#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#define TPB 32

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 520
#endif

#include "cuda_lyra2_vectors.h"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
__device__ void __threadfence_block();
#if __CUDA_ARCH__ >= 300
__device__ uint32_t __shfl_sync(uint32_t a, uint32_t b, uint32_t c);
#endif
#endif

#define Nrow 4
#define Ncol 4
#define memshift 3

__device__ uint2x4 *DState;

__device__ __forceinline__ uint2 LD4S(const int index)
{
	extern __shared__ uint2 shared_mem[];

	return shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
}

__device__ __forceinline__ void ST4S(const int index, const uint2 data)
{
	extern __shared__ uint2 shared_mem[];

	shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data;
}

__device__ __forceinline__
void Gfunc_v5(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; uint2 tmp = d; d.y = a.x ^ tmp.x; d.x = a.y ^ tmp.y;
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);
}

#if __CUDA_ARCH__ >= 300
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	return __shfl_sync(0xFFFFFFFF, a, b, c);
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	return make_uint2(__shfl_sync(0xFFFFFFFF, a.x, b, c), __shfl_sync(0xFFFFFFFF, a.y, b, c));
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	a1 = WarpShuffle(a1, b1, c);
	a2 = WarpShuffle(a2, b2, c);
	a3 = WarpShuffle(a3, b3, c);
}

#else
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;
	uint32_t *_ptr = (uint32_t*)shared_mem;

	__threadfence_block();
	uint32_t buf = _ptr[thread];

	_ptr[thread] = a;
	__threadfence_block();
	uint32_t result = _ptr[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	_ptr[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a;
	__threadfence_block();
	uint2 result = shared_mem[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a1;
	__threadfence_block();
	a1 = shared_mem[(thread&~(c - 1)) + (b1&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a2;
	__threadfence_block();
	a2 = shared_mem[(thread&~(c - 1)) + (b2&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a3;
	__threadfence_block();
	a3 = shared_mem[(thread&~(c - 1)) + (b3&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;
	__threadfence_block();
}

#endif


__device__ __forceinline__ void round_lyra(uint2 s[4])
{
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3, 4);
	Gfunc_v5(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1, 4);
}

__device__ __forceinline__
void round_lyra(uint2x4* s)
{
	Gfunc_v5(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v5(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v5(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v5(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v5(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v5(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v5(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v5(s[0].w, s[1].x, s[2].y, s[3].z);
}

__device__ __forceinline__ void reduceDuplexRowSetupV2(uint2 state[4], uint2 *DMatrix)
{
	int i, j;
	uint2 state1[Ncol][memshift], state0[Ncol][memshift], state2[memshift];

#pragma unroll
	for (int i = 0; i < Ncol; i++)
	{
		state0[Ncol - i - 1][0] = state[0];
		state0[Ncol - i - 1][1] = state[1];
		state0[Ncol - i - 1][2] = state[2];
		round_lyra(state);
	}

	//#pragma unroll 4
	for (i = 0; i < Ncol; i++)
	{
		state[0] ^= state0[i][0];
		state[1] ^= state0[i][1];
		state[2] ^= state0[i][2];

		round_lyra(state);

		state1[Ncol - i - 1][0] = state0[i][0] ^ state[0];
		state1[Ncol - i - 1][1] = state0[i][1] ^ state[1];
		state1[Ncol - i - 1][2] = state0[i][2] ^ state[2];
	}

	uint32_t s0 = 0 * Ncol*(memshift - 1);
	uint32_t s2 = 2 * Ncol*(memshift - 1) + (Ncol - 1)*(memshift - 1);

	for (i = 0; i < Ncol; i++)
	{
		state[0] ^= state1[i][0] + state0[i][0];
		state[1] ^= state1[i][1] + state0[i][1];
		state[2] ^= state1[i][2] + state0[i][2];

		round_lyra(state);

		state2[0] = state1[i][0] ^ state[0];
		state2[1] = state1[i][1] ^ state[1];
		state2[2] = state1[i][2] ^ state[2];

#if __CUDA_ARCH__ >= 520
		DMatrix[s2 >> 1] = state2[0];
		ST4S(s2 + 0, state2[1]);
		ST4S(s2 + 1, state2[2]);
#else
		DMatrix[s2 + 0] = state2[0];
		DMatrix[s2 + 1] = state2[1];
		ST4S(s2 >> 1, state2[2]);
#endif

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state0[i][0] ^= Data2;
			state0[i][1] ^= Data0;
			state0[i][2] ^= Data1;
		}
		else
		{
			state0[i][0] ^= Data0;
			state0[i][1] ^= Data1;
			state0[i][2] ^= Data2;
		}

#if __CUDA_ARCH__ >= 520
		DMatrix[s0 >> 1] = state0[i][0];
		ST4S(s0 + 0, state0[i][1]);
		ST4S(s0 + 1, state0[i][2]);
#else
		DMatrix[s0 + 0] = state0[i][0];
		DMatrix[s0 + 1] = state0[i][1];
		ST4S(s0 >> 1, state0[i][2]);
#endif

		state0[i][0] = state2[0];
		state0[i][1] = state2[1];
		state0[i][2] = state2[2];

		s0 += memshift - 1;
		s2 -= memshift - 1;
	}

	s2 += 2 * Ncol*(memshift - 1);
	// s0 = 1 * Ncol*(memshift - 1);
	// s2 = 3 * Ncol*(memshift - 1) + (Ncol - 1)*(memshift - 1);

	for (i = 0; i < Ncol; i++)
	{
		state[0] ^= state1[i][0] + state0[Ncol - i - 1][0];
		state[1] ^= state1[i][1] + state0[Ncol - i - 1][1];
		state[2] ^= state1[i][2] + state0[Ncol - i - 1][2];

		round_lyra(state);

		state0[Ncol - i - 1][0] ^= state[0];
		state0[Ncol - i - 1][1] ^= state[1];
		state0[Ncol - i - 1][2] ^= state[2];

#if __CUDA_ARCH__ >= 520
		DMatrix[s2 >> 1] = state0[Ncol - i - 1][0];
		ST4S(s2 + 0, state0[Ncol - i - 1][1]);
		ST4S(s2 + 1, state0[Ncol - i - 1][2]);
#else
		DMatrix[s2 + 0] = state0[Ncol - i - 1][0];
		DMatrix[s2 + 1] = state0[Ncol - i - 1][1];
		ST4S(s2 >> 1, state0[Ncol - i - 1][2]);
#endif

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state1[i][0] ^= Data2;
			state1[i][1] ^= Data0;
			state1[i][2] ^= Data1;
		}
		else
		{
			state1[i][0] ^= Data0;
			state1[i][1] ^= Data1;
			state1[i][2] ^= Data2;
		}

#if __CUDA_ARCH__ >= 520
		DMatrix[s0 >> 1] = state1[i][0];
		ST4S(s0 + 0, state1[i][1]);
		ST4S(s0 + 1, state1[i][2]);
#else
		DMatrix[s0 + 0] = state1[i][0];
		DMatrix[s0 + 1] = state1[i][1];
		ST4S(s0 >> 1, state1[i][2]);
#endif

		s0 += memshift - 1;
		s2 -= memshift - 1;
	}
}

__device__ __forceinline__ void reduceDuplexRowSetupV2_eco(uint2 state[4])
{
	int i, j;
	uint2 state1[Ncol][memshift], state0[Ncol][memshift], state2[memshift];

#pragma unroll
	for (int i = 0; i < Ncol; i++)
	{
		state0[Ncol - i - 1][0] = state[0];
		state0[Ncol - i - 1][1] = state[1];
		state0[Ncol - i - 1][2] = state[2];
		round_lyra(state);
	}

	//#pragma unroll 4
	for (i = 0; i < Ncol; i++)
	{
		state[0] ^= state0[i][0];
		state[1] ^= state0[i][1];
		state[2] ^= state0[i][2];

		round_lyra(state);

		state1[Ncol - i - 1][0] = state0[i][0] ^ state[0];
		state1[Ncol - i - 1][1] = state0[i][1] ^ state[1];
		state1[Ncol - i - 1][2] = state0[i][2] ^ state[2];
	}

	uint32_t s0 = 0 * Ncol * memshift;
	uint32_t s2 = 2 * Ncol * memshift + (Ncol - 1) * memshift;

	for (i = 0; i < Ncol; i++)
	{
		state[0] ^= state1[i][0] + state0[i][0];
		state[1] ^= state1[i][1] + state0[i][1];
		state[2] ^= state1[i][2] + state0[i][2];

		round_lyra(state);

		state2[0] = state1[i][0] ^ state[0];
		state2[1] = state1[i][1] ^ state[1];
		state2[2] = state1[i][2] ^ state[2];

		ST4S(s2 + 0, state2[0]);
		ST4S(s2 + 1, state2[1]);
		ST4S(s2 + 2, state2[2]);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state0[i][0] ^= Data2;
			state0[i][1] ^= Data0;
			state0[i][2] ^= Data1;
		}
		else
		{
			state0[i][0] ^= Data0;
			state0[i][1] ^= Data1;
			state0[i][2] ^= Data2;
		}

		ST4S(s0 + 0, state0[i][0]);
		ST4S(s0 + 1, state0[i][1]);
		ST4S(s0 + 2, state0[i][2]);

		state0[i][0] = state2[0];
		state0[i][1] = state2[1];
		state0[i][2] = state2[2];

		s0 += memshift;
		s2 -= memshift;
	}

	s2 += 2 * Ncol * memshift;
	// s0 = 1 * Ncol * memshift;
	// s2 = 3 * Ncol * memshift + (Ncol - 1) * memshift;

	for (i = 0; i < Ncol; i++)
	{
		state[0] ^= state1[i][0] + state0[Ncol - i - 1][0];
		state[1] ^= state1[i][1] + state0[Ncol - i - 1][1];
		state[2] ^= state1[i][2] + state0[Ncol - i - 1][2];

		round_lyra(state);

		state0[Ncol - i - 1][0] ^= state[0];
		state0[Ncol - i - 1][1] ^= state[1];
		state0[Ncol - i - 1][2] ^= state[2];

		ST4S(s2 + 0, state0[Ncol - i - 1][0]);
		ST4S(s2 + 1, state0[Ncol - i - 1][1]);
		ST4S(s2 + 2, state0[Ncol - i - 1][2]);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state1[i][0] ^= Data2;
			state1[i][1] ^= Data0;
			state1[i][2] ^= Data1;
		}
		else
		{
			state1[i][0] ^= Data0;
			state1[i][1] ^= Data1;
			state1[i][2] ^= Data2;
		}

		ST4S(s0 + 0, state1[i][0]);
		ST4S(s0 + 1, state1[i][1]);
		ST4S(s0 + 2, state1[i][2]);

		s0 += memshift;
		s2 -= memshift;
	}
}

__device__ void reduceDuplexRowtV2(uint2 state[4], uint2 *DMatrix)
{
	uint32_t rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;

	uint2 state2[memshift], state1[memshift];
	uint32_t s1 = 3 * Ncol * (memshift - 1);
	uint32_t s2 = rowInOut * Ncol * (memshift - 1);
	uint32_t s3 = 0 * Ncol * (memshift - 1);

	for (int i = 0; i < Ncol; i++)
	{
#if __CUDA_ARCH__ >= 520
		state2[0] = DMatrix[s2 >> 1];
		state2[1] = LD4S(s2 + 0);
		state2[2] = LD4S(s2 + 1);

		state[0] ^= DMatrix[s1 >> 1] + state2[0];
		state[1] ^= LD4S(s1 + 0) + state2[1];
		state[2] ^= LD4S(s1 + 1) + state2[2];
#else
		state2[0] = DMatrix[s2 + 0];
		state2[1] = DMatrix[s2 + 1];
		state2[2] = LD4S(s2 >> 1);

		state[0] ^= DMatrix[s1 + 0] + state2[0];
		state[1] ^= DMatrix[s1 + 1] + state2[1];
		state[2] ^= LD4S(s1 >> 1) + state2[2];
#endif

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

#if __CUDA_ARCH__ >= 520
		DMatrix[s2 >> 1] = state2[0];
		ST4S(s2 + 0, state2[1]);
		ST4S(s2 + 1, state2[2]);

		DMatrix[s3 >> 1] ^= state[0];
		ST4S(s3 + 0, LD4S(s3 + 0) ^ state[1]);
		ST4S(s3 + 1, LD4S(s3 + 1) ^ state[2]);
#else
		DMatrix[s2 + 0] = state2[0];
		DMatrix[s2 + 1] = state2[1];
		ST4S(s2 >> 1, state2[2]);

		DMatrix[s3 + 0] ^= state[0];
		DMatrix[s3 + 1] ^= state[1];
		ST4S(s3 >> 1, LD4S(s3 >> 1) ^ state[2]);
#endif

		s1 += memshift - 1;
		s2 += memshift - 1;
		s3 += memshift - 1;
	}

	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;

	s1 = 0 * Ncol * (memshift - 1);
	s2 = rowInOut * Ncol * (memshift - 1);
	// s3 = 1        * Ncol * (memshift - 1);

	for (int i = 0; i < Ncol; i++)
	{
#if __CUDA_ARCH__ >= 520
		state2[0] = DMatrix[s2 >> 1];
		state2[1] = LD4S(s2 + 0);
		state2[2] = LD4S(s2 + 1);

		state[0] ^= DMatrix[s1 >> 1] + state2[0];
		state[1] ^= LD4S(s1 + 0) + state2[1];
		state[2] ^= LD4S(s1 + 1) + state2[2];
#else
		state2[0] = DMatrix[s2 + 0];
		state2[1] = DMatrix[s2 + 1];
		state2[2] = LD4S(s2 >> 1);

		state[0] ^= DMatrix[s1 + 0] + state2[0];
		state[1] ^= DMatrix[s1 + 1] + state2[1];
		state[2] ^= LD4S(s1 >> 1) + state2[2];
#endif

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

#if __CUDA_ARCH__ >= 520
		DMatrix[s2 >> 1] = state2[0];
		ST4S(s2 + 0, state2[1]);
		ST4S(s2 + 1, state2[2]);

		DMatrix[s3 >> 1] ^= state[0];
		ST4S(s3 + 0, LD4S(s3 + 0) ^ state[1]);
		ST4S(s3 + 1, LD4S(s3 + 1) ^ state[2]);
#else
		DMatrix[s2 + 0] = state2[0];
		DMatrix[s2 + 1] = state2[1];
		ST4S(s2 >> 1, state2[2]);

		DMatrix[s3 + 0] ^= state[0];
		DMatrix[s3 + 1] ^= state[1];
		ST4S(s3 >> 1, LD4S(s3 >> 1) ^ state[2]);
#endif

		s1 += memshift - 1;
		s2 += memshift - 1;
		s3 += memshift - 1;
	}

	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;

	// s1 = 1        * Ncol * (memshift - 1);
	s2 = rowInOut * Ncol * (memshift - 1);
	// s3 = 2        * Ncol * (memshift - 1);

	for (int i = 0; i < Ncol; i++)
	{
#if __CUDA_ARCH__ >= 520
		state2[0] = DMatrix[s2 >> 1];
		state2[1] = LD4S(s2 + 0);
		state2[2] = LD4S(s2 + 1);

		state[0] ^= DMatrix[s1 >> 1] + state2[0];
		state[1] ^= LD4S(s1 + 0) + state2[1];
		state[2] ^= LD4S(s1 + 1) + state2[2];
#else
		state2[0] = DMatrix[s2 + 0];
		state2[1] = DMatrix[s2 + 1];
		state2[2] = LD4S(s2 >> 1);

		state[0] ^= DMatrix[s1 + 0] + state2[0];
		state[1] ^= DMatrix[s1 + 1] + state2[1];
		state[2] ^= LD4S(s1 >> 1) + state2[2];
#endif

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

#if __CUDA_ARCH__ >= 520
		DMatrix[s2 >> 1] = state2[0];
		ST4S(s2 + 0, state2[1]);
		ST4S(s2 + 1, state2[2]);

		DMatrix[s3 >> 1] ^= state[0];
		ST4S(s3 + 0, LD4S(s3 + 0) ^ state[1]);
		ST4S(s3 + 1, LD4S(s3 + 1) ^ state[2]);
#else
		DMatrix[s2 + 0] = state2[0];
		DMatrix[s2 + 1] = state2[1];
		ST4S(s2 >> 1, state2[2]);

		DMatrix[s3 + 0] ^= state[0];
		DMatrix[s3 + 1] ^= state[1];
		ST4S(s3 >> 1, LD4S(s3 >> 1) ^ state[2]);
#endif

		s1 += memshift - 1;
		s2 += memshift - 1;
		s3 += memshift - 1;
	}

	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;
	// s1 = 2        * Ncol * (memshift - 1);
	s2 = rowInOut * Ncol * (memshift - 1);
	// s3 = 3        * Ncol * (memshift - 1);

#if __CUDA_ARCH__ >= 520
	state2[0] = DMatrix[s2 >> 1];
	state2[1] = LD4S(s2 + 0);
	state2[2] = LD4S(s2 + 1);

	state[0] ^= DMatrix[s1 >> 1] + state2[0];
	state[1] ^= LD4S(s1 + 0) + state2[1];
	state[2] ^= LD4S(s1 + 1) + state2[2];
#else
	state2[0] = DMatrix[s2 + 0];
	state2[1] = DMatrix[s2 + 1];
	state2[2] = LD4S(s2 >> 1);

	state[0] ^= DMatrix[s1 + 0] + state2[0];
	state[1] ^= DMatrix[s1 + 1] + state2[1];
	state[2] ^= LD4S(s1 >> 1) + state2[2];
#endif

	round_lyra(state);

	//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
	uint2 Data0 = state[0];
	uint2 Data1 = state[1];
	uint2 Data2 = state[2];
	WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

	if (threadIdx.x == 0)
	{
		state2[0] ^= Data2;
		state2[1] ^= Data0;
		state2[2] ^= Data1;
	}
	else
	{
		state2[0] ^= Data0;
		state2[1] ^= Data1;
		state2[2] ^= Data2;
	}

	if (rowInOut == 3)
	{
		state2[0] ^= state[0];
		state2[1] ^= state[1];
		state2[2] ^= state[2];
	}

	s1 += memshift - 1;
	s2 += memshift - 1;

	for (int i = 1; i < Ncol; i++)
	{
#if __CUDA_ARCH__ >= 520
		state[0] ^= DMatrix[s1 >> 1] + DMatrix[s2 >> 1];
		state[1] ^= LD4S(s1 + 0) + LD4S(s2 + 0);
		state[2] ^= LD4S(s1 + 1) + LD4S(s2 + 1);
#else
		state[0] ^= DMatrix[s1 + 0] + DMatrix[s2 + 0];
		state[1] ^= DMatrix[s1 + 1] + DMatrix[s2 + 1];
		state[2] ^= LD4S(s1 >> 1) + LD4S(s2 >> 1);
#endif

		round_lyra(state);

		s1 += memshift - 1;
		s2 += memshift - 1;
	}

#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= state2[j];
}

__device__ void reduceDuplexRowtV2_eco(uint2 state[4])
{
	uint32_t rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;

	uint2 state2[memshift], state1[memshift];
	uint32_t s1 = 3 * Ncol * memshift;
	uint32_t s2 = rowInOut * Ncol * memshift;
	uint32_t s3 = 0 * Ncol * memshift;

	for (int i = 0; i < Ncol; i++)
	{
		state2[0] = LD4S(s2 + 0);
		state2[1] = LD4S(s2 + 1);
		state2[2] = LD4S(s2 + 2);

		state[0] ^= LD4S(s1 + 0) + state2[0];
		state[1] ^= LD4S(s1 + 1) + state2[1];
		state[2] ^= LD4S(s1 + 2) + state2[2];

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		ST4S(s2 + 0, state2[0]);
		ST4S(s2 + 1, state2[1]);
		ST4S(s2 + 2, state2[2]);

		ST4S(s3 + 0, LD4S(s3 + 0) ^ state[0]);
		ST4S(s3 + 1, LD4S(s3 + 1) ^ state[1]);
		ST4S(s3 + 2, LD4S(s3 + 2) ^ state[2]);

		s1 += memshift;
		s2 += memshift;
		s3 += memshift;
	}

	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;
	s1 = 0 * Ncol * memshift;
	s2 = rowInOut * Ncol * memshift;
	// s3 = 1        * Ncol * memshift;

	for (int i = 0; i < Ncol; i++)
	{
		state2[0] = LD4S(s2 + 0);
		state2[1] = LD4S(s2 + 1);
		state2[2] = LD4S(s2 + 2);

		state[0] ^= LD4S(s1 + 0) + state2[0];
		state[1] ^= LD4S(s1 + 1) + state2[1];
		state[2] ^= LD4S(s1 + 2) + state2[2];

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		ST4S(s2 + 0, state2[0]);
		ST4S(s2 + 1, state2[1]);
		ST4S(s2 + 2, state2[2]);

		ST4S(s3 + 0, LD4S(s3 + 0) ^ state[0]);
		ST4S(s3 + 1, LD4S(s3 + 1) ^ state[1]);
		ST4S(s3 + 2, LD4S(s3 + 2) ^ state[2]);

		s1 += memshift;
		s2 += memshift;
		s3 += memshift;
	}

	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;

	// s1 = 1        * Ncol * memshift;
	s2 = rowInOut * Ncol * memshift;
	// s3 = 2        * Ncol * memshift;

	for (int i = 0; i < Ncol; i++)
	{
		state2[0] = LD4S(s2 + 0);
		state2[1] = LD4S(s2 + 1);
		state2[2] = LD4S(s2 + 2);

		state[0] ^= LD4S(s1 + 0) + state2[0];
		state[1] ^= LD4S(s1 + 1) + state2[1];
		state[2] ^= LD4S(s1 + 2) + state2[2];

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		ST4S(s2 + 0, state2[0]);
		ST4S(s2 + 1, state2[1]);
		ST4S(s2 + 2, state2[2]);

		ST4S(s3 + 0, LD4S(s3 + 0) ^ state[0]);
		ST4S(s3 + 1, LD4S(s3 + 1) ^ state[1]);
		ST4S(s3 + 2, LD4S(s3 + 2) ^ state[2]);

		s1 += memshift;
		s2 += memshift;
		s3 += memshift;
	}

	rowInOut = WarpShuffle(state[0].x, 0, 4) & 3;
	// s1 = 2        * Ncol * memshift;
	s2 = rowInOut * Ncol * memshift;
	// s3 = 3        * Ncol * memshift;

	state2[0] = LD4S(s2 + 0);
	state2[1] = LD4S(s2 + 1);
	state2[2] = LD4S(s2 + 2);

	state[0] ^= LD4S(s1 + 0) + state2[0];
	state[1] ^= LD4S(s1 + 1) + state2[1];
	state[2] ^= LD4S(s1 + 2) + state2[2];

	round_lyra(state);

	//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
	uint2 Data0 = state[0];
	uint2 Data1 = state[1];
	uint2 Data2 = state[2];
	WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

	if (threadIdx.x == 0)
	{
		state2[0] ^= Data2;
		state2[1] ^= Data0;
		state2[2] ^= Data1;
	}
	else
	{
		state2[0] ^= Data0;
		state2[1] ^= Data1;
		state2[2] ^= Data2;
	}

	if (rowInOut == 3)
	{
		state2[0] ^= state[0];
		state2[1] ^= state[1];
		state2[2] ^= state[2];
	}

	s1 += memshift;
	s2 += memshift;

	for (int i = 1; i < Ncol; i++)
	{
		state[0] ^= LD4S(s1 + 0) + LD4S(s2 + 0);
		state[1] ^= LD4S(s1 + 1) + LD4S(s2 + 1);
		state[2] ^= LD4S(s1 + 2) + LD4S(s2 + 2);

		round_lyra(state);

		s1 += memshift;
		s2 += memshift;
	}

	state[0] ^= state2[0];
	state[1] ^= state2[1];
	state[2] ^= state2[2];
}

__constant__ uint28 blake2b_IV[2] = {
	0xf3bcc908lu, 0x6a09e667lu,
	0x84caa73blu, 0xbb67ae85lu,
	0xfe94f82blu, 0x3c6ef372lu,
	0x5f1d36f1lu, 0xa54ff53alu,
	0xade682d1lu, 0x510e527flu,
	0x2b3e6c1flu, 0x9b05688clu,
	0xfb41bd6blu, 0x1f83d9ablu,
	0x137e2179lu, 0x5be0cd19lu
};

__constant__ uint28 Mask[2] = {
	0x00000020lu, 0x00000000lu,
	0x00000020lu, 0x00000000lu,
	0x00000020lu, 0x00000000lu,
	0x00000001lu, 0x00000000lu,
	0x00000004lu, 0x00000000lu,
	0x00000004lu, 0x00000000lu,
	0x00000080lu, 0x00000000lu,
	0x00000000lu, 0x01000000lu
};

__global__ __launch_bounds__(64, 1)
void lyra2v2_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint28 state[4];

	if (thread < threads)
	{
		state[0].x = state[1].x = __ldg(&outputHash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&outputHash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&outputHash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&outputHash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

#pragma unroll 2
		for (int i = 0; i<12; i++)
			round_lyra(state);

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];

#pragma unroll 2
		for (int i = 0; i<12; i++)
			round_lyra(state);

		outputHash[thread + threads * 0] = state[0].x;
		outputHash[thread + threads * 1] = state[0].y;
		outputHash[thread + threads * 2] = state[0].z;
		outputHash[thread + threads * 3] = state[0].w;
		DState[blockDim.x * gridDim.x * 0 + thread] = state[1];
		DState[blockDim.x * gridDim.x * 1 + thread] = state[2];
		DState[blockDim.x * gridDim.x * 2 + thread] = state[3];

	} //thread
}

__global__ __launch_bounds__(TPB, 1)
void lyra2v2_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = outputHash[thread + threads * threadIdx.x];
		state[1] = ((uint2*)DState)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[2] = ((uint2*)DState)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[3] = ((uint2*)DState)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];

#if __CUDA_ARCH__ >= 520
		uint2 DMatrix[16];
#else
		uint2 DMatrix[32];
#endif

		reduceDuplexRowSetupV2(state, DMatrix);

		reduceDuplexRowtV2(state, DMatrix);

		outputHash[thread + threads * threadIdx.x] = state[0];
		((uint2*)DState)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[1];
		((uint2*)DState)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[2];
		((uint2*)DState)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[3];
	} //thread
}

__global__ __launch_bounds__(TPB, 1)
void lyra2v2_gpu_hash_32_2_eco(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = outputHash[thread + threads * threadIdx.x];
		state[1] = ((uint2*)DState)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[2] = ((uint2*)DState)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[3] = ((uint2*)DState)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];

		reduceDuplexRowSetupV2_eco(state);

		reduceDuplexRowtV2_eco(state);

		outputHash[thread + threads * threadIdx.x] = state[0];
		((uint2*)DState)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[1];
		((uint2*)DState)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[2];
		((uint2*)DState)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[3];
	} //thread
}

__global__ __launch_bounds__(64, 1)
void lyra2v2_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint28 state[4];

	if (thread < threads)
	{
		state[0].x = __ldg(&outputHash[thread + threads * 0]);
		state[0].y = __ldg(&outputHash[thread + threads * 1]);
		state[0].z = __ldg(&outputHash[thread + threads * 2]);
		state[0].w = __ldg(&outputHash[thread + threads * 3]);
		state[1] = __ldg4(&DState[blockDim.x * gridDim.x * 0 + thread]);
		state[2] = __ldg4(&DState[blockDim.x * gridDim.x * 1 + thread]);
		state[3] = __ldg4(&DState[blockDim.x * gridDim.x * 2 + thread]);

#pragma unroll 2
		for (int i = 0; i < 12; i++)
			round_lyra(state);

		outputHash[thread + threads * 0] = state[0].x;
		outputHash[thread + threads * 1] = state[0].y;
		outputHash[thread + threads * 2] = state[0].z;
		outputHash[thread + threads * 3] = state[0].w;

	} //thread
}

__host__
void lyra2v2_cpu_init(int thr_id, uint64_t *d_matrix)
{
	int dev_id = device_map[thr_id % MAX_GPUS];
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DState), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}

__host__
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, bool eco_mode)
{
	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = TPB;

	dim3 grid1((threads * 4 + tpb - 1) / tpb);
	dim3 block1(4, tpb >> 2);

	dim3 grid2((threads + 64 - 1) / 64);
	dim3 block2(64);

	if (cuda_arch[dev_id] < 500)
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(lyra2v2_gpu_hash_32_2), hipFuncCachePreferShared);
	else if (cuda_arch[dev_id] >= 700)
		hipFuncSetAttribute(reinterpret_cast<const void*>(lyra2v2_gpu_hash_32_2), hipFuncAttributePreferredSharedMemoryCarveout, 100);

	lyra2v2_gpu_hash_32_1 << <grid2, block2 >> > (threads, startNounce, (uint2*)g_hash);

	if (eco_mode)
		lyra2v2_gpu_hash_32_2_eco << <grid1, block1, 48 * sizeof(uint2) * tpb >> > (threads, startNounce, (uint2*)g_hash);
	else if (cuda_arch[dev_id] >= 520)
		lyra2v2_gpu_hash_32_2 << <grid1, block1, 32 * sizeof(uint2) * tpb >> > (threads, startNounce, (uint2*)g_hash);
	else
		lyra2v2_gpu_hash_32_2 << <grid1, block1, 16 * sizeof(uint2) * tpb >> > (threads, startNounce, (uint2*)g_hash);

	lyra2v2_gpu_hash_32_3 << <grid2, block2 >> > (threads, startNounce, (uint2*)g_hash);
	//MyStreamSynchronize(NULL, order, thr_id);
}
