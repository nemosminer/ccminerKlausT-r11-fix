#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}
#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

// Speicher zur Generierung der Noncevektoren f�r die bedingten Hashes
static uint32_t *d_branch1Nonces[MAX_GPUS];
static uint32_t *d_branch2Nonces[MAX_GPUS];
static uint32_t *d_branch3Nonces[MAX_GPUS];

extern void quark_blake512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_bmw512_cpu_init(int thr_id, uint32_t threads);
extern void quark_bmw512_cpu_setBlock_80(int thr_id, void *pdata);
extern void quark_bmw512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_outputHash);
extern void quark_bmw512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_outputHash);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_doublegroestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_keccak512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_jh512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
											uint32_t *d_nonces1, uint32_t *nrm1,
											uint32_t *d_nonces2, uint32_t *nrm2);
extern void quark_compactTest_single_false_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *inpHashes, uint32_t *d_validNonceTable,
											uint32_t *d_nonces1, uint32_t *nrm1);

extern uint32_t cuda_check_hash_branch(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash);
extern void cuda_check_quarkcoin(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, uint32_t *foundnonces);

/* CPU Hash */
extern "C" void animehash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_bmw512_context ctx_bmw;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    unsigned char hash[64];

    sph_bmw512_init(&ctx_bmw);
    // ZBMW;
    sph_bmw512 (&ctx_bmw, (const void*) input, 80);
    sph_bmw512_close(&ctx_bmw, (void*) hash);

    sph_blake512_init(&ctx_blake);
    // ZBLAKE;
    sph_blake512 (&ctx_blake, hash, 64);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    if (hash[0] & 0x8)
    {
        sph_groestl512_init(&ctx_groestl);
        // ZGROESTL;
        sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
        sph_groestl512_close(&ctx_groestl, (void*) hash);
    }
    else
    {
        sph_skein512_init(&ctx_skein);
        // ZSKEIN;
        sph_skein512 (&ctx_skein, (const void*) hash, 64);
        sph_skein512_close(&ctx_skein, (void*) hash);
    }
    
    sph_groestl512_init(&ctx_groestl);
    // ZGROESTL;
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    // ZJH;
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_blake512_init(&ctx_blake);
        // ZBLAKE;
        sph_blake512 (&ctx_blake, (const void*) hash, 64);
        sph_blake512_close(&ctx_blake, (void*) hash);
    }
    else
    {
        sph_bmw512_init(&ctx_bmw);
        // ZBMW;
        sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
        sph_bmw512_close(&ctx_bmw, (void*) hash);
    }

    sph_keccak512_init(&ctx_keccak);
    // ZKECCAK;
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    // SKEIN;
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    if (hash[0] & 0x8)
    {
        sph_keccak512_init(&ctx_keccak);
        // ZKECCAK;
        sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
        sph_keccak512_close(&ctx_keccak, (void*) hash);
    }
    else
    {
        sph_jh512_init(&ctx_jh);
        // ZJH;
        sph_jh512 (&ctx_jh, (const void*) hash, 64);
        sph_jh512_close(&ctx_jh, (void*) hash);
    }

    memcpy(state, hash, 32);
}

/*
struct HashPredicate
{    
    HashPredicate(uint32_t *hashes, uint32_t startNonce) :
        m_hashes(hashes),
        m_startNonce(startNonce)
        { }

    __device__
    bool operator()(const uint32_t x)
    {
        uint32_t *const Hash = &m_hashes[(x - m_startNonce)*16];
        return hash[0] & 0x8;
    }

    uint32_t *m_hashes;
    uint32_t  m_startNonce;
};
*/

static volatile bool init[MAX_GPUS] = { false };

extern int scanhash_anime(int thr_id, uint32_t *pdata,
    uint32_t *ptarget, uint32_t max_nonce,
    uint32_t *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, 1 << 20); // 256*256*8
	throughput = min(throughput, (max_nonce - first_nonce)) & 0xfffffc00;

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	if (!init[thr_id])
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		get_cuda_arch(&cuda_arch[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id);
		quark_bmw512_cpu_init(thr_id, throughput);
		cuda_check_cpu_init(thr_id, throughput);
		quark_compactTest_cpu_init(thr_id, throughput);

		CUDA_SAFE_CALL(hipMalloc(&d_branch1Nonces[thr_id], sizeof(uint32_t)*throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_branch2Nonces[thr_id], sizeof(uint32_t)*throughput));
		CUDA_SAFE_CALL(hipMalloc(&d_branch3Nonces[thr_id], sizeof(uint32_t)*throughput));

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_bmw512_cpu_setBlock_80(thr_id, (void*)endiandata);
	cuda_check_cpu_setTarget(ptarget, thr_id);

	do {
		uint32_t nrm1=0, nrm2=0, nrm3=0;

		// erstes BMW512 Hash mit CUDA
		quark_bmw512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);

		// das ist der unbedingte Branch f�r Blake512
		quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id]);

		quark_compactTest_single_false_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], NULL,
				d_branch3Nonces[thr_id], &nrm3);
		
		// nur den Skein Branch weiterverfolgen
		quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// das ist der unbedingte Branch f�r Groestl512
		quark_groestl512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// das ist der unbedingte Branch f�r JH512
		quark_jh512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// quarkNonces in branch1 und branch2 aufsplitten gem�ss if (hash[0] & 0x8)
		quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2);

		// das ist der bedingte Branch f�r Blake512
		quark_blake512_cpu_hash_64(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id]);

		// das ist der bedingte Branch f�r Bmw512
		quark_bmw512_cpu_hash_64(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id]);

		// das ist der unbedingte Branch f�r Keccak512
		quark_keccak512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// das ist der unbedingte Branch f�r Skein512
		quark_skein512_cpu_hash_64(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id]);

		// quarkNonces in branch1 und branch2 aufsplitten gem�ss if (hash[0] & 0x8)
		quark_compactTest_cpu_hash_64(thr_id, nrm3, pdata[19], d_hash[thr_id], d_branch3Nonces[thr_id],
			d_branch1Nonces[thr_id], &nrm1,
			d_branch2Nonces[thr_id], &nrm2);

		quark_keccak512_cpu_hash_64_final(thr_id, nrm1, pdata[19], d_branch1Nonces[thr_id], d_hash[thr_id]);
		quark_jh512_cpu_hash_64_final(thr_id, nrm2, pdata[19], d_branch2Nonces[thr_id], d_hash[thr_id]);

		uint32_t foundnonces[2];
		cuda_check_quarkcoin(thr_id, nrm3, pdata[19], d_branch3Nonces[thr_id], d_hash[thr_id], foundnonces);
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundnonces[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundnonces[0]);
			animehash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				// check if there was some other ones...
				if (foundnonces[1] != 0xffffffff)
				{
					be32enc(&endiandata[19], foundnonces[1]);
					animehash(vhash64, endiandata);
					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						pdata[21] = foundnonces[1];
						res++;
						if (opt_benchmark)  applog(LOG_INFO, "GPU #%d: Found second nonce $%08X", device_map[thr_id], foundnonces[1]);
					}
				}
				pdata[19] = foundnonces[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: Found nonce $%08X", device_map[thr_id], foundnonces[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundnonces[0]);
			}
		}
		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
