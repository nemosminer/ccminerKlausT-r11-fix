#include "hip/hip_runtime.h"
// Auf QuarkCoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"
#include "cuda_vector.h"

#define TPB 512
#define THF 4

// aus cpu-miner.c
//extern short device_map[8];


// diese Struktur wird in der Init Funktion angefordert
//static hipDeviceProp_t props[8];

// 64 Register Variante f�r Compute 3.0
#include "groestl_functions_quad.cu"
#include "bitslice_transformations_quad.cu"

__global__ __launch_bounds__(TPB, 2)
void quark_groestl512_gpu_hash_64_quad(uint32_t threads, uint32_t startNounce, uint32_t *const __restrict__ g_hash, const uint32_t *const __restrict__ g_nonceVector)
{
	uint32_t __align__(16) msgBitsliced[8];
	uint32_t __align__(16) state[8];
	uint32_t __align__(16) hash[16];
	// durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
    const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x) >> 2;
    if (thread < threads)
    {
        // GROESTL
        const uint32_t nounce = g_nonceVector ? g_nonceVector[thread] : (startNounce + thread);
		const uint32_t hashPosition = nounce - startNounce;
        uint32_t *const inpHash = &g_hash[hashPosition * 16];

        const uint32_t thr = threadIdx.x & (THF-1);

		uint32_t message[8] =
		{
			inpHash[thr], inpHash[(THF)+thr], inpHash[(2 * THF) + thr], inpHash[(3 * THF) + thr],0, 0, 0, 
		};
		if (thr == 0) message[4] = 0x80UL;
		if (thr == 3) message[7] = 0x01000000UL;

		to_bitslice_quad(message, msgBitsliced);

        groestl512_progressMessage_quad(state, msgBitsliced);

		from_bitslice_quad(state, hash);

		if (thr == 0)
		{
			uint28 *phash = (uint28*)hash;
			uint28 *outpt = (uint28*)inpHash; /* var kept for hash align */
			outpt[0] = phash[0];
			outpt[1] = phash[1];
//			outpt[2] = phash[2];
//			outpt[3] = phash[3];
		}
    }
}


__host__ void quark_groestl512_cpu_init(int thr_id, uint32_t threads)
{
//    hipGetDeviceProperties(&props[thr_id], device_map[thr_id]);
}

__host__ void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash)
{

    // berechne wie viele Thread Blocks wir brauchen
	dim3 grid(THF*((threads + TPB - 1) / TPB));
	dim3 block(TPB);

    quark_groestl512_gpu_hash_64_quad<<<grid, block, 0, gpustream[thr_id]>>>(threads, startNounce, d_hash, d_nonceVector);
	CUDA_SAFE_CALL(hipGetLastError());
}

