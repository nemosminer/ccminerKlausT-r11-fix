#include "hip/hip_runtime.h"
/**
* SKEIN512 80 + SHA256 64
* by tpruvot@github - 2015
* Optimized by sp-hash@github - 2015
*/

extern "C" {
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include <openssl/sha.h>

extern void skein512_cpu_setBlock_80(int thr_id,void *pdata);
extern void skein512_cpu_hash_80_6x(int thr_id, uint32_t threads, uint32_t startNounce, int swapu, uint64_t target, uint32_t *h_found);
extern void skein512_cpu_hash_80_50(int thr_id, uint32_t threads, uint32_t startNounce, int swapu, uint64_t target, uint32_t *h_found);
extern void skein512_cpu_hash_80_52(int thr_id, uint32_t threads, uint32_t startNounce, int swapu, uint64_t target, uint32_t *h_found);

void skeincoinhash(void *output, const void *input)
{
	sph_skein512_context ctx_skein;
	SHA256_CTX sha256;

	uint32_t hash[16];

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, hash);

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, (unsigned char *)hash, 64);
	SHA256_Final((unsigned char *)hash, &sha256);

	memcpy(output, hash, 32);
}

static __inline uint32_t swab32_if(uint32_t val, bool iftrue)
{
	return iftrue ? swab32(val) : val;
}

int scanhash_skeincoin(int thr_id, uint32_t *pdata,
								  uint32_t *ptarget, uint32_t max_nonce,
								  uint32_t *hashes_done)
{
	static THREAD uint32_t *foundnonces = nullptr;

	const uint32_t first_nonce = pdata[19];
	const int swap = 1;

	uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 28 : 1 << 27;;
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	uint32_t throughput = min(throughputmax, max_nonce - first_nonce) & 0xfffffc00;

	if (opt_benchmark)
	{
		((uint64_t*)ptarget)[3] = 0x3000f0000;
	}
	uint64_t target = ((uint64_t*)ptarget)[3];

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		CUDA_SAFE_CALL(hipHostMalloc(&foundnonces, 2 * 4));
		mining_has_stopped[thr_id] = false;
		init = true;
	}

	uint32_t endiandata[20];
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80(thr_id, (void*)endiandata);
	do
	{
		*hashes_done = pdata[19] - first_nonce + throughput;

		if(device_sm[device_map[thr_id]] >= 600)
			skein512_cpu_hash_80_6x(thr_id, throughput, pdata[19], swap, target, foundnonces);
		else
			if(device_sm[device_map[thr_id]] > 500)
				skein512_cpu_hash_80_52(thr_id, throughput, pdata[19], swap, target, foundnonces);
			else
				skein512_cpu_hash_80_50(thr_id, throughput, pdata[19], swap, target, foundnonces);

		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundnonces[0] != 0xffffffff)
		{
			uint32_t vhash64[8]={0};

			endiandata[19] = swab32_if(foundnonces[0], swap);
			
			skeincoinhash(vhash64, endiandata);

			uint64_t test = ((uint64_t*)vhash64)[3];
			if (test <= target && fulltest(vhash64, ptarget))
			{
				int res = 1;
				if (opt_debug || opt_benchmark)
					applog(LOG_INFO, "GPU #%d: found nonce $%08X", device_map[thr_id], foundnonces[0]);
				if (foundnonces[1] != 0xffffffff)
				{
					endiandata[19] = swab32_if(foundnonces[1], swap);
					skeincoinhash(vhash64, endiandata);
					uint64_t test2 = ((uint64_t*)vhash64)[3];
					if (test2 <= target && fulltest(vhash64, ptarget))
					{
						if (opt_debug || opt_benchmark)
							applog(LOG_INFO, "GPU #%d: found nonce $%08X", device_map[thr_id], foundnonces[1]);
						pdata[19 + res] = swab32_if(foundnonces[1], !swap);
						res++;
					}
					else
					{
						if (test2 != target) applog(LOG_WARNING, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundnonces[1]);
					}
				}
				pdata[19] = swab32_if(foundnonces[0], !swap);
				return res;
			}
			else 
			{
				if (test != target)
					applog(LOG_WARNING, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundnonces[0]);
				else
					applog(LOG_WARNING, "Lost work: #%d", test);

			}
		}

		pdata[19] += throughput;

	} while(!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
