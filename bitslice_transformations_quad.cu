
#include <hip/hip_runtime.h>

#define merge8(z, x, y, b)\
		z=__byte_perm(x, y, b); \

#define SWAP8(x,y)\
		x=__byte_perm(x, y, 0x5410); \
		y=__byte_perm(x, y, 0x7632);

#define SWAP4(x,y)\
		t = 0xf0f0f0f0UL & (x ^ (y<<4)); \
		x = (x ^ t); \
		t=  t>>4;\
		y=  y ^ t;

#ifndef NOASM
#define SWAP4_final(x,y)\
	asm("and.b32 %0, %0, 0x0f0f0f0f;"\
	    "and.b32 %1, %1, 0x0f0f0f0f;"\
	    "vshl.u32.u32.u32.clamp.add %0, %1, 4, %0;\n\t"\
	    : "+r"(x) : "r"(y));
#else
#define SWAP4_final(x,y)\
	t = 0xf0f0f0f0UL & (x ^ (y << 4)); \
	x = (x ^ (0xf0f0f0f0UL & (x ^ (y << 4)))); 
#endif

#define SWAP2(x,y)\
		t = 0xccccccccUL & (x ^ (y<<2)); \
		x = (x ^ t); \
		t=  t>>2;\
		y=  y ^ t;

#define SWAP1(x,y)\
		t = 0xaaaaaaaaUL & (x ^ (y<<1)); \
		x = (x ^ t); \
		t = t>>1;\
		y = y ^ t;

static __device__ __forceinline__
void to_bitslice_quad(uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{
	uint32_t other[8];
	uint32_t t;

	const uint32_t perm = (threadIdx.x & 1) ? 0x7362 : 0x5140;
	const unsigned int n = threadIdx.x & 3;
#pragma unroll
	for(int i = 0; i < 4; i++)
	{
		input[i] = __shfl_sync(0xffffffff, (int)input[i], n ^ (3 * (n >= 1 && n <= 2)), 4);
		other[i] = __shfl_sync(0xffffffff, (int)input[i], (threadIdx.x + 1) & 3, 4);
		input[i] = __shfl_sync(0xffffffff, (int)input[i], threadIdx.x & 2, 4);
		other[i] = __shfl_sync(0xffffffff, (int)other[i], threadIdx.x & 2, 4);
	}

	if((threadIdx.x & 3) < 2)
	{
		input[4] = 0x80;
	}
	else
	{
		input[4] = 0;
	}

	if((threadIdx.x & 3) > 1)
		other[7] = 0x01000000;
	else
		other[7] = 0;
	input[7] = 0;

	merge8(output[0], input[0], input[4], perm);
	merge8(output[1], other[0],        0, perm);
	merge8(output[2], input[1],        0, perm);
	merge8(output[3], other[1],        0, perm);
	merge8(output[4], input[2],        0, perm);
	merge8(output[5], other[2],        0, perm);
	merge8(output[6], input[3],        0, perm);
	merge8(output[7], other[3], other[7], perm);

	SWAP1(output[0], output[1]);
	SWAP1(output[2], output[3]);
	SWAP1(output[4], output[5]);
	SWAP1(output[6], output[7]);

	SWAP2(output[0], output[2]);
	SWAP2(output[1], output[3]);
	SWAP2(output[4], output[6]);
	SWAP2(output[5], output[7]);

	SWAP4(output[0], output[4]);
	SWAP4(output[1], output[5]);
	SWAP4(output[2], output[6]);
	SWAP4(output[3], output[7]);
}

static __device__ __forceinline__
void myr_to_bitslice_quad(uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{
	uint32_t other[8];
	uint32_t t;

	const uint32_t perm = (threadIdx.x & 1) ? 0x7362 : 0x5140;
	const unsigned int n = threadIdx.x & 3;
#pragma unroll
	for(int i = 0; i < 5; i++)
	{
		input[i] = __shfl_sync(0xffffffff, (int)input[i], n ^ (3 * (n >= 1 && n <= 2)), 4);
		other[i] = __shfl_sync(0xffffffff, (int)input[i], (threadIdx.x + 1) & 3, 4);
		input[i] = __shfl_sync(0xffffffff, (int)input[i], threadIdx.x & 2, 4);
		other[i] = __shfl_sync(0xffffffff, (int)other[i], threadIdx.x & 2, 4);
	}
	if(n < 2)
	{
		input[5] = 0x80;
		other[7] = 0;
	}
	else
	{
		input[5] = 0;
		other[7] = 0x01000000;
	}

	merge8(output[0], input[0], input[4], perm);
	merge8(output[1], other[0], other[4], perm);
	merge8(output[2], input[1], input[5], perm);
	output[3] = __byte_perm(other[1], 0, perm);
	output[4] = __byte_perm(input[2], 0, perm);
	output[5] = __byte_perm(other[2], 0, perm);
	output[6] = __byte_perm(input[3], 0, perm);
	merge8(output[7], other[3], other[7], perm);

	SWAP1(output[0], output[1]);
	SWAP1(output[2], output[3]);
	SWAP1(output[4], output[5]);
	SWAP1(output[6], output[7]);

	SWAP2(output[0], output[2]);
	SWAP2(output[1], output[3]);
	SWAP2(output[4], output[6]);
	SWAP2(output[5], output[7]);

	SWAP4(output[0], output[4]);
	SWAP4(output[1], output[5]);
	SWAP4(output[2], output[6]);
	SWAP4(output[3], output[7]);
}

static __device__ __forceinline__
void from_bitslice_quad(const uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{
	uint32_t t;
	const uint32_t perm = 0x7531;//(threadIdx.x & 1) ? 0x3175 : 0x7531;

	output[0] = __byte_perm(input[0], input[4], perm);
	output[2] = __byte_perm(input[1], input[5], perm);
	output[8] = __byte_perm(input[2], input[6], perm);
	output[10] = __byte_perm(input[3], input[7], perm);

	SWAP1(output[0], output[2]);
	SWAP1(output[8], output[10]);

	SWAP2(output[0], output[8]);
	SWAP2(output[2], output[10]);

	output[4] = __byte_perm(output[0], output[8], 0x5410);
	output[8] = __byte_perm(output[0], output[8], 0x7632);
	output[0] = output[4];

	output[6] = __byte_perm(output[2], output[10], 0x5410);
	output[10] = __byte_perm(output[2], output[10], 0x7632);
	output[2] = output[6];

	SWAP4(output[0], output[8]);
	SWAP4(output[2], output[10]);

	if(threadIdx.x & 1)
	{
		output[14] = __byte_perm(output[10], 0, 0x3232);
		output[12] = __byte_perm(output[8], 0, 0x3232);
		output[6] = __byte_perm(output[2], 0, 0x3232);
		output[4] = __byte_perm(output[0], 0, 0x3232);

		output[0] = __byte_perm(output[0], 0, 0x1032);
		output[2] = __byte_perm(output[2], 0, 0x1032);
		output[8] = __byte_perm(output[8], 0, 0x1032);
		output[10] = __byte_perm(output[10], 0, 0x1032);
	}
	else
	{
		output[4] = output[0];
		output[6] = output[2];
		output[12] = output[8];
		output[14] = output[10];
	}

	output[0] = __byte_perm(output[0], __shfl_sync(0xffffffff, (int)output[0], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[2] = __byte_perm(output[2], __shfl_sync(0xffffffff, (int)output[2], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[4] = __byte_perm(output[4], __shfl_sync(0xffffffff, (int)output[4], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[6] = __byte_perm(output[6], __shfl_sync(0xffffffff, (int)output[6], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[8] = __byte_perm(output[8], __shfl_sync(0xffffffff, (int)output[8], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[10] = __byte_perm(output[10], __shfl_sync(0xffffffff, (int)output[10], (threadIdx.x + 1) & 3, 4), 0x7610);
	output[12] = __byte_perm(output[12], __shfl_sync(0xffffffff, (int)output[12], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[14] = __byte_perm(output[14], __shfl_sync(0xffffffff, (int)output[14], (threadIdx.x + 1) & 3, 4), 0x7632);

	output[0 + 1] = __shfl_sync(0xffffffff, (int)output[0], (threadIdx.x + 2) & 3, 4);
	output[2 + 1] = __shfl_sync(0xffffffff, (int)output[2], (threadIdx.x + 2) & 3, 4);
	output[4 + 1] = __shfl_sync(0xffffffff, (int)output[4], (threadIdx.x + 2) & 3, 4);
	output[6 + 1] = __shfl_sync(0xffffffff, (int)output[6], (threadIdx.x + 2) & 3, 4);
	output[8 + 1] = __shfl_sync(0xffffffff, (int)output[8], (threadIdx.x + 2) & 3, 4);
	output[10 + 1] = __shfl_sync(0xffffffff, (int)output[10], (threadIdx.x + 2) & 3, 4);
	output[12 + 1] = __shfl_sync(0xffffffff, (int)output[12], (threadIdx.x + 2) & 3, 4);
	output[14 + 1] = __shfl_sync(0xffffffff, (int)output[14], (threadIdx.x + 2) & 3, 4);

}

static __device__ __forceinline__
void from_bitslice_quad_final(const uint32_t *const __restrict__ input, uint32_t *const __restrict__ output)
{
	uint32_t t;
	const uint32_t perm = 0x7531;//(threadIdx.x & 1) ? 0x3175 : 0x7531;

	output[0] = __byte_perm(input[0], input[4], perm);
	output[2] = __byte_perm(input[1], input[5], perm);
	output[8] = __byte_perm(input[2], input[6], perm);
	output[10] = __byte_perm(input[3], input[7], perm);

	SWAP1(output[0], output[2]);
	SWAP1(output[8], output[10]);

	SWAP2(output[2], output[10]);

	output[6] = __byte_perm(output[2], output[10], 0x5410);
	output[10] = __byte_perm(output[2], output[10], 0x7632);

	if(threadIdx.x & 3)
	{
		SWAP4_final(output[6], output[10]);
		output[6] = __byte_perm(output[6], 0, 0x3232);
	}
	else
	{
		output[2] = output[6];

		SWAP4(output[2], output[10]);

		if(threadIdx.x & 1)
		{
			output[6] = __byte_perm(output[2], 0, 0x3232);
		}
		else
		{
			output[6] = output[2];
		}
	}

	output[6] = __byte_perm(output[6], __shfl_sync(0xffffffff, (int)output[6], (threadIdx.x + 1) & 3, 4), 0x7632);
	output[7] = __shfl_sync(0xffffffff, (int)output[6], (threadIdx.x + 2) & 3, 4);

}
