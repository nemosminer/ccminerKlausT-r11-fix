#include "hip/hip_runtime.h"
/*
 * qubit algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
}

#include "miner.h"

#include "cuda_helper.h"

extern void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void qubit_luffa512_cpu_setBlock_80(int thr_id, void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void qubit_luffa512_cpufinal_setBlock_80(int thr_id, void *pdata, const void *ptarget);
extern uint32_t qubit_luffa512_cpu_finalhash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void doomhash(void *state, const void *input)
{
	// luffa512
	sph_luffa512_context ctx_luffa;

	uint8_t hash[64];

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	memcpy(state, hash, 32);
}

extern int scanhash_doom(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	static THREAD uint32_t *d_hash = nullptr;

	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, 1U << 22); // 256*256*8*8
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;

	if (opt_benchmark)
		ptarget[7] = 0x0000f;

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == 1<<22)
			applog(LOG_INFO, "GPU #%d: using default intensity 22", device_map[thr_id]);
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (16 * sizeof(uint32_t)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughputmax));

		qubit_luffa512_cpu_init(thr_id, (int) throughputmax);
		mining_has_stopped[thr_id] = false;

		init = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	qubit_luffa512_cpufinal_setBlock_80(thr_id, (void*)endiandata,ptarget);

	do {

		uint32_t foundNonce = qubit_luffa512_cpu_finalhash_80(thr_id, (int) throughput, pdata[19], d_hash);
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify){ be32enc(&endiandata[19], foundNonce);
			doomhash(vhash64, endiandata);

			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				*hashes_done = min(max_nonce - first_nonce, (uint64_t) pdata[19] - first_nonce + throughput);
				pdata[19] = foundNonce;
				return 1;
			}
			else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
