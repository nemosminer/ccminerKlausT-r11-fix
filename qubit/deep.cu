#include "hip/hip_runtime.h"
/*
 * deepcoin algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"

#include "cuda_helper.h"

extern void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void qubit_luffa512_cpu_setBlock_80(int thr_id, void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void qubit_luffa512_cpufinal_setBlock_80(int thr_id, void *pdata, const void *ptarget);
extern uint32_t qubit_luffa512_cpu_finalhash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x11_echo512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *d_hash, uint32_t target, uint32_t *h_found);

void deephash(void *state, const void *input)
{
	// luffa1-cubehash2-shavite3-simd4-echo5
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_echo512_context ctx_echo;

	uint8_t hash[64];

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(state, hash, 32);
}

extern int scanhash_deep(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	static THREAD uint32_t *d_hash = nullptr;
	static THREAD uint32_t *h_found = nullptr;

	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, 1U << 19); // 256*256*8
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	static THREAD volatile bool init = false;
	if (!init)
	{
		if(throughputmax == 1<<19)
			applog(LOG_INFO, "GPU #%d: using default intensity 19", device_map[thr_id]);
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (16 * sizeof(uint32_t)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughputmax));

		qubit_luffa512_cpu_init(thr_id, throughputmax);
		x11_echo512_cpu_init(thr_id, throughputmax);
		CUDA_SAFE_CALL(hipHostMalloc(&(h_found), 4 * sizeof(uint32_t)));

		cuda_check_cpu_init(thr_id, throughputmax);
		mining_has_stopped[thr_id] = false;

		init = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	qubit_luffa512_cpufinal_setBlock_80(thr_id, (void*)endiandata,ptarget);
	cuda_check_cpu_setTarget(ptarget, thr_id);

	do {

		qubit_luffa512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x11_echo512_cpu_hash_64_final(thr_id, throughput, pdata[19], d_hash, ptarget[7], h_found);
		hipStreamSynchronize(gpustream[thr_id]);
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(h_found[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify){ be32enc(&endiandata[19], h_found[0]);
			deephash(vhash64, endiandata);

			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_found[1] != 0xffffffff)
				{
					if(opt_verify){ be32enc(&endiandata[19], h_found[1]);
					deephash(vhash64, endiandata);
					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						pdata[21] = h_found[1];
						res++;
						if (opt_benchmark)
							applog(LOG_INFO, "GPU #%d Found second nounce %08x", device_map[thr_id], h_found[1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[1]);
						}
					}

				}
				pdata[19] = h_found[0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", device_map[thr_id], h_found[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[0]);
				}
			}
		}
		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
