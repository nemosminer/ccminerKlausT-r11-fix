#include "hip/hip_runtime.h"
/*
 * qubit algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"
#include "cuda_helper.h"

extern void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void qubit_luffa512_cpu_setBlock_80(int thr_id, void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_cubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern int x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash,const uint32_t simdthreads);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
//extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x11_echo512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *d_hash, uint32_t target, uint32_t *h_found);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *inpHashes,
											const uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse);

void qubithash(void *state, const void *input)
{
	// luffa1-cubehash2-shavite3-simd4-echo5

	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	uint8_t hash[64];

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512 (&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(state, hash, 32);
}

extern int scanhash_qubit(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	static THREAD uint32_t *d_hash = nullptr;
	static THREAD uint32_t *h_found = nullptr;

	uint32_t endiandata[20];
	const uint32_t first_nonce = pdata[19];

	uint32_t intensity = 256 * 256 * 10;
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 256 : 32;

	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);
	if(strstr(props.name, "1080"))
	{
		intensity = 256 * 256 * 24;
	}
	else if(strstr(props.name, "1070"))
	{
		intensity = 256 * 256 * 24;
	}
	else if(strstr(props.name, "970"))
	{
		intensity = 256 * 256 * 16;
	}
	else if (strstr(props.name, "980"))
	{
		intensity = 256 * 256 * 24;
	}
	else if (strstr(props.name, "750 Ti"))
	{
		intensity = 256 * 256 * 12;
	}
	else if (strstr(props.name, "750"))
	{
		intensity = 256 * 256 * 10;
	}
	else if (strstr(props.name, "960"))
	{
		intensity = 256 * 256 * 16;
	}
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, intensity);
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;

	if (opt_benchmark)
		ptarget[7] = 0x0000ff;

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));

#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (64 * sizeof(uint4)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		qubit_luffa512_cpu_init(thr_id, throughputmax);
		x11_simd512_cpu_init(thr_id, throughputmax);
		x11_echo512_cpu_init(thr_id, throughputmax);

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughputmax));
		CUDA_SAFE_CALL(hipHostMalloc(&(h_found), 4 * sizeof(uint32_t)));
		mining_has_stopped[thr_id] = false;

		init = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	qubit_luffa512_cpu_setBlock_80(thr_id, (void*)endiandata);

	do {

		// Hash with CUDA
		qubit_luffa512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x11_simd512_cpu_hash_64(thr_id,throughput, pdata[19], d_hash,simdthreads);
		x11_echo512_cpu_hash_64_final(thr_id, throughput, pdata[19], d_hash, ptarget[7], h_found);
		hipStreamSynchronize(gpustream[thr_id]);
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(h_found[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify){ be32enc(&endiandata[19], h_found[0]);
			qubithash(vhash64, endiandata);

			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_found[1] != 0xffffffff)
				{
					if(opt_verify){ be32enc(&endiandata[19], h_found[1]);
					qubithash(vhash64, endiandata);
					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						pdata[21] = h_found[1];
						res++;
						if (opt_benchmark)
							applog(LOG_INFO, "GPU #%d Found second nounce %08x", device_map[thr_id], h_found[1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[1]);
						}
					}

				}
				pdata[19] = h_found[0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", device_map[thr_id], h_found[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[0]);
				}
			}
		}
		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
