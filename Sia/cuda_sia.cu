#include "hip/hip_runtime.h"
/*
Copyright (c) 2015 KlausT and Vorksholk

Permission to use, copy, modify, and/or distribute this software for any
purpose with or without fee is hereby granted, provided that the above
copyright notice and this permission notice appear in all copies.

THE SOFTWARE IS PROVIDED "AS IS" AND THE AUTHOR DISCLAIMS ALL WARRANTIES WITH
REGARD TO THIS SOFTWARE INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS. IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY SPECIAL, DIRECT,
INDIRECT, OR CONSEQUENTIAL DAMAGES OR ANY DAMAGES WHATSOEVER RESULTING FROM
LOSS OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR
PERFORMANCE OF THIS SOFTWARE.
*/


#include <stdint.h>
#include "cuda_helper.h"
#include "sia.h"

#ifdef _MSC_VER
#define THREAD __declspec(thread)
#else
#define THREAD __thread
#endif

#ifdef __INTELLISENSE__
#define __launch_bounds__(blocksize)
#endif

static THREAD uint64_t *vpre_h;
static THREAD uint32_t *nonceOut_d;
static THREAD uint64_t *hash_d;
__constant__ uint64_t vpre[16];
__constant__ uint64_t header[10];

__device__ __forceinline__
static uint64_t __byte_perm_64(const uint64_t source, const uint32_t grab1, const uint32_t grab2)
{
	uint64_t r;
	uint32_t r1;
	uint32_t r2;

	uint32_t i1;
	uint32_t i2;

	asm("mov.b64 {%0, %1}, %2;" : "=r"(i1), "=r"(i2) : "l"(source));
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r1) : "r"(i1), "r"(i2), "r"(grab1));
	asm("prmt.b32 %0, %1, %2, %3;" : "=r"(r2) : "r"(i1), "r"(i2), "r"(grab2));
	asm("mov.b64 %0, {%1, %2};" : "=l"(r) : "r"(r1), "r"(r2));

	return r;
}

__device__ __forceinline__
static uint64_t __swap_hilo(const uint64_t source)
{
	uint64_t r;
	uint32_t s1;
	uint32_t s2;

	asm("mov.b64 {%0, %1}, %2;" : "=r"(s1), "=r"(s2) : "l"(source));
	asm("mov.b64 %0, {%1, %2};" : "=l"(r) : "r"(s2), "r"(s1));

	return r;
}

__device__ unsigned int numberofresults;

__global__ void __launch_bounds__(blocksize, 3) siakernel(uint32_t * __restrict__ nonceOut, uint64_t target, uint64_t startnonce)
{
	uint64_t v[16];
	const uint64_t start = startnonce + (blockDim.x * blockIdx.x + threadIdx.x)*npt;
	const uint64_t end = start + npt;

	numberofresults = 0;

	for(uint64_t n = start; n < end; n++)
	{
		v[2] = 0x5BF2CD1EF9D6B596u + n; v[14] = __swap_hilo(~0x1f83d9abfb41bd6bu ^ v[2]); v[10] = 0x3c6ef372fe94f82bu + v[14]; v[6] = __byte_perm_64(0x1f83d9abfb41bd6bu ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[5]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = 0x130C253729B586Au + header[6]; v[15] = __swap_hilo(0x5be0cd19137e2179u ^ v[3]); v[11] = 0xa54ff53a5f1d36f1u + v[15]; v[7] = __byte_perm_64(0x5be0cd19137e2179u ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[7]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = vpre[0] + vpre[5] + header[8]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(vpre[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[9]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = vpre[1] + v[6];          v[12] = __swap_hilo(vpre[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = __swap_hilo(vpre[13] ^ v[2]); v[8] = vpre[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + vpre[4];          v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = vpre[9] + v[14]; v[4] = __byte_perm_64(vpre[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + n;         v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[8]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[9]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[6]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[1]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[0]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[2]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[7]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[5]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[3]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[8]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];             v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[0]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[5]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[2]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[3]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[6]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[7]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[1]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[9]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + n;         v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[7]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[9]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[3]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[1]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[2]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[6]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[5]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + n;         v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[0]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[8]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[9]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[0]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[5]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[7]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[2]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + n;         v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[1]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[6]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[8]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[3]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[2]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[6]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[0]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[8]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[3]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + n;         v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[7]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[5]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[1]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[9]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[5]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[1]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + n;         v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7];             v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[0]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[7]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[6]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[3]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[9]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[2]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[8]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[7]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5];             v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[1]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[3]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[9]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[5]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[0]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + n;         v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[8]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + header[6]; v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4] + header[2]; v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[6]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5];             v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[9]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6];             v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[3]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[0]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[8]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[2]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6] + header[7]; v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[1]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7] + n;         v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[5]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[2]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[8]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + n;         v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[7]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[6]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[1]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[5]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5];             v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5];             v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6] + header[9]; v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7] + header[3]; v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4] + header[0]; v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4] + header[0]; v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4] + header[1]; v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + header[2]; v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[3]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + n;         v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6] + header[5]; v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7] + header[6]; v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[7]; v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[8]; v[15] = __swap_hilo(v[15] ^ v[0]); v[10] = v[10] + v[15]; v[5] = __byte_perm_64(v[5] ^ v[10], 0x6543, 0x2107);
		v[0] = v[0] + v[5] + header[9]; v[15] = __byte_perm_64(v[15] ^ v[0], 0x5432, 0x1076); v[10] = v[10] + v[15]; v[5] = ROTR64(v[5] ^ v[10], 63);
		v[1] = v[1] + v[6];             v[12] = __swap_hilo(v[12] ^ v[1]); v[11] = v[11] + v[12]; v[6] = __byte_perm_64(v[6] ^ v[11], 0x6543, 0x2107);
		v[1] = v[1] + v[6];             v[12] = __byte_perm_64(v[12] ^ v[1], 0x5432, 0x1076); v[11] = v[11] + v[12]; v[6] = ROTR64(v[6] ^ v[11], 63);
		v[2] = v[2] + v[7];             v[13] = __swap_hilo(v[13] ^ v[2]); v[8] = v[8] + v[13]; v[7] = __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107);
		v[2] = v[2] + v[7];             v[13] = __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076); v[8] = v[8] + v[13]; v[7] = ROTR64(v[7] ^ v[8], 63);
		v[3] = v[3] + v[4];             v[14] = __swap_hilo(v[14] ^ v[3]); v[9] = v[9] + v[14]; v[4] = __byte_perm_64(v[4] ^ v[9], 0x6543, 0x2107);
		v[3] = v[3] + v[4];             v[14] = __byte_perm_64(v[14] ^ v[3], 0x5432, 0x1076); v[9] = v[9] + v[14]; v[4] = ROTR64(v[4] ^ v[9], 63);

		v[0] = v[0] + v[4];             v[12] = __swap_hilo(v[12] ^ v[0]); v[8] = v[8] + v[12]; v[4] = __byte_perm_64(v[4] ^ v[8], 0x6543, 0x2107);
		v[0] = v[0] + v[4];             v[12] = __byte_perm_64(v[12] ^ v[0], 0x5432, 0x1076); v[8] = v[8] + v[12]; v[4] = ROTR64(v[4] ^ v[8], 63);
		v[1] = v[1] + v[5] + n;         v[13] = __swap_hilo(v[13] ^ v[1]); v[9] = v[9] + v[13]; v[5] = __byte_perm_64(v[5] ^ v[9], 0x6543, 0x2107);
		v[1] = v[1] + v[5] + header[8]; v[13] = __byte_perm_64(v[13] ^ v[1], 0x5432, 0x1076); v[9] = v[9] + v[13]; v[5] = ROTR64(v[5] ^ v[9], 63);
		v[2] = v[2] + v[6] + header[9]; v[14] = __swap_hilo(v[14] ^ v[2]); v[10] = v[10] + v[14]; v[6] = __byte_perm_64(v[6] ^ v[10], 0x6543, 0x2107);
		v[2] = v[2] + v[6];             v[14] = __byte_perm_64(v[14] ^ v[2], 0x5432, 0x1076); v[10] = v[10] + v[14]; v[6] = ROTR64(v[6] ^ v[10], 63);
		v[3] = v[3] + v[7];             v[15] = __swap_hilo(v[15] ^ v[3]); v[11] = v[11] + v[15]; v[7] = __byte_perm_64(v[7] ^ v[11], 0x6543, 0x2107);
		v[3] = v[3] + v[7] + header[6];	v[15] = __byte_perm_64(v[15] ^ v[3], 0x5432, 0x1076); v[11] = v[11] + v[15]; v[7] = ROTR64(v[7] ^ v[11], 63);
		v[0] = v[0] + v[5] + header[1];
		v[0] = v[0] + __byte_perm_64(v[5] ^ (v[10] + __swap_hilo(v[15] ^ v[0])), 0x6543, 0x2107);
		v[2] = v[2] + v[7];
		v[13] = __swap_hilo(v[13] ^ v[2]);
		v[8] = v[8] + v[13];
		v[2] = v[2] + __byte_perm_64(v[7] ^ v[8], 0x6543, 0x2107) + header[7];

		if(cuda_swab64(0x6A09E667F2BDC928 ^ v[0] ^ (v[8] + __byte_perm_64(v[13] ^ v[2], 0x5432, 0x1076))) < target)
		{
			int i = atomicAdd(&numberofresults, 1);
			if(i < MAXRESULTS)
				nonceOut[i] = n & 0xffffffff;
			return;
		}
	}
}

void sia_gpu_hash(hipStream_t cudastream, int thr_id, uint32_t threads, uint32_t *nonceOut, uint64_t target, uint64_t startnonce)
{
	siakernel << <threads / blocksize / npt, blocksize, 0, cudastream >> >(nonceOut_d, target, startnonce);
	CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipMemcpyAsync(nonceOut, nonceOut_d, 4 * MAXRESULTS, hipMemcpyDeviceToHost, cudastream));
	CUDA_SAFE_CALL(hipStreamSynchronize(cudastream));
}

void sia_gpu_init(int thr_id)
{
	CUDA_SAFE_CALL(hipHostMalloc(&vpre_h, 16 * 8));
	CUDA_SAFE_CALL(hipMalloc(&nonceOut_d, MAXRESULTS * 4));
	CUDA_SAFE_CALL(hipMalloc(&hash_d, 4 * 8));
}

void sia_precalc(int thr_id, hipStream_t cudastream, const uint64_t *blockHeader)
{
	vpre_h[0] = 0xBB1838E7A0A44BF9u + blockHeader[0]; vpre_h[12] = ROTR64(0x510E527FADE68281u ^ vpre_h[0], 32); vpre_h[8] = 0x6a09e667f3bcc908u + vpre_h[12]; vpre_h[4] = ROTR64(0x510e527fade682d1u ^ vpre_h[8], 24);
	vpre_h[0] = vpre_h[0] + vpre_h[4] + blockHeader[1];       vpre_h[12] = ROTR64(vpre_h[12] ^ vpre_h[0], 16);              vpre_h[8] = vpre_h[8] + vpre_h[12];               vpre_h[4] = ROTR64(vpre_h[4] ^ vpre_h[8], 63);
	vpre_h[1] = 0x566D1711B009135Au + blockHeader[2]; vpre_h[13] = ROTR64(0x9b05688c2b3e6c1fu ^ vpre_h[1], 32); vpre_h[9] = 0xbb67ae8584caa73bu + vpre_h[13]; vpre_h[5] = ROTR64(0x9b05688c2b3e6c1fu ^ vpre_h[9], 24);
	vpre_h[1] = vpre_h[1] + vpre_h[5] + blockHeader[3];       vpre_h[13] = ROTR64(vpre_h[13] ^ vpre_h[1], 16);              vpre_h[9] = vpre_h[9] + vpre_h[13];               vpre_h[5] = ROTR64(vpre_h[5] ^ vpre_h[9], 63);

	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(vpre), vpre_h, 16 * 8, 0, hipMemcpyHostToDevice, cudastream));
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(header), blockHeader, 10 * 8, 0, hipMemcpyHostToDevice, cudastream));
	CUDA_SAFE_CALL(hipMemsetAsync(nonceOut_d, 0, 4 * MAXRESULTS, cudastream));
}