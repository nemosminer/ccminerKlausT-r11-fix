#include "hip/hip_runtime.h"
/*-
* blake2b C code from https://github.com/SiaMining/sgminer/blob/master/algorithm/sia.c
*
* Copyright 2009 Colin Percival, 2014 savale
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
* 1. Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
* 2. Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*
* THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
* ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
* ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
* OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
* HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
* LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
* OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
* SUCH DAMAGE.
*
* This file was originally written by Colin Percival as part of the Tarsnap
* online backup system.
*/
#include "miner.h"
#include "cuda_helper.h"
#include <cstdio>
using namespace std;
#include <hip/hip_runtime_api.h>
#include "sia.h"

extern void applog_hex(void *data, int len);
extern bool fulltest_sia(const uint64_t *hash, const uint64_t *target);

#define B2B_GET64(p)                            \
    (((uint64_t) ((uint8_t *) (p))[0]) ^        \
    (((uint64_t) ((uint8_t *) (p))[1]) << 8) ^  \
    (((uint64_t) ((uint8_t *) (p))[2]) << 16) ^ \
    (((uint64_t) ((uint8_t *) (p))[3]) << 24) ^ \
    (((uint64_t) ((uint8_t *) (p))[4]) << 32) ^ \
    (((uint64_t) ((uint8_t *) (p))[5]) << 40) ^ \
    (((uint64_t) ((uint8_t *) (p))[6]) << 48) ^ \
    (((uint64_t) ((uint8_t *) (p))[7]) << 56))

#define B2B_G(a, b, c, d, x, y) {   \
    v[a] = v[a] + v[b] + x;         \
    v[d] = ROTR64(v[d] ^ v[a], 32); \
    v[c] = v[c] + v[d];             \
    v[b] = ROTR64(v[b] ^ v[c], 24); \
    v[a] = v[a] + v[b] + y;         \
    v[d] = ROTR64(v[d] ^ v[a], 16); \
    v[c] = v[c] + v[d];             \
    v[b] = ROTR64(v[b] ^ v[c], 63); }

static const uint64_t blake2b_iv[8] =
{
	0x6A09E667F3BCC908, 0xBB67AE8584CAA73B,
	0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
	0x510E527FADE682D1, 0x9B05688C2B3E6C1F,
	0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
};

typedef struct
{
	uint8_t b[128];                     // input buffer
	uint64_t h[8];                      // chained state
	uint64_t t[2];                      // total number of bytes
	size_t c;                           // pointer for b[]
	size_t outlen;                      // digest size
} blake2b_ctx;

void blake2b_update(blake2b_ctx *ctx, const void *in, size_t inlen);

static void blake2b_compress(blake2b_ctx *ctx, int last)
{
	const uint8_t sigma[12][16] =
	{
		{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
		{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3},
		{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4},
		{7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8},
		{9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13},
		{2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9},
		{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11},
		{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10},
		{6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5},
		{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0},
		{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
		{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3}
	};
	int i;
	uint64_t v[16], m[16];

	for(i = 0; i < 8; i++)
	{           // init work variables
		v[i] = ctx->h[i];
		v[i + 8] = blake2b_iv[i];
	}

	v[12] ^= ctx->t[0];                 // low 64 bits of offset
	v[13] ^= ctx->t[1];                 // high 64 bits
	if(last)                           // last block flag set ?
		v[14] = ~v[14];

	for(i = 0; i < 16; i++)            // get little-endian words
		m[i] = B2B_GET64(&ctx->b[8 * i]);

	for(i = 0; i < 12; i++)
	{          // twelve rounds
		B2B_G(0, 4, 8, 12, m[sigma[i][0]], m[sigma[i][1]]);
		B2B_G(1, 5, 9, 13, m[sigma[i][2]], m[sigma[i][3]]);
		B2B_G(2, 6, 10, 14, m[sigma[i][4]], m[sigma[i][5]]);
		B2B_G(3, 7, 11, 15, m[sigma[i][6]], m[sigma[i][7]]);
		B2B_G(0, 5, 10, 15, m[sigma[i][8]], m[sigma[i][9]]);
		B2B_G(1, 6, 11, 12, m[sigma[i][10]], m[sigma[i][11]]);
		B2B_G(2, 7, 8, 13, m[sigma[i][12]], m[sigma[i][13]]);
		B2B_G(3, 4, 9, 14, m[sigma[i][14]], m[sigma[i][15]]);
	}

	for(i = 0; i < 8; ++i)
		ctx->h[i] ^= v[i] ^ v[i + 8];
}

// Initialize the hashing context "ctx" with optional key "key".
//      1 <= outlen <= 64 gives the digest size in bytes.
//      Secret key (also <= 64 bytes) is optional (keylen = 0).
int blake2b_init(blake2b_ctx *ctx, size_t outlen, const void *key, size_t keylen)        // (keylen=0: no key)
{
	size_t i;

	if(outlen == 0 || outlen > 64 || keylen > 64)
		return -1;                      // illegal parameters

	for(i = 0; i < 8; i++)             // state, "param block"
		ctx->h[i] = blake2b_iv[i];
	ctx->h[0] ^= 0x01010000 ^ (keylen << 8) ^ outlen;

	ctx->t[0] = 0;                      // input count low word
	ctx->t[1] = 0;                      // input count high word
	ctx->c = 0;                         // pointer within buffer
	ctx->outlen = outlen;

	for(i = keylen; i < 128; i++)      // zero input block
		ctx->b[i] = 0;
	if(keylen > 0)
	{
		blake2b_update(ctx, key, keylen);
		ctx->c = 128;                   // at the end
	}

	return 0;
}

// Add "inlen" bytes from "in" into the hash.
void blake2b_update(blake2b_ctx *ctx,	const void *in, size_t inlen)
{
	size_t i;

	for(i = 0; i < inlen; i++)
	{
		if(ctx->c == 128)
		{            // buffer full ?
			ctx->t[0] += ctx->c;        // add counters
			if(ctx->t[0] < ctx->c)     // carry overflow ?
				ctx->t[1]++;            // high word
			blake2b_compress(ctx, 0);   // compress (not last)
			ctx->c = 0;                 // counter to zero
		}
		ctx->b[ctx->c++] = ((const uint8_t *)in)[i];
	}
}

// Generate the message digest (size given in init).
//      Result placed in "out".
void blake2b_final(blake2b_ctx *ctx, void *out)
{
	size_t i;

	ctx->t[0] += ctx->c;                // mark last block offset
	if(ctx->t[0] < ctx->c)             // carry overflow
		ctx->t[1]++;                    // high word

	while(ctx->c < 128)                // fill up with zeros
		ctx->b[ctx->c++] = 0;
	blake2b_compress(ctx, 1);           // final block flag = 1

	// little endian convert and store
	for(i = 0; i < ctx->outlen; i++)
	{
		((uint8_t *)out)[i] =
			(ctx->h[i >> 3] >> (8 * (i & 7))) & 0xFF;
	}
}

void siahash(const void *data, unsigned int len, void *hash)
{
	blake2b_ctx ctx;
	blake2b_init(&ctx, 32, NULL, 0);
	blake2b_update(&ctx, data, len);
	blake2b_final(&ctx, hash);
}

/***************************************************************************/

int scanhash_sia(int thr_id, uint32_t *pdata, uint32_t *ptarget, uint32_t max_nonce, uint32_t *hashes_done)
{
	static THREAD uint32_t *h_nounce = nullptr;
	const uint32_t first_nonce = pdata[8];
	static THREAD uint32_t throughputmax;

	if(opt_benchmark)
		ptarget[7] = 0x00000001;

	static THREAD volatile bool init = false;
	if(!init)
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		CUDA_SAFE_CALL(hipHostMalloc(&h_nounce, MAXRESULTS * sizeof(uint32_t)));
		sia_gpu_init(thr_id);

		throughputmax = device_intensity(device_map[thr_id], __func__, 1U << 28);
		if(throughputmax == 1<<28)
			applog(LOG_INFO, "GPU #%d: using default intensity 28", device_map[thr_id]);
		mining_has_stopped[thr_id] = false;
		init = true;
	}
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce));
	throughput -= throughput % (blocksize*npt);

	sia_precalc(thr_id, gpustream[thr_id], (uint64_t *)pdata);

	uint32_t endiandata[20];
	for(int k = 0; k < 20; k++)
		le32enc(&endiandata[k], pdata[k]);

	do
	{
		sia_gpu_hash(gpustream[thr_id], thr_id, throughput, h_nounce, ((uint64_t*)ptarget)[3], ((uint64_t*)pdata)[4]);
		if(stop_mining)
		{
			hipDeviceSynchronize();
			hipStreamDestroy(gpustream[thr_id]);
			hipProfilerStop();
			mining_has_stopped[thr_id] = true;
			pthread_exit(nullptr);
		}
		if(h_nounce[0] != 0)
		{
			const uint64_t Htarg = ((uint64_t*)ptarget)[3];
			uint64_t vhash64[4] = {0};
			if(opt_verify)
			{
				le32enc(&endiandata[8], h_nounce[0]);
				siahash(endiandata, 80, vhash64);
			}
			if(swab64(vhash64[0]) <= Htarg && fulltest_sia(vhash64, (uint64_t*)ptarget))
			{
				int res = 1;
				*hashes_done = pdata[8] - first_nonce + throughput;
				if(opt_benchmark || opt_debug)  applog(LOG_INFO, "GPU #%d: Found nonce %08x", device_map[thr_id], h_nounce[0]);
				// check if there was some other ones...
				if(h_nounce[1] != 0)
				{
					if(opt_verify)
					{
						le32enc(&endiandata[8], h_nounce[1]);
						siahash(vhash64, 80, endiandata);

					}
					if(swab64(vhash64[0]) <= Htarg && fulltest_sia(vhash64, (uint64_t*)ptarget))
					{
						pdata[20] = h_nounce[1];
						res++;
						if(opt_benchmark || opt_debug)  applog(LOG_INFO, "GPU #%d: Found second nonce", device_map[thr_id]);
					}
					else
					{
						if(vhash64[0] != Htarg) // don't show message if it is equal but fails fulltest
							applog(LOG_INFO, "GPU #%d: result does not validate on CPU!", device_map[thr_id]);
					}
				}
				pdata[8] = h_nounce[0];
//				applog(LOG_INFO, "hashes done = %08x", *hashes_done);
				return res;
			}
			else
			{
				if(vhash64[0] != Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_INFO, "GPU #%d: result does not validate on CPU!", device_map[thr_id]);
			}
		}
		pdata[8] += throughput;
		CUDA_SAFE_CALL(hipGetLastError());

	} while(!work_restart[thr_id].restart && ((uint64_t)max_nonce >((uint64_t)pdata[8] + (uint64_t)throughput)));
	*hashes_done = pdata[8] - first_nonce;
	return 0;
}