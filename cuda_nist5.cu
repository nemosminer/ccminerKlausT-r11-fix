#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"

extern void quark_blake512_cpu_init(int thr_id);
extern void quark_blake512_cpu_setBlock_80(int thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_setBlock_80_multi(int thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_80_multi(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_skein512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t *h_found, uint32_t target);


// Original nist5hash Funktion aus einem miner Quelltext
void nist5hash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    uint8_t hash[64];

    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    memcpy(state, hash, 32);
}

extern int scanhash_nist5(int thr_id, uint32_t *pdata,
    uint32_t *ptarget, uint32_t max_nonce,
    uint32_t *hashes_done)
{
	static THREAD uint32_t *d_hash = nullptr;
	static THREAD uint32_t *h_found = nullptr;
	static THREAD uint32_t oldthroughput;

	const uint32_t first_nonce = pdata[19];

	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, 1 << 19); // 256*256*16
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;

	if (opt_benchmark)
		ptarget[7] = 0x0Fu;

	static THREAD volatile bool init = false;
	if(!init)
	{
		oldthroughput = throughput;
		if(throughputmax == 1<<19)
			applog(LOG_INFO, "GPU #%d: using default intensity 19", device_map[thr_id]);
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughput > 0x7fffffffULL / (16 * sizeof(uint32_t)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		// Konstanten kopieren, Speicher belegen
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id);

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughput));
		CUDA_SAFE_CALL(hipHostMalloc(&(h_found), 2 * sizeof(uint32_t)));

//		cuda_check_cpu_init(thr_id, throughput);
		mining_has_stopped[thr_id] = false;
		init = true;
	}
	if(throughput > oldthroughput)
	{
		oldthroughput = throughput;
		CUDA_SAFE_CALL(hipFree(d_hash));
		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughput));
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, (uint64_t *)endiandata);
//	cuda_check_cpu_setTarget(ptarget, thr_id);

	do {

		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		quark_skein512_cpu_hash_64_final(thr_id, throughput, pdata[19], NULL, d_hash, h_found, ptarget[7]);

		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(h_found[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify){ be32enc(&endiandata[19], h_found[0]);
			nist5hash(vhash64, endiandata);

			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_found[1] != 0xffffffff)
				{
					if(opt_verify){ be32enc(&endiandata[19], h_found[1]);
					nist5hash(vhash64, endiandata);
					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						pdata[21] = h_found[1];
						res++;
						if (opt_benchmark)
							applog(LOG_INFO, "GPU #%d Found second nounce %08x", device_map[thr_id], h_found[1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[1]);
						}
					}

				}
				pdata[19] = h_found[0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", device_map[thr_id], h_found[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[0]);
				}
			}
		}
		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
