#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

static uint32_t *h_nonce[MAX_GPUS];
static uint32_t *d_nonce[MAX_GPUS];

#define shl(x, n)            ((x) << (n))
#define shr(x, n)            ((x) >> (n))
//#define SHR(x, n) SHR2(x, n) 
//#define SHL(x, n) SHL2(x, n) 

#undef SPH_ROTL32
#define SPH_ROTL32 ROTL32


#define ROTL32host(x, n) ROTL32(x,n)
// #define SPH_ROTL32 SPH_ROTL32
#define ss0(x)  (shr((x), 1) ^ shl((x), 3) ^ SPH_ROTL32((x),  4) ^ SPH_ROTL32((x), 19))
#define ss1(x)  (shr((x), 1) ^ shl((x), 2) ^ __byte_perm(x,0,0x2103) ^ SPH_ROTL32((x), 23))
#define ss2(x)  (shr((x), 2) ^ shl((x), 1) ^ SPH_ROTL32((x), 12) ^ SPH_ROTL32((x), 25))
#define ss3(x)  (shr((x), 2) ^ shl((x), 2) ^ SPH_ROTL32((x), 15) ^ SPH_ROTL32((x), 29))
#define ss4(x)  (shr((x), 1) ^ (x))
#define ss5(x)  (shr((x), 2) ^ (x))
#define rs1(x) SPH_ROTL32((x),  3)
#define rs2(x) SPH_ROTL32((x),  7)
#define rs3(x) SPH_ROTL32((x), 13)
#define rs4(x) __byte_perm(x,0,0x1032)
#define rs5(x) SPH_ROTL32((x), 19)
#define rs6(x) SPH_ROTL32((x), 23)
#define rs7(x) SPH_ROTL32((x), 27)


/* Message expansion function 1 */
static __forceinline__ __device__ uint32_t expand32_1(int i, const uint32_t *message, const uint32_t *H, const uint32_t *Q)
{
	return (ss1(Q[i - 16]) + ss2(Q[i - 15]) + ss3(Q[i - 14]) + ss0(Q[i - 13])
					+ ss1(Q[i - 12]) + ss2(Q[i - 11]) + ss3(Q[i - 10]) + ss0(Q[i - 9])
					+ ss1(Q[i - 8]) + ss2(Q[i - 7]) + ss3(Q[i - 6]) + ss0(Q[i - 5])
					+ ss1(Q[i - 4]) + ss2(Q[i - 3]) + ss3(Q[i - 2]) + ss0(Q[i - 1])
					+ ((i*(0x05555555ul) + SPH_ROTL32(message[(i - 16) % 16], ((i - 16) % 16) + 1) + SPH_ROTL32(message[(i - 13) % 16], ((i - 13) % 16) + 1) - SPH_ROTL32(message[(i - 6) % 16], ((i - 6) % 16) + 1)) ^ H[(i - 16 + 7) % 16]));
}

/* Message expansion function 2 */
static __forceinline__ __device__ uint32_t expand32_2(const int i, const uint32_t *message, const uint32_t *H, const uint32_t *Q)
{
	return (
		rs2(Q[i - 13]) + rs3(Q[i - 11]) + rs4(Q[i - 9]) + rs1(Q[i - 15]) +
		+rs5(Q[i - 7]) + rs6(Q[i - 5]) + rs7(Q[i - 3]) + ss4(Q[i - 2]) + ss5(Q[i - 1]));
}

#define TPB 512
__global__	__launch_bounds__(TPB, 2)
void bmw256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *g_hash, uint32_t *const __restrict__ nonceVector, uint32_t Target)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t backup = Target;
		uint32_t message[16] = {0};

		((uint2*)message)[0] = __ldg(&g_hash[thread]);
		((uint2*)message)[1] = __ldg(&g_hash[thread + 1 * threads]);
		((uint2*)message)[2] = __ldg(&g_hash[thread + 2 * threads]);
		((uint2*)message)[3] = __ldg(&g_hash[thread + 3 * threads]);


		const uint32_t h1[16] = {
			(0x40414243), (0x44454647),
			(0x48494A4B), (0x4C4D4E4F),
			(0x50515253), (0x54555657),
			(0x58595A5B), (0x5C5D5E5F),
			(0x60616263), (0x64656667),
			(0x68696A6B), (0x6C6D6E6F),
			(0x70717273), (0x74757677),
			(0x78797A7B), (0x7C7D7E7F)
		};

		message[8] = 0x80;
		message[14] = 0x100;

		uint32_t XL32, XH32, Q[32];

		Q[0] = (message[5] ^ h1[5]) - (message[7] ^ h1[7]) + (message[10] ^ h1[10]) + (message[13] ^ h1[13]) + (message[14] ^ h1[14]);
		Q[1] = (message[6] ^ h1[6]) - (message[8] ^ h1[8]) + (message[11] ^ h1[11]) + (message[14] ^ h1[14]) - (message[15] ^ h1[15]);
		Q[2] = (message[0] ^ h1[0]) + (message[7] ^ h1[7]) + (message[9] ^ h1[9]) - (message[12] ^ h1[12]) + (message[15] ^ h1[15]);
		Q[3] = (message[0] ^ h1[0]) - (message[1] ^ h1[1]) + (message[8] ^ h1[8]) - (message[10] ^ h1[10]) + (message[13] ^ h1[13]);
		Q[4] = (message[1] ^ h1[1]) + (message[2] ^ h1[2]) + (message[9] ^ h1[9]) - (message[11] ^ h1[11]) - (message[14] ^ h1[14]);
		Q[5] = (message[3] ^ h1[3]) - (message[2] ^ h1[2]) + (message[10] ^ h1[10]) - (message[12] ^ h1[12]) + (message[15] ^ h1[15]);
		Q[6] = (message[4] ^ h1[4]) - (message[0] ^ h1[0]) - (message[3] ^ h1[3]) - (message[11] ^ h1[11]) + (message[13] ^ h1[13]);
		Q[7] = (message[1] ^ h1[1]) - (message[4] ^ h1[4]) - (message[5] ^ h1[5]) - (message[12] ^ h1[12]) - (message[14] ^ h1[14]);
		Q[8] = (message[2] ^ h1[2]) - (message[5] ^ h1[5]) - (message[6] ^ h1[6]) + (message[13] ^ h1[13]) - (message[15] ^ h1[15]);
		Q[9] = (message[0] ^ h1[0]) - (message[3] ^ h1[3]) + (message[6] ^ h1[6]) - (message[7] ^ h1[7]) + (message[14] ^ h1[14]);
		Q[10] = (message[8] ^ h1[8]) - (message[1] ^ h1[1]) - (message[4] ^ h1[4]) - (message[7] ^ h1[7]) + (message[15] ^ h1[15]);
		Q[11] = (message[8] ^ h1[8]) - (message[0] ^ h1[0]) - (message[2] ^ h1[2]) - (message[5] ^ h1[5]) + (message[9] ^ h1[9]);
		Q[12] = (message[1] ^ h1[1]) + (message[3] ^ h1[3]) - (message[6] ^ h1[6]) - (message[9] ^ h1[9]) + (message[10] ^ h1[10]);
		Q[13] = (message[2] ^ h1[2]) + (message[4] ^ h1[4]) + (message[7] ^ h1[7]) + (message[10] ^ h1[10]) + (message[11] ^ h1[11]);
		Q[14] = (message[3] ^ h1[3]) - (message[5] ^ h1[5]) + (message[8] ^ h1[8]) - (message[11] ^ h1[11]) - (message[12] ^ h1[12]);
		Q[15] = (message[12] ^ h1[12]) - (message[4] ^ h1[4]) - (message[6] ^ h1[6]) - (message[9] ^ h1[9]) + (message[13] ^ h1[13]);

		Q[0] = ss0(Q[0]) + h1[1];
		Q[1] = ss1(Q[1]) + h1[2];
		Q[2] = ss2(Q[2]) + h1[3];
		Q[3] = ss3(Q[3]) + h1[4];
		Q[4] = ss4(Q[4]) + h1[5];
		Q[5] = ss0(Q[5]) + h1[6];
		Q[6] = ss1(Q[6]) + h1[7];
		Q[7] = ss2(Q[7]) + h1[8];
		Q[8] = ss3(Q[8]) + h1[9];
		Q[9] = ss4(Q[9]) + h1[10];
		Q[10] = ss0(Q[10]) + h1[11];
		Q[11] = ss1(Q[11]) + h1[12];
		Q[12] = ss2(Q[12]) + h1[13];
		Q[13] = ss3(Q[13]) + h1[14];
		Q[14] = ss4(Q[14]) + h1[15];
		Q[15] = ss0(Q[15]) + h1[0];

		Q[16] = ss1(Q[16 - 16]) + ss2(Q[16 - 15]) + ss3(Q[16 - 14]) + ss0(Q[16 - 13])
			+ ss1(Q[16 - 12]) + ss2(Q[16 - 11]) + ss3(Q[16 - 10]) + ss0(Q[16 - 9])
			+ ss1(Q[16 - 8]) + ss2(Q[16 - 7]) + ss3(Q[16 - 6]) + ss0(Q[16 - 5])
			+ ss1(Q[16 - 4]) + ss2(Q[16 - 3]) + ss3(Q[16 - 2]) + ss0(Q[16 - 1])
			+ ((16 * (0x05555555ul) + SPH_ROTL32(message[0], ((16 - 16) % 16) + 1) + SPH_ROTL32(message[3], ((16 - 13) % 16) + 1)) ^ h1[(16 - 16 + 7) % 16]);

		Q[17] = ss1(Q[17 - 16]) + ss2(Q[17 - 15]) + ss3(Q[17 - 14]) + ss0(Q[17 - 13])
			+ ss1(Q[17 - 12]) + ss2(Q[17 - 11]) + ss3(Q[17 - 10]) + ss0(Q[17 - 9])
			+ ss1(Q[17 - 8]) + ss2(Q[17 - 7]) + ss3(Q[17 - 6]) + ss0(Q[17 - 5])
			+ ss1(Q[17 - 4]) + ss2(Q[17 - 3]) + ss3(Q[17 - 2]) + ss0(Q[17 - 1])
			+ ((17 * (0x05555555ul) + SPH_ROTL32(message[(17 - 16) % 16], ((17 - 16) % 16) + 1) + SPH_ROTL32(message[(17 - 13) % 16], ((17 - 13) % 16) + 1)) ^ h1[(17 - 16 + 7) % 16]);

		uint32_t precalc = Q[18 - 16] + Q[18 - 14] + Q[18 - 12] + Q[18 - 10] + Q[18 - 8] + Q[18 - 6]; //+ Q[18 - 4]
		uint32_t precalc2 = Q[19 - 16] + Q[19 - 14] + Q[19 - 12] + Q[19 - 10] + Q[19 - 8] + Q[19 - 6];//+ Q[19 - 4]

		precalc = precalc + Q[18 - 4];
		precalc2 = precalc2 + Q[18 + 1 - 4];
		uint32_t p1 = ((18 * (0x05555555ul) + SPH_ROTL32(message[2], ((18 - 16) % 16) + 1) + SPH_ROTL32(message[5], ((18 - 13) % 16) + 1)) ^ h1[(18 - 16 + 7) % 16]);
		uint32_t p2 = (((18 + 1)*(0x05555555ul) + SPH_ROTL32(message[3], (((18 + 1) - 16) % 16) + 1) + SPH_ROTL32(message[6], (((18 + 1) - 13) % 16) + 1)) ^ h1[((18 + 1) - 16 + 7) % 16]);
		Q[18] = precalc + expand32_2(18, message, h1, Q) + p1;
		Q[18 + 1] = precalc2 + expand32_2(18 + 1, message, h1, Q) + p2;
		precalc = precalc - Q[18 - 16];
		precalc2 = precalc2 - Q[18 + 1 - 16];

		precalc = precalc + Q[20 - 4];
		precalc2 = precalc2 + Q[20 + 1 - 4];
		p1 = ((20 * (0x05555555ul) + SPH_ROTL32(message[4], ((20 - 16) % 16) + 1) + SPH_ROTL32(message[7], ((20 - 13) % 16) + 1) - (0x100 << 15)) ^ h1[(20 - 16 + 7) % 16]);
		p2 = (((20 + 1)*(0x05555555ul) + SPH_ROTL32(message[5], (((20 + 1) - 16) % 16) + 1) + (0x80 << 9)) ^ h1[((20 + 1) - 16 + 7) % 16]);
		Q[20] = precalc + expand32_2(20, message, h1, Q) + p1;
		Q[20 + 1] = precalc2 + expand32_2(20 + 1, message, h1, Q) + p2;
		precalc = precalc - Q[20 - 16];
		precalc2 = precalc2 - Q[20 + 1 - 16];

		precalc = precalc + Q[22 - 4];
		precalc2 = precalc2 + Q[22 + 1 - 4];
		p1 = ((22 * (0x05555555ul) + SPH_ROTL32(message[6], ((22 - 16) % 16) + 1) - SPH_ROTL32(message[0], ((22 - 6) % 16) + 1)) ^ h1[(22 - 16 + 7) % 16]);
		p2 = (((22 + 1)*(0x05555555ul) + SPH_ROTL32(message[7], (((22 + 1) - 16) % 16) + 1) - SPH_ROTL32(message[1], (((22 + 1) - 6) % 16) + 1)) ^ h1[((22 + 1) - 16 + 7) % 16]);
		Q[22] = precalc + expand32_2(22, message, h1, Q) + p1;
		Q[22 + 1] = precalc2 + expand32_2(22 + 1, message, h1, Q) + p2;
		precalc = precalc - Q[22 - 16];
		precalc2 = precalc2 - Q[22 + 1 - 16];

		precalc = precalc + Q[24 - 4];
		precalc2 = precalc2 + Q[24 + 1 - 4];
		p1 = ((24 * (0x05555555ul) + (0x80 << 9) - SPH_ROTL32(message[2], ((24 - 6) % 16) + 1)) ^ h1[(24 - 16 + 7) % 16]);
		p2 = (((24 + 1)*(0x05555555ul) - SPH_ROTL32(message[3], (((24 + 1) - 6) % 16) + 1)) ^ h1[((24 + 1) - 16 + 7) % 16]);
		Q[24] = precalc + expand32_2(24, message, h1, Q) + p1;
		Q[24 + 1] = precalc2 + expand32_2(24 + 1, message, h1, Q) + p2;
		precalc = precalc - Q[24 - 16];
		precalc2 = precalc2 - Q[24 + 1 - 16];

		precalc = precalc + Q[26 - 4];
		precalc2 = precalc2 + Q[26 + 1 - 4];
		p1 = ((26 * (0x05555555ul) - SPH_ROTL32(message[4], ((26 - 6) % 16) + 1)) ^ h1[(26 - 16 + 7) % 16]);
		p2 = (((26 + 1)*(0x05555555ul) + (0x100 << 15) - SPH_ROTL32(message[5], (((26 + 1) - 6) % 16) + 1)) ^ h1[((26 + 1) - 16 + 7) % 16]);
		Q[26] = precalc + expand32_2(26, message, h1, Q) + p1;
		Q[26 + 1] = precalc2 + expand32_2(26 + 1, message, h1, Q) + p2;
		precalc = precalc - Q[26 - 16];
		precalc2 = precalc2 - Q[26 + 1 - 16];

		precalc = precalc + Q[28 - 4];
		precalc2 = precalc2 + Q[28 + 1 - 4];
		p1 = ((28 * (0x05555555ul) - SPH_ROTL32(message[6], ((28 - 6) % 16) + 1)) ^ h1[(28 - 16 + 7) % 16]);
		p2 = (((28 + 1)*(0x05555555ul) + SPH_ROTL32(message[0], (((28 + 1) - 13) % 16) + 1) - SPH_ROTL32(message[7], (((28 + 1) - 6) % 16) + 1)) ^ h1[((28 + 1) - 16 + 7) % 16]);
		Q[28] = precalc + expand32_2(28, message, h1, Q) + p1;
		Q[28 + 1] = precalc2 + expand32_2(28 + 1, message, h1, Q) + p2;
		precalc = precalc - Q[28 - 16];
		precalc2 = precalc2 - Q[28 + 1 - 16];

		precalc = precalc + Q[30 - 4];
		precalc2 = precalc2 + Q[30 + 1 - 4];
		p1 = ((30 * (0x05555555ul) + (0x100 << 15) + SPH_ROTL32(message[1], ((30 - 13) % 16) + 1) - (0x80 << 9)) ^ h1[(30 - 16 + 7) % 16]);
		p2 = (((30 + 1)*(0x05555555ul) + SPH_ROTL32(message[2], (((30 + 1) - 13) % 16) + 1)) ^ h1[((30 + 1) - 16 + 7) % 16]);
		Q[30] = precalc + expand32_2(30, message, h1, Q) + p1;
		Q[30 + 1] = precalc2 + expand32_2(30 + 1, message, h1, Q) + p2;
		precalc = precalc - Q[30 - 16];
		precalc2 = precalc2 - Q[30 + 1 - 16];

		XL32 = Q[16] ^ Q[17] ^ Q[18] ^ Q[19] ^ Q[20] ^ Q[21] ^ Q[22] ^ Q[23];
		XH32 = XL32^Q[24] ^ Q[25] ^ Q[26] ^ Q[27] ^ Q[28] ^ Q[29] ^ Q[30] ^ Q[31];

		message[0] = (shl(XH32, 5) ^ shr(Q[16], 5) ^ message[0]) + (XL32    ^ Q[24] ^ Q[0]);
		message[1] = (shr(XH32, 7) ^ shl(Q[17], 8) ^ message[1]) + (XL32    ^ Q[25] ^ Q[1]);
		message[2] = (shr(XH32, 5) ^ shl(Q[18], 5) ^ message[2]) + (XL32    ^ Q[26] ^ Q[2]);
		message[3] = (shr(XH32, 1) ^ shl(Q[19], 5) ^ message[3]) + (XL32    ^ Q[27] ^ Q[3]);
		message[4] = (shr(XH32, 3) ^ Q[20] ^ message[4]) + (XL32    ^ Q[28] ^ Q[4]);
		message[5] = (shl(XH32, 6) ^ shr(Q[21], 6) ^ message[5]) + (XL32    ^ Q[29] ^ Q[5]);
		message[6] = (shr(XH32, 4) ^ shl(Q[22], 6) ^ message[6]) + (XL32    ^ Q[30] ^ Q[6]);
		message[7] = (shr(XH32, 11) ^ shl(Q[23], 2) ^ message[7]) + (XL32    ^ Q[31] ^ Q[7]);

		message[8] = SPH_ROTL32(message[4], 9) + (XH32     ^     Q[24] ^ message[8]) + (shl(XL32, 8) ^ Q[23] ^ Q[8]);
		message[9] = SPH_ROTL32(message[5], 10) + (XH32     ^     Q[25] ^ message[9]) + (shr(XL32, 6) ^ Q[16] ^ Q[9]);
		message[10] = SPH_ROTL32(message[6], 11) + (XH32     ^     Q[26] ^ message[10]) + (shl(XL32, 6) ^ Q[17] ^ Q[10]);
		message[11] = SPH_ROTL32(message[7], 12) + (XH32     ^     Q[27] ^ message[11]) + (shl(XL32, 4) ^ Q[18] ^ Q[11]);
		message[12] = SPH_ROTL32(message[0], 13) + (XH32     ^     Q[28] ^ message[12]) + (shr(XL32, 3) ^ Q[19] ^ Q[12]);
		message[13] = SPH_ROTL32(message[1], 14) + (XH32     ^     Q[29] ^ message[13]) + (shr(XL32, 4) ^ Q[20] ^ Q[13]);
		message[14] = SPH_ROTL32(message[2], 15) + (XH32     ^     Q[30] ^ message[14]) + (shr(XL32, 7) ^ Q[21] ^ Q[14]);
		message[15] = SPH_ROTL32(message[3], 16) + (XH32     ^     Q[31] ^ message[15]) + (shr(XL32, 2) ^ Q[22] ^ Q[15]);

		const uint32_t h2[16] = {
			(0xaaaaaaa0), (0xaaaaaaa1), (0xaaaaaaa2),
			(0xaaaaaaa3), (0xaaaaaaa4), (0xaaaaaaa5),
			(0xaaaaaaa6), (0xaaaaaaa7), (0xaaaaaaa8),
			(0xaaaaaaa9), (0xaaaaaaaa), (0xaaaaaaab),
			(0xaaaaaaac), (0xaaaaaaad), (0xaaaaaaae),
			(0xaaaaaaaf)
		};

		Q[0] = (message[5] ^ h2[5]) - (message[7] ^ h2[7]) + (message[10] ^ h2[10]) + (message[13] ^ h2[13]) + (message[14] ^ h2[14]);
		Q[1] = (message[6] ^ h2[6]) - (message[8] ^ h2[8]) + (message[11] ^ h2[11]) + (message[14] ^ h2[14]) - (message[15] ^ h2[15]);
		Q[2] = (message[0] ^ h2[0]) + (message[7] ^ h2[7]) + (message[9] ^ h2[9]) - (message[12] ^ h2[12]) + (message[15] ^ h2[15]);
		Q[3] = (message[0] ^ h2[0]) - (message[1] ^ h2[1]) + (message[8] ^ h2[8]) - (message[10] ^ h2[10]) + (message[13] ^ h2[13]);
		Q[4] = (message[1] ^ h2[1]) + (message[2] ^ h2[2]) + (message[9] ^ h2[9]) - (message[11] ^ h2[11]) - (message[14] ^ h2[14]);
		Q[5] = (message[3] ^ h2[3]) - (message[2] ^ h2[2]) + (message[10] ^ h2[10]) - (message[12] ^ h2[12]) + (message[15] ^ h2[15]);
		Q[6] = (message[4] ^ h2[4]) - (message[0] ^ h2[0]) - (message[3] ^ h2[3]) - (message[11] ^ h2[11]) + (message[13] ^ h2[13]);
		Q[7] = (message[1] ^ h2[1]) - (message[4] ^ h2[4]) - (message[5] ^ h2[5]) - (message[12] ^ h2[12]) - (message[14] ^ h2[14]);
		Q[8] = (message[2] ^ h2[2]) - (message[5] ^ h2[5]) - (message[6] ^ h2[6]) + (message[13] ^ h2[13]) - (message[15] ^ h2[15]);
		Q[9] = (message[0] ^ h2[0]) - (message[3] ^ h2[3]) + (message[6] ^ h2[6]) - (message[7] ^ h2[7]) + (message[14] ^ h2[14]);
		Q[10] = (message[8] ^ h2[8]) - (message[1] ^ h2[1]) - (message[4] ^ h2[4]) - (message[7] ^ h2[7]) + (message[15] ^ h2[15]);
		Q[11] = (message[8] ^ h2[8]) - (message[0] ^ h2[0]) - (message[2] ^ h2[2]) - (message[5] ^ h2[5]) + (message[9] ^ h2[9]);
		Q[12] = (message[1] ^ h2[1]) + (message[3] ^ h2[3]) - (message[6] ^ h2[6]) - (message[9] ^ h2[9]) + (message[10] ^ h2[10]);
		Q[13] = (message[2] ^ h2[2]) + (message[4] ^ h2[4]) + (message[7] ^ h2[7]) + (message[10] ^ h2[10]) + (message[11] ^ h2[11]);
		Q[14] = (message[3] ^ h2[3]) - (message[5] ^ h2[5]) + (message[8] ^ h2[8]) - (message[11] ^ h2[11]) - (message[12] ^ h2[12]);
		Q[15] = (message[12] ^ h2[12]) - (message[4] ^ h2[4]) - (message[6] ^ h2[6]) - (message[9] ^ h2[9]) + (message[13] ^ h2[13]);

		Q[0] = ss0(Q[0]) + h2[1];
		Q[1] = ss1(Q[1]) + h2[2];
		Q[2] = ss2(Q[2]) + h2[3];
		Q[3] = ss3(Q[3]) + h2[4];
		Q[4] = ss4(Q[4]) + h2[5];
		Q[5] = ss0(Q[5]) + h2[6];
		Q[6] = ss1(Q[6]) + h2[7];
		Q[7] = ss2(Q[7]) + h2[8];
		Q[8] = ss3(Q[8]) + h2[9];
		Q[9] = ss4(Q[9]) + h2[10];
		Q[10] = ss0(Q[10]) + h2[11];
		Q[11] = ss1(Q[11]) + h2[12];
		Q[12] = ss2(Q[12]) + h2[13];
		Q[13] = ss3(Q[13]) + h2[14];
		Q[14] = ss4(Q[14]) + h2[15];
		Q[15] = ss0(Q[15]) + h2[0];

#pragma unroll
		for(int i = 0; i<2; i++)
			Q[i + 16] = expand32_1(i + 16, message, h2, Q);

		precalc = Q[18 - 16] + Q[18 - 14] + Q[18 - 12] + Q[18 - 10] + Q[18 - 8] + Q[18 - 6];
		precalc2 = Q[19 - 16] + Q[19 - 14] + Q[19 - 12] + Q[19 - 10] + Q[19 - 8] + Q[19 - 6];

#pragma unroll
		for(int i = 2 + 16; i < 16 + 16; i += 2)
		{
			precalc = precalc + Q[i - 4];
			precalc2 = precalc2 + Q[i + 1 - 4];
			p1 = ((i*(0x05555555ul) + SPH_ROTL32(message[(i - 16) % 16], ((i - 16) % 16) + 1) + SPH_ROTL32(message[(i - 13) % 16], ((i - 13) % 16) + 1) - SPH_ROTL32(message[(i - 6) % 16], ((i - 6) % 16) + 1)) ^ h2[(i - 16 + 7) % 16]);
			p2 = (((i + 1)*(0x05555555ul) + SPH_ROTL32(message[((i + 1) - 16) % 16], (((i + 1) - 16) % 16) + 1) + SPH_ROTL32(message[((i + 1) - 13) % 16], (((i + 1) - 13) % 16) + 1) - SPH_ROTL32(message[((i + 1) - 6) % 16], (((i + 1) - 6) % 16) + 1)) ^ h2[((i + 1) - 16 + 7) % 16]);
			Q[i] = precalc + expand32_2(i, message, h2, Q) + p1;
			Q[i + 1] = precalc2 + expand32_2(i + 1, message, h2, Q) + p2;
			precalc = precalc - Q[i - 16];
			precalc2 = precalc2 - Q[i + 1 - 16];
		}

		XL32 = Q[16] ^ Q[17] ^ Q[18] ^ Q[19] ^ Q[20] ^ Q[21] ^ Q[22] ^ Q[23];
		XH32 = XL32^Q[24] ^ Q[25] ^ Q[26] ^ Q[27] ^ Q[28] ^ Q[29] ^ Q[30] ^ Q[31];

		message[3] = (shr(XH32, 1) ^ shl(Q[19], 5) ^ message[3]) + (XL32    ^ Q[27] ^ Q[3]);
		message[15] = SPH_ROTL32(message[3], 16) + (XH32     ^     Q[31] ^ message[15]) + (shr(XL32, 2) ^ Q[22] ^ Q[15]);

		if(message[15] <= backup)
		{

			uint32_t tmp = atomicExch(nonceVector, startNounce + thread);
			if(tmp != 0)
				nonceVector[1] = tmp;
		}
	}
}


__host__
void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces, uint32_t Target)
{
	CUDA_SAFE_CALL(hipMemsetAsync(d_nonce[thr_id], 0x0, 2 * sizeof(uint32_t), gpustream[thr_id]));

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + TPB - 1) / TPB);
	dim3 block(TPB);

	bmw256_gpu_hash_32 << <grid, block >> >(threads, startNounce, (uint2 *)g_hash, d_nonce[thr_id], Target);
	CUDA_SAFE_CALL(hipGetLastError());
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy(h_nonce[thr_id], d_nonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
	resultnonces[0] = *(h_nonce[thr_id]);
	resultnonces[1] = *(h_nonce[thr_id] + 1);
}


__host__
void bmw256_cpu_init(int thr_id)
{
	CUDA_SAFE_CALL(hipMalloc(&d_nonce[thr_id], 2 * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipHostMalloc(&h_nonce[thr_id], 2 * sizeof(uint32_t)));
}

/*
__host__
void bmw256_setTarget(int thr_id, const void *pTargetIn)
{
hipMemcpyToSymbolAsync(HIP_SYMBOL(pTarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]);
}
*/