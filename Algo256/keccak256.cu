#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

extern "C"
{
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_keccak.h"
}
#include "miner.h"


#include "cuda_helper.h"

extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void keccak256_setBlock_80(int thr_id, void *pdata,const void *ptarget);
extern void keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *h_nounce);

// CPU Hash
void keccak256_hash(void *state, const void *input)
{
	sph_keccak_context ctx_keccak;

	uint32_t hash[16];

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256 (&ctx_keccak, input, 80);
	sph_keccak256_close(&ctx_keccak, (void*) hash);

	memcpy(state, hash, 32);
}

extern int scanhash_keccak256(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	static THREAD uint32_t *h_nounce = nullptr;

	const uint32_t first_nonce = pdata[19];
	uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 28 : 1 << 27;;
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	uint32_t throughput = min(throughputmax, max_nonce - first_nonce) & 0xfffffc00;


	if (opt_benchmark)
		ptarget[7] = 0x0002;

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		CUDA_SAFE_CALL(hipHostMalloc(&h_nounce, 2 * sizeof(uint32_t)));
		keccak256_cpu_init(thr_id, (int)throughputmax);
//		CUDA_SAFE_CALL(hipHostMalloc(&h_nounce, 2 * sizeof(uint32_t)));
		mining_has_stopped[thr_id] = false;
		init = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	keccak256_setBlock_80(thr_id, (void*)endiandata, ptarget);

	do {

		keccak256_cpu_hash_80(thr_id, (int) throughput, pdata[19], h_nounce);
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(h_nounce[0] != UINT32_MAX)
		{
			uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify){ be32enc(&endiandata[19], h_nounce[0]);
			keccak256_hash(vhash64, endiandata);

			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_nounce[1] != 0xffffffff)
				{
					if(opt_verify){ be32enc(&endiandata[19], h_nounce[1]);
					keccak256_hash(vhash64, endiandata);

					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = h_nounce[1];
						res++;
						if (opt_benchmark)
							applog(LOG_INFO, "GPU #%d Found second nounce %08x", device_map[thr_id], h_nounce[1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_nounce[1]);
						}
					}
				}
				pdata[19] = h_nounce[0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", device_map[thr_id], h_nounce[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_nounce[0]);
				}
			}
		}

		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
