#include "hip/hip_runtime.h"
#include "miner.h"
#ifdef __cplusplus
#include <cstdint>
#else
#include <stdint.h>
#endif
#include <memory.h>
#include "cuda_helper.h"


#define UINT2(x,y) make_uint2(x,y)

static uint32_t *d_KNonce[MAX_GPUS];

__constant__ uint32_t pTarget[8];
__constant__ uint64_t keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint2 keccak_round_constants35[24] = {
		{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
		{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
		{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
		{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
		{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
		{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
		{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
		{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
		{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
		{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
		{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
};


__constant__ uint2 c_PaddedMessage80[10]; // padded message (80 bytes + padding?)

#define bitselect(a, b, c) ((a) ^ ((c) & ((b) ^ (a))))

static void __forceinline__ __device__ keccak_block(uint2 *s)
{
	uint2 bc[5], tmpxor[5], tmp1, tmp2;
//	uint2 s[25];

#pragma unroll 1
	for (int i= 0; i < 24; i++) 
	{
#pragma unroll
		for (uint32_t x = 0; x < 5; x++)
			tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		tmp1 = s[1] ^ bc[0];

		s[0] ^= bc[4];
		s[1] = ROL2(s[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(s[22] ^ bc[1], 61);
		s[22] = ROL2(s[14] ^ bc[3], 39);
		s[14] = ROL2(s[20] ^ bc[4], 18);
		s[20] = ROL2(s[2] ^ bc[1], 62);
		s[2] = ROL2(s[12] ^ bc[1], 43);
		s[12] = ROL2(s[13] ^ bc[2], 25);
		s[13] = ROL8(s[19] ^ bc[3]);
		s[19] = ROR8(s[23] ^ bc[2]);
		s[23] = ROL2(s[15] ^ bc[4], 41);
		s[15] = ROL2(s[4] ^ bc[3], 27);
		s[4] = ROL2(s[24] ^ bc[3], 14);
		s[24] = ROL2(s[21] ^ bc[0], 2);
		s[21] = ROL2(s[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(s[5] ^ bc[4], 36);
		s[5] = ROL2(s[3] ^ bc[2], 28);
		s[3] = ROL2(s[18] ^ bc[2], 21);
		s[18] = ROL2(s[17] ^ bc[1], 15);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[11] = ROL2(s[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(tmp1, 1);

		tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
		tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
		tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
		tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
		tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
		s[0] ^= keccak_round_constants35[i];
	}
}

__global__	__launch_bounds__(512)
void keccak256_gpu_hash_80(uint32_t threads, uint32_t startNounce,  uint32_t *const __restrict__ resNounce)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		const uint32_t nounce = startNounce + thread;
		uint2 bc[5], tmpxor[5], tmp1, tmp2;
		uint2 s[25];
		
		s[9] = make_uint2(c_PaddedMessage80[9].x, cuda_swab32(nounce));
		s[10] = make_uint2(1, 0);
		s[16] = make_uint2(0, 0x80000000);

		tmpxor[0] = c_PaddedMessage80[0] ^ c_PaddedMessage80[5] ^ s[10];
		tmpxor[1] = c_PaddedMessage80[1] ^ c_PaddedMessage80[6] ^ s[16];
		tmpxor[2] = c_PaddedMessage80[2] ^ c_PaddedMessage80[7];
		tmpxor[3] = c_PaddedMessage80[3] ^ c_PaddedMessage80[8];
		tmpxor[4] = c_PaddedMessage80[4] ^ s[9];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		tmp1 = c_PaddedMessage80[1] ^ bc[0];

		s[0] = c_PaddedMessage80[0] ^ bc[4];
		s[1] = ROL2(c_PaddedMessage80[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(bc[1], 61);
		s[22] = ROL2(bc[3], 39);
		s[14] = ROL2(bc[4], 18);
		s[20] = ROL2(c_PaddedMessage80[2] ^ bc[1], 62);
		s[2] = ROL2(bc[1], 43);
		s[12] = ROL2(bc[2], 25);
		s[13] = ROL8(bc[3]);
		s[19] = ROR8(bc[2]);
		s[23] = ROL2(bc[4], 41);
		s[15] = ROL2(c_PaddedMessage80[4] ^ bc[3], 27);
		s[4] = ROL2(bc[3], 14);
		s[24] = ROL2(bc[0], 2);
		s[21] = ROL2(c_PaddedMessage80[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(c_PaddedMessage80[5] ^ bc[4], 36);
		s[5] = ROL2(c_PaddedMessage80[3] ^ bc[2], 28);
		s[3] = ROL2( bc[2], 21);
		s[18] = ROL2(bc[1], 15);
		s[17] = ROL2(bc[0], 10);
		s[11] = ROL2(c_PaddedMessage80[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(tmp1, 1);

		tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
		tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
		tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
		tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
		tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
		s[0].x ^= 1;

#pragma unroll 2
		for (int i = 1; i < 23; i++) 
		{

#pragma unroll
			for (uint32_t x = 0; x < 5; x++)
				tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

			bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
			bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
			bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
			bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
			bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

			tmp1 = s[1] ^ bc[0];

			s[0] ^= bc[4];
			s[1] = ROL2(s[6] ^ bc[0], 44);
			s[6] = ROL2(s[9] ^ bc[3], 20);
			s[9] = ROL2(s[22] ^ bc[1], 61);
			s[22] = ROL2(s[14] ^ bc[3], 39);
			s[14] = ROL2(s[20] ^ bc[4], 18);
			s[20] = ROL2(s[2] ^ bc[1], 62);
			s[2] = ROL2(s[12] ^ bc[1], 43);
			s[12] = ROL2(s[13] ^ bc[2], 25);
			s[13] = ROL8(s[19] ^ bc[3]);
			s[19] = ROR8(s[23] ^ bc[2]);
			s[23] = ROL2(s[15] ^ bc[4], 41);
			s[15] = ROL2(s[4] ^ bc[3], 27);
			s[4] = ROL2(s[24] ^ bc[3], 14);
			s[24] = ROL2(s[21] ^ bc[0], 2);
			s[21] = ROL2(s[8] ^ bc[2], 55);
			s[8] = ROL2(s[16] ^ bc[0], 45);
			s[16] = ROL2(s[5] ^ bc[4], 36);
			s[5] = ROL2(s[3] ^ bc[2], 28);
			s[3] = ROL2(s[18] ^ bc[2], 21);
			s[18] = ROL2(s[17] ^ bc[1], 15);
			s[17] = ROL2(s[11] ^ bc[0], 10);
			s[11] = ROL2(s[7] ^ bc[1], 6);
			s[7] = ROL2(s[10] ^ bc[4], 3);
			s[10] = ROL2(tmp1, 1);

			tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
			tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
			tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
			tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
			tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
			s[0] ^= keccak_round_constants35[i];
		}
		uint2 t[5];
		t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
		t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
		t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
		t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
		t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

		s[0] ^= t[4] ^ ROL2(t[1], 1);
		s[18] ^= t[2] ^ ROL2(t[4], 1);
		s[24] ^= t[3] ^ ROL2(t[0], 1);

		s[3] = ROL2(s[18], 21) ^ ((~ROL2(s[24], 14)) & s[0]);


		if (devectorize(s[3]) <= ((uint64_t*)pTarget)[3])
		{
			uint32_t tmp = atomicCAS(resNounce, 0xffffffff, nounce);
			if (tmp != 0xffffffff)
				resNounce[1] = nounce;
		}
	}
}

__host__
void keccak256_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *h_nounce)
{
	CUDA_SAFE_CALL(hipMemsetAsync(d_KNonce[thr_id], 0xff, 2 * sizeof(uint32_t), gpustream[thr_id]));
	const uint32_t threadsperblock = 512;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	keccak256_gpu_hash_80<<<grid, block, 0, gpustream[thr_id]>>>(threads, startNounce, d_KNonce[thr_id]);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy(h_nounce, d_KNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
}

__global__ __launch_bounds__(256,3)
void keccak256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		uint2 keccak_gpu_state[25];
		#pragma unroll 25
		for (int i = 0; i<25; i++) {
			if (i<4) keccak_gpu_state[i] = vectorize(outputHash[i*threads+thread]);
			else     keccak_gpu_state[i] = UINT2(0, 0);
		}
		keccak_gpu_state[4]  = UINT2(1, 0);
		keccak_gpu_state[16] = UINT2(0, 0x80000000);
		keccak_block(keccak_gpu_state);

		#pragma unroll 4
		for (int i=0; i<4; i++)
			outputHash[i*threads+thread] = devectorize(keccak_gpu_state[i]);
	}
}

__host__
void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	keccak256_gpu_hash_32 <<<grid, block, 0, gpustream[thr_id]>>> (threads, startNounce, d_outputHash);
	CUDA_SAFE_CALL(hipGetLastError());
}

__host__
void keccak256_setBlock_80(int thr_id, void *pdata,const void *pTargetIn)
{
	unsigned char PaddedMessage[80];
	memcpy(PaddedMessage, pdata, 80);
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(pTarget), pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]));
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 10 * sizeof(uint64_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]));
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
}

__host__
void keccak256_cpu_init(int thr_id, uint32_t threads)
{
	CUDA_SAFE_CALL(hipMalloc(&d_KNonce[thr_id], 2*sizeof(uint32_t)));
}