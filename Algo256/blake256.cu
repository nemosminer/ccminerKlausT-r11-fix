#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Nov. 2014
 */

#define PRECALC64 1

#include "miner.h"

extern "C" {
#include "sph/sph_blake.h"
}

#include <cstdint>
#include <memory.h>


/* threads per block and throughput (intensity) */
#define TPB 128

/* added in sph_blake.c */
extern "C" int blake256_rounds = 14;

/* hash by cpu with blake 256 */
void blake256hash(void *output, const void *input, int8_t rounds = 14)
{
	uchar hash[64];
	sph_blake256_context ctx;

	blake256_rounds = rounds;

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 80);
	sph_blake256_close(&ctx, hash);

	memcpy(output, hash, 32);
}

#include "cuda_helper.h"

#if PRECALC64
__constant__ uint32_t _ALIGN(32) d_data[15];
static THREAD uint32_t *h_data;
#else
__constant__ static uint32_t _ALIGN(32) c_data[20];
/* midstate hash cache, this algo is run on 2 parts */
__device__ static uint32_t cache[8];
__device__ static uint32_t prevsum = 0;
/* crc32.c */
extern "C" uint32_t crc32_u32t(const uint32_t *buf, size_t size);
#endif

/* 8 adapters max */
static uint32_t *d_resNonce[MAX_GPUS];
static THREAD uint32_t *h_resNonce;

/* max count of found nonces in one call */
#define NBN 2
static uint32_t extra_results[MAX_GPUS][NBN] = { UINT32_MAX };

#if !PRECALC64
__device__ __constant__
static const uint32_t __align__(32) c_IV256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85),
	SPH_C32(0x3C6EF372), SPH_C32(0xA54FF53A),
	SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};
#endif

#define GSPREC(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ c_u256[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ c_u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
	}

/* Second part (64-80) msg never change, store it */
__device__ static
void blake256_compress(uint32_t *h, const uint32_t *block, const uint32_t T0, const int rounds)
{
	uint32_t /*_ALIGN(8)*/ m[16];
	uint32_t v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

	const uint32_t c_u256[16] = 
	{
		SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
		SPH_C32(0x13198A2E), SPH_C32(0x03707344),
		SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
		SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
		SPH_C32(0x452821E6), SPH_C32(0x38D01377),
		SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
		SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
		SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
	};

	 const uint32_t c_Padding[16] = {
		0, 0, 0, 0,
		0x80000000UL, 0, 0, 0,
		0, 0, 0, 0,
		0, 1, 0, 640,
	};


	#pragma unroll
	for (uint32_t i = 4; i < 16; i++) 
	{
#if PRECALC64
		m[i] = c_Padding[i];
#else
		m[i] = (T0 == 0x200) ? block[i] : c_Padding[i];
#endif
	}

#pragma unroll
	for(uint32_t i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 8] = c_u256[0];
	v[ 9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	GSPREC(0, 4, 0x8, 0xC,0,1);
	GSPREC(1, 5, 0x9, 0xD,2,3);
	GSPREC(2, 6, 0xA, 0xE, 4,5);
	GSPREC(3, 7, 0xB, 0xF, 6,7);
	GSPREC(0, 5, 0xA, 0xF, 8,9);
	GSPREC(1, 6, 0xB, 0xC, 10,11);
	GSPREC(2, 7, 0x8, 0xD, 12,13);
	GSPREC(3, 4, 0x9, 0xE, 14,15);
	//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	GSPREC(0, 4, 0x8, 0xC, 14, 10);
	GSPREC(1, 5, 0x9, 0xD, 4, 8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0, 2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5, 3);
	//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5, 2);
	GSPREC(3, 7, 0xB, 0xF, 15, 13);
	GSPREC(0, 5, 0xA, 0xF, 10, 14);
	GSPREC(1, 6, 0xB, 0xC, 3, 6);
	GSPREC(2, 7, 0x8, 0xD, 7, 1);
	GSPREC(3, 4, 0x9, 0xE, 9, 4);
	//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	GSPREC(0, 4, 0x8, 0xC, 7, 9);
	GSPREC(1, 5, 0x9, 0xD, 3, 1);
	GSPREC(2, 6, 0xA, 0xE, 13, 12);
	GSPREC(3, 7, 0xB, 0xF, 11, 14);
	GSPREC(0, 5, 0xA, 0xF, 2, 6);
	GSPREC(1, 6, 0xB, 0xC, 5, 10);
	GSPREC(2, 7, 0x8, 0xD, 4, 0);
	GSPREC(3, 4, 0x9, 0xE, 15, 8);

	//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	GSPREC(0, 4, 0x8, 0xC, 9, 0);
	GSPREC(1, 5, 0x9, 0xD, 5, 7);
	GSPREC(2, 6, 0xA, 0xE, 2, 4);
	GSPREC(3, 7, 0xB, 0xF, 10, 15);
	GSPREC(0, 5, 0xA, 0xF, 14, 1);
	GSPREC(1, 6, 0xB, 0xC, 11, 12);
	GSPREC(2, 7, 0x8, 0xD, 6, 8);
	GSPREC(3, 4, 0x9, 0xE, 3, 13);
	//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	GSPREC(0, 4, 0x8, 0xC, 2, 12);
	GSPREC(1, 5, 0x9, 0xD, 6, 10);
	GSPREC(2, 6, 0xA, 0xE, 0, 11);
	GSPREC(3, 7, 0xB, 0xF, 8, 3);
	GSPREC(0, 5, 0xA, 0xF, 4, 13);
	GSPREC(1, 6, 0xB, 0xC, 7, 5);
	GSPREC(2, 7, 0x8, 0xD, 15, 14);
	GSPREC(3, 4, 0x9, 0xE, 1, 9);

	//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	GSPREC(0, 4, 0x8, 0xC, 12, 5);
	GSPREC(1, 5, 0x9, 0xD, 1, 15);
	GSPREC(2, 6, 0xA, 0xE, 14, 13);
	GSPREC(3, 7, 0xB, 0xF, 4, 10);
	GSPREC(0, 5, 0xA, 0xF, 0, 7);
	GSPREC(1, 6, 0xB, 0xC, 6, 3);
	GSPREC(2, 7, 0x8, 0xD, 9, 2);
	GSPREC(3, 4, 0x9, 0xE, 8, 11);

//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	GSPREC(0, 4, 0x8, 0xC, 13, 11);
	GSPREC(1, 5, 0x9, 0xD, 7, 14);
	GSPREC(2, 6, 0xA, 0xE, 12, 1);
	GSPREC(3, 7, 0xB, 0xF, 3, 9);
	GSPREC(0, 5, 0xA, 0xF, 5, 0);
	GSPREC(1, 6, 0xB, 0xC, 15, 4);
	GSPREC(2, 7, 0x8, 0xD, 8, 6);
	GSPREC(3, 4, 0x9, 0xE, 2, 10);
//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	GSPREC(0, 4, 0x8, 0xC, 6, 15);
	GSPREC(1, 5, 0x9, 0xD, 14, 9);
	GSPREC(2, 6, 0xA, 0xE, 11, 3);
	GSPREC(3, 7, 0xB, 0xF, 0, 8);
	GSPREC(0, 5, 0xA, 0xF, 12, 2);
	GSPREC(1, 6, 0xB, 0xC, 13, 7);
	GSPREC(2, 7, 0x8, 0xD, 1, 4);
	GSPREC(3, 4, 0x9, 0xE, 10, 5);
//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	GSPREC(0, 4, 0x8, 0xC, 10, 2);
	GSPREC(1, 5, 0x9, 0xD, 8, 4);
	GSPREC(2, 6, 0xA, 0xE, 7, 6);
	GSPREC(3, 7, 0xB, 0xF, 1, 5);
	GSPREC(0, 5, 0xA, 0xF, 15, 11);
	GSPREC(1, 6, 0xB, 0xC, 9, 14);
	GSPREC(2, 7, 0x8, 0xD, 3, 12);
	GSPREC(3, 4, 0x9, 0xE, 13, 0);
//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	GSPREC(0, 4, 0x8, 0xC, 0, 1);
	GSPREC(1, 5, 0x9, 0xD, 2, 3);
	GSPREC(2, 6, 0xA, 0xE, 4, 5);
	GSPREC(3, 7, 0xB, 0xF, 6, 7);
	GSPREC(0, 5, 0xA, 0xF, 8, 9);
	GSPREC(1, 6, 0xB, 0xC, 10, 11);
	GSPREC(2, 7, 0x8, 0xD, 12, 13);
	GSPREC(3, 4, 0x9, 0xE, 14, 15);

//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	GSPREC(0, 4, 0x8, 0xC, 14, 10);
	GSPREC(1, 5, 0x9, 0xD, 4, 8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0, 2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5, 3);

//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5, 2);
	GSPREC(3, 7, 0xB, 0xF, 15, 13);
	GSPREC(0, 5, 0xA, 0xF, 10, 14);
	GSPREC(1, 6, 0xB, 0xC, 3, 6);
	GSPREC(2, 7, 0x8, 0xD, 7, 1);
	GSPREC(3, 4, 0x9, 0xE, 9, 4);
	//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	GSPREC(0, 4, 0x8, 0xC, 7, 9);
	GSPREC(1, 5, 0x9, 0xD, 3, 1);
	GSPREC(2, 6, 0xA, 0xE, 13, 12);
	GSPREC(3, 7, 0xB, 0xF, 11, 14);
	GSPREC(0, 5, 0xA, 0xF, 2, 6);
	GSPREC(1, 6, 0xB, 0xC, 5, 10);
	GSPREC(2, 7, 0x8, 0xD, 4, 0);
//	GSPREC(3, 4, 0x9, 0xE, 15, 8);


#if PRECALC64
	// only compute h6 & 7
//	h[6U] ^= v[6U] ^ v[14U];
	h[7] ^= v[7] ^ v[15];
#else
	//#pragma unroll 16
	for (uint32_t i = 0; i < 16; i++) {
		uint32_t j = i & 7U;
		h[j] ^= v[i];
	}
#endif
}


/* Second part (64-80) msg never change, store it */
__device__ static
void blake256_compress_8(uint32_t *const __restrict__ h, const uint32_t *const __restrict__ block)
{
	uint32_t /*_ALIGN(8)*/ m[16];
	uint32_t v[16];

	m[0] = block[0];
	m[1] = block[1];
	m[2] = block[2];
	m[3] = block[3];

	const uint32_t c_u256[16] = 
	{
		SPH_C32(0x243F6A88), SPH_C32(0x85A308D3),
		SPH_C32(0x13198A2E), SPH_C32(0x03707344),
		SPH_C32(0xA4093822), SPH_C32(0x299F31D0),
		SPH_C32(0x082EFA98), SPH_C32(0xEC4E6C89),
		SPH_C32(0x452821E6), SPH_C32(0x38D01377),
		SPH_C32(0xBE5466CF), SPH_C32(0x34E90C6C),
		SPH_C32(0xC0AC29B7), SPH_C32(0xC97C50DD),
		SPH_C32(0x3F84D5B5), SPH_C32(0xB5470917)
	};

	const uint32_t c_Padding[16] = {
		0, 0, 0, 0,
		0x80000000UL, 0, 0, 0,
		0, 0, 0, 0,
		0, 1, 0, 640,
	};


#pragma unroll
	for (int i = 4; i < 16; i++) 
	{
		m[i] = c_Padding[i];
	}

#pragma unroll
	for(int i = 0; i < 8; i++)
		v[i] = h[i];

	v[ 9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[13] = c_u256[5] ^ 640;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	v[0] = d_data[11];
	v[4] = d_data[12];
	v[8] = d_data[13];
	v[12] = d_data[14];

	GSPREC(1, 5, 0x9, 0xD,2,3);
	GSPREC(2, 6, 0xA, 0xE, 4,5);
	GSPREC(3, 7, 0xB, 0xF, 6,7);
	GSPREC(0, 5, 0xA, 0xF, 8,9);
	GSPREC(1, 6, 0xB, 0xC, 10,11);
	GSPREC(2, 7, 0x8, 0xD, 12,13);
	GSPREC(3, 4, 0x9, 0xE, 14,15);
	//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	GSPREC(0, 4, 0x8, 0xC, 14, 10);
	GSPREC(1, 5, 0x9, 0xD, 4, 8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0, 2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5, 3);
	//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5, 2);
	GSPREC(3, 7, 0xB, 0xF, 15, 13);
	GSPREC(0, 5, 0xA, 0xF, 10, 14);
	GSPREC(1, 6, 0xB, 0xC, 3, 6);
	GSPREC(2, 7, 0x8, 0xD, 7, 1);
	GSPREC(3, 4, 0x9, 0xE, 9, 4);
	//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	GSPREC(0, 4, 0x8, 0xC, 7, 9);
	GSPREC(1, 5, 0x9, 0xD, 3, 1);
	GSPREC(2, 6, 0xA, 0xE, 13, 12);
	GSPREC(3, 7, 0xB, 0xF, 11, 14);
	GSPREC(0, 5, 0xA, 0xF, 2, 6);
	GSPREC(1, 6, 0xB, 0xC, 5, 10);
	GSPREC(2, 7, 0x8, 0xD, 4, 0);
	GSPREC(3, 4, 0x9, 0xE, 15, 8);

	//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	GSPREC(0, 4, 0x8, 0xC, 9, 0);
	GSPREC(1, 5, 0x9, 0xD, 5, 7);
	GSPREC(2, 6, 0xA, 0xE, 2, 4);
	GSPREC(3, 7, 0xB, 0xF, 10, 15);
	GSPREC(0, 5, 0xA, 0xF, 14, 1);
	GSPREC(1, 6, 0xB, 0xC, 11, 12);
	GSPREC(2, 7, 0x8, 0xD, 6, 8);
	GSPREC(3, 4, 0x9, 0xE, 3, 13);
	//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	GSPREC(0, 4, 0x8, 0xC, 2, 12);
	GSPREC(1, 5, 0x9, 0xD, 6, 10);
	GSPREC(2, 6, 0xA, 0xE, 0, 11);
	GSPREC(3, 7, 0xB, 0xF, 8, 3);
	GSPREC(0, 5, 0xA, 0xF, 4, 13);
	GSPREC(1, 6, 0xB, 0xC, 7, 5);
	GSPREC(2, 7, 0x8, 0xD, 15, 14);
	GSPREC(3, 4, 0x9, 0xE, 1, 9);

	//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	GSPREC(0, 4, 0x8, 0xC, 12, 5);
	GSPREC(1, 5, 0x9, 0xD, 1, 15);
	GSPREC(2, 6, 0xA, 0xE, 14, 13);
	GSPREC(3, 7, 0xB, 0xF, 4, 10);
	GSPREC(0, 5, 0xA, 0xF, 0, 7);
	GSPREC(1, 6, 0xB, 0xC, 6, 3);
	GSPREC(2, 7, 0x8, 0xD, 9, 2);
	GSPREC(3, 4, 0x9, 0xE, 8, 11);

	//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	GSPREC(0, 4, 0x8, 0xC, 13, 11);
	GSPREC(1, 5, 0x9, 0xD, 7, 14);
	GSPREC(2, 6, 0xA, 0xE, 12, 1);
	GSPREC(3, 7, 0xB, 0xF, 3, 9);
	GSPREC(0, 5, 0xA, 0xF, 5, 0);
	GSPREC(1, 6, 0xB, 0xC, 15, 4);
	GSPREC(2, 7, 0x8, 0xD, 8, 6);
//	GSPREC(3, 4, 0x9, 0xE, 2, 10);
	//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },


#if PRECALC64
	// only compute h6 & 7
//	h[6] ^= v[6] ^ v[14];
	h[7] ^= v[7] ^ v[15];
#else
	//#pragma unroll 16
	for (uint32_t i = 0; i < 16; i++) {
		uint32_t j = i & 7U;
		h[j] ^= v[i];
	}
#endif
}


#if !PRECALC64 /* original method */
__global__
void blake256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce,
	const uint64_t highTarget, const int crcsum, const int rounds)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t h[8];

		#pragma unroll
		for(int i=0; i<8; i++) {
			h[i] = c_IV256[i];
		}

		if (crcsum != prevsum) {
			prevsum = crcsum;
			blake256_compress(h, c_data, 512, rounds);
			#pragma unroll
			for(int i=0; i<8; i++) {
				cache[i] = h[i];
			}
		} else {
			#pragma unroll
			for(int i=0; i<8; i++) {
				h[i] = cache[i];
			}
		}

		// ------ Close: Bytes 64 to 80 ------

		uint32_t ending[4];
		ending[0] = c_data[16];
		ending[1] = c_data[17];
		ending[2] = c_data[18];
		ending[3] = nonce; /* our tested value */

		blake256_compress(h, ending, 640, rounds);

		// not sure why, h[7] is ok
		h[6] = cuda_swab32(h[6]);

		// compare count of leading zeros h[6] + h[7]
		uint64_t high64 = ((uint64_t*)h)[3];
		if (high64 <= highTarget)
#if NBN == 2
		/* keep the smallest nonce, + extra one if found */
		if (resNonce[0] > nonce) {
			// printf("%llx %llx \n", high64, highTarget);
			resNonce[1] = resNonce[0];
			resNonce[0] = nonce;
		}
		else
			resNonce[1] = nonce;
#else
		resNonce[0] = nonce;
#endif
	}
}

__host__
uint32_t blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint64_t highTarget,
	const uint32_t crcsum, const int8_t rounds)
{
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);
	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake256_gpu_hash_80<<<grid, block, 0, gpustream[thr_id]>>>(threads, startNonce, d_resNonce[thr_id], highTarget, crcsum, (int) rounds);
	//hipDeviceSynchronize();
	if (hipSuccess == hipMemcpyAsync(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		//hipDeviceSynchronize(); /* seems no more required */
		result = h_resNonce[thr_id][0];
		for (int n=0; n < (NBN-1); n++)
			extra_results[thr_id][n] = h_resNonce[thr_id][n+1];
	}
	return result;
}

__host__
void blake256_cpu_setBlock_80(int thr_id, uint32_t *pdata, const uint32_t *ptarget)
{
	uint32_t data[20];
	memcpy(data, pdata, 80);
	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_data), data, sizeof(data), 0, hipMemcpyHostToDevice));
}
#else

/* ############################################################################################################################### */
/* Precalculated 1st 64-bytes block (midstate) method */

__global__
void blake256_gpu_hash_16(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce,
                          const uint32_t Target6, const uint32_t Target7, const int rounds, const bool trace)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t _ALIGN(16) h[8];

		#pragma unroll
		for(int i=0; i < 8; i++) {
			h[i] = d_data[i];
		}

		// ------ Close: Bytes 64 to 80 ------

		uint32_t _ALIGN(16) ending[4];
		ending[0] = d_data[8];
		ending[1] = d_data[9];
		ending[2] = d_data[10];
		ending[3] = nonce; /* our tested value */

		blake256_compress(h, ending, 640, rounds);

		if (h[7] <= Target7)
		{
#if NBN == 2
			uint32_t tmp = atomicCAS(resNonce, 0xffffffff, nonce);
			if(tmp != 0xffffffff)
				resNonce[1] = nonce;
#else
			resNonce[0] = nonce;
#endif
#ifdef _DEBUG
			if (trace) {
				uint64_t high64 = ((uint64_t*)h)[3];
				printf("gpu:  %16llx\n", high64);
				printf("gpu: %08x.%08x\n", h[7], h[6]);
				printf("tgt:  %16llx\n", highTarget);
			}
#endif
		}
	}
}


__global__
void blake256_gpu_hash_16_8(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce,
                            const uint32_t Target6, const uint32_t Target7, const int rounds, const bool trace)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t _ALIGN(16) h[8];

#pragma unroll
		for (int i = 0; i < 8; i++)
			h[i] = d_data[i];

		// ------ Close: Bytes 64 to 80 ------

		uint32_t _ALIGN(16) ending[4];
		ending[0] = d_data[8];
		ending[1] = d_data[9];
		ending[2] = d_data[10];
		ending[3] = nonce; /* our tested value */

		blake256_compress_8(h, ending);

		if (h[7] <= Target7)
		{
#if NBN == 2
			uint32_t tmp = atomicCAS(resNonce, 0xffffffff, nonce);
			if(tmp != 0xffffffff)
				resNonce[1] = nonce;
#else
			resNonce[0] = nonce;
#endif
#ifdef _DEBUG
			if (trace) {
				uint64_t high64 = ((uint64_t*)h)[3];
				printf("gpu:  %16llx\n", high64);
				printf("gpu: %08x.%08x\n", h[7], h[6]);
				printf("tgt:  %16llx\n", highTarget);
			}
#endif
		}
	}
}


__host__
static uint32_t blake256_cpu_hash_16(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint32_t Target6, const uint32_t Target7,
	const int8_t rounds)
{
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);

	CUDA_SAFE_CALL(hipMemsetAsync(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t), gpustream[thr_id]));

	if(rounds == 8)
		blake256_gpu_hash_16_8 << <grid, block, 0, gpustream[thr_id] >> > (threads, startNonce, d_resNonce[thr_id], Target6, Target7, (int)rounds, opt_tracegpu);
	else
	{
		if(rounds == 14)
			blake256_gpu_hash_16 << <grid, block, 0, gpustream[thr_id] >> > (threads, startNonce, d_resNonce[thr_id], Target6, Target7, (int)rounds, opt_tracegpu);
		else
			applog(LOG_ERR, "Number of blake rounds not supported");
	}
	CUDA_SAFE_CALL(hipMemcpyAsync(h_resNonce, d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost, gpustream[thr_id])); hipStreamSynchronize(gpustream[thr_id]);
	CUDA_SAFE_CALL(hipStreamSynchronize(gpustream[thr_id]));
	result = h_resNonce[0];

	for (int n=0; n < (NBN-1); n++)
		extra_results[thr_id][n] = h_resNonce[n + 1];
	return result;
}

__host__
static void blake256mid(uint32_t *output, const uint32_t *input, int8_t rounds = 14)
{
	sph_blake256_context ctx;

	/* in sph_blake.c */
	blake256_rounds = rounds;

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 64);

	memcpy(output, (void*)ctx.H, 32);
}

__host__
static void blake256_cpu_setBlock_16(int thr_id, uint32_t *penddata, const uint32_t *midstate, const uint32_t *ptarget)
{
	memcpy(h_data, midstate, 32);
	h_data[8] = penddata[0];
	h_data[9] = penddata[1];
	h_data[10] = penddata[2];

	// precalc v[0], v[4], v[8], v[12]
	h_data[11] = h_data[0] + (h_data[8] ^ 0x85A308D3) + h_data[4];
	h_data[14] = ROTL32(0xA4093822 ^ 640 ^ h_data[11], 16);
	h_data[13] = 0x243F6A88 + h_data[14];
	h_data[12] = ROTR32(h_data[4] ^ h_data[13], 12);
	h_data[11] += (h_data[9] ^ 0x243F6A88) + h_data[12];
	h_data[14] = ROTR32(h_data[14] ^ h_data[11], 8);
	h_data[13] += h_data[14];
	h_data[12] = ROTR32(h_data[12] ^ h_data[13], 7);

	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_data), h_data, 15 * 4, 0, hipMemcpyHostToDevice, gpustream[thr_id]));
}
#endif

extern int scanhash_blake256(int thr_id, uint32_t *pdata, uint32_t *ptarget,
	uint32_t max_nonce, uint32_t *hashes_done, int8_t blakerounds=14)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t _ALIGN(64) endiandata[20];
#if PRECALC64
	uint32_t _ALIGN(64) midstate[8];
#else
	uint32_t crcsum;
#endif
	unsigned int intensity = 28;
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, 1U << intensity);
	uint32_t throughput = min(throughputmax, max_nonce - first_nonce) & 0xfffffc00;

	int rc = 0;

	if (opt_benchmark)
	{
		ptarget[7] = 0x00000000;
		ptarget[6] = 0xffffffff;
	}
	uint32_t target6 = ptarget[6];
	uint32_t target7 = swab32(ptarget[7]); // don't ask me why

	if (opt_tracegpu)
	{
		/* test call from util.c */
		throughput = 1;
		for (int k = 0; k < 20; k++)
			pdata[k] = swab32(pdata[k]);
	}

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		CUDA_SAFE_CALL(hipHostMalloc(&h_data, 15 * sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipHostMalloc(&h_resNonce, NBN * sizeof(uint32_t)));
		CUDA_SAFE_CALL(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)));
		mining_has_stopped[thr_id] = false;
		init = true;
	}

#if PRECALC64
	for (int k = 0; k < 16; k++)
		be32enc(&endiandata[k], pdata[k]);
	blake256mid(midstate, endiandata, blakerounds);
	blake256_cpu_setBlock_16(thr_id, &pdata[16], midstate, ptarget);
#else
	blake256_cpu_setBlock_80(thr_id, pdata, ptarget);
	crcsum = crc32_u32t(pdata, 64);
#endif /* PRECALC64 */

	do {
#if PRECALC64
		// GPU HASH (second block only, first is midstate)
		uint32_t foundNonce =	blake256_cpu_hash_16(thr_id, throughput, pdata[19], target6, target7, blakerounds);
#else
		// GPU FULL HASH
		uint32_t foundNonce =	blake256_cpu_hash_80(thr_id, throughput, pdata[19], targetHigh, crcsum, blakerounds);
#endif
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundNonce != UINT32_MAX)
		{
			uint32_t vhashcpu[8] = { 0 };

			for (int k=0; k < 19; k++)
				be32enc(&endiandata[k], pdata[k]);

			if(opt_verify)
			{
				be32enc(&endiandata[19], foundNonce);
				blake256hash(vhashcpu, endiandata, blakerounds);
			}
			//applog(LOG_BLUE, "%08x %16llx", vhashcpu[6], targetHigh);
			if (vhashcpu[7] <= target7 && fulltest(vhashcpu, ptarget))
			{
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nounce %08x", thr_id, foundNonce);
				rc = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				pdata[19] = foundNonce;
#if NBN > 1
				if (extra_results[thr_id][0] != UINT32_MAX)
				{
					if(opt_verify)
					{
						be32enc(&endiandata[19], extra_results[thr_id][0]);
						blake256hash(vhashcpu, endiandata, blakerounds);
					}
					if (vhashcpu[7] <= target7 && fulltest(vhashcpu, ptarget))
					{
						pdata[21] = extra_results[thr_id][0];
						if(opt_benchmark) applog(LOG_INFO, "GPU #%d Found second nounce %08x", thr_id, extra_results[thr_id][0]);
//						applog(LOG_BLUE, "1:%x 2:%x", foundNonce, extra_results[thr_id][0]);
						rc = 2;
					}
					else
					{
						if(vhashcpu[7]>target7)
							applog(LOG_ERR, "GPU #%d: result for second nonce %08x does not validate on CPU!", device_map[thr_id], extra_results[thr_id][0]);
					}
					extra_results[thr_id][0] = UINT32_MAX;
				}
#endif
				//applog_hash((uint8_t*)ptarget);
				//applog_compare_hash((uint8_t*)vhashcpu,(uint8_t*)ptarget);
				return rc;
			}
			else
			{
				if(opt_debug)
				{
					applog_hash((uchar*)ptarget);
					applog_compare_hash((uchar*)vhashcpu, (uchar*)ptarget);
				}
				if(vhashcpu[7]>target7)
					applog(LOG_ERR, "GPU #%d: result for nonce %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;

	return rc;
}
