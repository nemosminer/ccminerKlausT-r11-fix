#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Nov. 2014
 */
extern "C" {
#include "sph/sph_blake.h"
}

#include "cuda_helper.h"
#include <memory.h>

#define UINT2(x,y) make_uint2(x,y)

//static __device__ __forceinline__ uint64_t cuda_swab32ll(uint64_t x) {
//	return MAKE_UINT64(cuda_swab32(_LOWORD(x)), cuda_swab32(_HIWORD(x)));
//}

__constant__ static uint32_t  c_data[3];

//__constant__ static uint8_t sigma[16][16];
static uint8_t  c_sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

__device__ __constant__ static uint32_t cpu_h[8];
/*
__device__ __constant__ static  uint32_t  u256[16] =
{
	0x243F6A88, 0x85A308D3,
	0x13198A2E, 0x03707344,
	0xA4093822, 0x299F31D0,
	0x082EFA98, 0xEC4E6C89,
	0x452821E6, 0x38D01377,
	0xBE5466CF, 0x34E90C6C,
	0xC0AC29B7, 0xC97C50DD,
	0x3F84D5B5, 0xB5470917
};
*/
static const uint32_t  c_u256[16] =
{
	0x243F6A88, 0x85A308D3,
	0x13198A2E, 0x03707344,
	0xA4093822, 0x299F31D0,
	0x082EFA98, 0xEC4E6C89,
	0x452821E6, 0x38D01377,
	0xBE5466CF, 0x34E90C6C,
	0xC0AC29B7, 0xC97C50DD,
	0x3F84D5B5, 0xB5470917
};

#define hostGS(a,b,c,d,x) { \
	const uint8_t idx1 = c_sigma[r][x]; \
	const uint8_t idx2 = c_sigma[r][x + 1]; \
	v[a] += (m[idx1] ^ c_u256[idx2]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ c_u256[idx1]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
	}

#define GSPREC(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ u256[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
		}

__constant__ uint64_t keccak_round_constants[24] = {
	0x0000000000000001ull, 0x0000000000008082ull,
	0x800000000000808aull, 0x8000000080008000ull,
	0x000000000000808bull, 0x0000000080000001ull,
	0x8000000080008081ull, 0x8000000000008009ull,
	0x000000000000008aull, 0x0000000000000088ull,
	0x0000000080008009ull, 0x000000008000000aull,
	0x000000008000808bull, 0x800000000000008bull,
	0x8000000000008089ull, 0x8000000000008003ull,
	0x8000000000008002ull, 0x8000000000000080ull,
	0x000000000000800aull, 0x800000008000000aull,
	0x8000000080008081ull, 0x8000000000008080ull,
	0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint2 keccak_round_constants35[24] = {
	{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
	{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
	{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
	{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
	{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
	{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
	{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
	{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
	{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
	{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
	{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
	{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
};

__host__ __forceinline__
static void blake256_compress1st(uint32_t *h, const uint32_t *block)
{
	uint32_t m[16];
	uint32_t v[16] =
	{
		0x6A09E667, 0xBB67AE85,
		0x3C6EF372, 0xA54FF53A,
		0x510E527F, 0x9B05688C,
		0x1F83D9AB, 0x5BE0CD19,
		0x243F6A88, 0x85A308D3,
		0x13198A2E, 0x03707344,
		0xA4093A22, 0x299F33D0,
		0x082EFA98, 0xEC4E6C89
	};
	
	for (int i = 0; i < 16; i++) {
		m[i] = block[i];
	}

	for (int r = 0; r < 14; r++) {
		/* column step */
		hostGS(0, 4, 0x8, 0xC, 0x0);
		hostGS(1, 5, 0x9, 0xD, 0x2);
		hostGS(2, 6, 0xA, 0xE, 0x4);
		hostGS(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		hostGS(0, 5, 0xA, 0xF, 0x8);
		hostGS(1, 6, 0xB, 0xC, 0xA);
		hostGS(2, 7, 0x8, 0xD, 0xC);
		hostGS(3, 4, 0x9, 0xE, 0xE);
	}

	h[0] ^= v[0] ^ v[8];
	h[1] ^= v[1] ^ v[9];
	h[2] ^= v[2] ^ v[10];
	h[3] ^= v[3] ^ v[11];
	h[4] ^= v[4] ^ v[12];
	h[5] ^= v[5] ^ v[13];
	h[6] ^= v[6] ^ v[14];
	h[7] ^= v[7] ^ v[15];
}
#define bitselect(a, b, c) ((a) ^ ((c) & ((b) ^ (a))))

static void __forceinline__ __device__ keccak_block(uint2 *s)
{
	uint2 bc[5], tmpxor[5], tmp1, tmp2;
	//	uint2 s[25];

#pragma unroll 1
	for (int i = 0; i < 24; i++)
	{
#pragma unroll
		for (uint32_t x = 0; x < 5; x++)
			tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		tmp1 = s[1] ^ bc[0];

		s[0] ^= bc[4];
		s[1] = ROL2(s[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(s[22] ^ bc[1], 61);
		s[22] = ROL2(s[14] ^ bc[3], 39);
		s[14] = ROL2(s[20] ^ bc[4], 18);
		s[20] = ROL2(s[2] ^ bc[1], 62);
		s[2] = ROL2(s[12] ^ bc[1], 43);
		s[12] = ROL2(s[13] ^ bc[2], 25);
		s[13] = ROL8(s[19] ^ bc[3]);
		s[19] = ROR8(s[23] ^ bc[2]);
		s[23] = ROL2(s[15] ^ bc[4], 41);
		s[15] = ROL2(s[4] ^ bc[3], 27);
		s[4] = ROL2(s[24] ^ bc[3], 14);
		s[24] = ROL2(s[21] ^ bc[0], 2);
		s[21] = ROL2(s[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(s[5] ^ bc[4], 36);
		s[5] = ROL2(s[3] ^ bc[2], 28);
		s[3] = ROL2(s[18] ^ bc[2], 21);
		s[18] = ROL2(s[17] ^ bc[1], 15);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[11] = ROL2(s[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(tmp1, 1);

		tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
		tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
		tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
		tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
		tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
		s[0] ^= keccak_round_constants35[i];
	}
}

//__launch_bounds__(256)
__global__ 
void blakeKeccak256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint32_t * Hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t h[8];
//		uint32_t input[4];
		const uint32_t T0 = 640;
		#pragma unroll 8
		for (int i = 0; i<8; i++) { h[i] = cpu_h[i];}

		uint32_t v[16];

		const uint32_t c_Padding[12] = {
			0x80000000, 0, 0, 0,
			0, 0, 0, 0,
			0, 1, 0, 640
		};

		const uint32_t  u256[16] = 
		{
			0x243F6A88, 0x85A308D3,
			0x13198A2E, 0x03707344,
			0xA4093822, 0x299F31D0,
			0x082EFA98, 0xEC4E6C89,
			0x452821E6, 0x38D01377,
			0xBE5466CF, 0x34E90C6C,
			0xC0AC29B7, 0xC97C50DD,
			0x3F84D5B5, 0xB5470917
		};

		uint32_t m[16] =
		{
			c_data[0], c_data[1], c_data[2], nonce,
			c_Padding[0], c_Padding[1], c_Padding[2], c_Padding[3],
			c_Padding[4], c_Padding[5], c_Padding[6], c_Padding[7],
			c_Padding[8], c_Padding[9], c_Padding[10], c_Padding[11]
		};

#pragma unroll 8
		for (int i = 0; i < 8; i++)
			v[i] = h[i];

		v[8] = u256[0];
		v[9] = u256[1];
		v[10] = u256[2];
		v[11] = u256[3];
		v[12] = u256[4] ^ T0;
		v[13] = u256[5] ^ T0;
		v[14] = u256[6];
		v[15] = u256[7];

		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);

		//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		GSPREC(0, 4, 0x8, 0xC, 9, 0);
		GSPREC(1, 5, 0x9, 0xD, 5, 7);
		GSPREC(2, 6, 0xA, 0xE, 2, 4);
		GSPREC(3, 7, 0xB, 0xF, 10, 15);
		GSPREC(0, 5, 0xA, 0xF, 14, 1);
		GSPREC(1, 6, 0xB, 0xC, 11, 12);
		GSPREC(2, 7, 0x8, 0xD, 6, 8);
		GSPREC(3, 4, 0x9, 0xE, 3, 13);
		//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		GSPREC(0, 4, 0x8, 0xC, 2, 12);
		GSPREC(1, 5, 0x9, 0xD, 6, 10);
		GSPREC(2, 6, 0xA, 0xE, 0, 11);
		GSPREC(3, 7, 0xB, 0xF, 8, 3);
		GSPREC(0, 5, 0xA, 0xF, 4, 13);
		GSPREC(1, 6, 0xB, 0xC, 7, 5);
		GSPREC(2, 7, 0x8, 0xD, 15, 14);
		GSPREC(3, 4, 0x9, 0xE, 1, 9);

		//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		GSPREC(0, 4, 0x8, 0xC, 12, 5);
		GSPREC(1, 5, 0x9, 0xD, 1, 15);
		GSPREC(2, 6, 0xA, 0xE, 14, 13);
		GSPREC(3, 7, 0xB, 0xF, 4, 10);
		GSPREC(0, 5, 0xA, 0xF, 0, 7);
		GSPREC(1, 6, 0xB, 0xC, 6, 3);
		GSPREC(2, 7, 0x8, 0xD, 9, 2);
		GSPREC(3, 4, 0x9, 0xE, 8, 11);

		//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		GSPREC(0, 4, 0x8, 0xC, 13, 11);
		GSPREC(1, 5, 0x9, 0xD, 7, 14);
		GSPREC(2, 6, 0xA, 0xE, 12, 1);
		GSPREC(3, 7, 0xB, 0xF, 3, 9);
		GSPREC(0, 5, 0xA, 0xF, 5, 0);
		GSPREC(1, 6, 0xB, 0xC, 15, 4);
		GSPREC(2, 7, 0x8, 0xD, 8, 6);
		GSPREC(3, 4, 0x9, 0xE, 2, 10);
		//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
		GSPREC(0, 4, 0x8, 0xC, 6, 15);
		GSPREC(1, 5, 0x9, 0xD, 14, 9);
		GSPREC(2, 6, 0xA, 0xE, 11, 3);
		GSPREC(3, 7, 0xB, 0xF, 0, 8);
		GSPREC(0, 5, 0xA, 0xF, 12, 2);
		GSPREC(1, 6, 0xB, 0xC, 13, 7);
		GSPREC(2, 7, 0x8, 0xD, 1, 4);
		GSPREC(3, 4, 0x9, 0xE, 10, 5);
		//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
		GSPREC(0, 4, 0x8, 0xC, 10, 2);
		GSPREC(1, 5, 0x9, 0xD, 8, 4);
		GSPREC(2, 6, 0xA, 0xE, 7, 6);
		GSPREC(3, 7, 0xB, 0xF, 1, 5);
		GSPREC(0, 5, 0xA, 0xF, 15, 11);
		GSPREC(1, 6, 0xB, 0xC, 9, 14);
		GSPREC(2, 7, 0x8, 0xD, 3, 12);
		GSPREC(3, 4, 0x9, 0xE, 13, 0);
		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);

		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);

		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);




		h[0] = cuda_swab32(h[0] ^ v[0] ^ v[8]);
		h[1] = cuda_swab32(h[1] ^ v[1] ^ v[9]);
		h[2] = cuda_swab32(h[2] ^ v[2] ^ v[10]);
		h[3] = cuda_swab32(h[3] ^ v[3] ^ v[11]);
		h[4] = cuda_swab32(h[4] ^ v[4] ^ v[12]);
		h[5] = cuda_swab32(h[5] ^ v[5] ^ v[13]);
		h[6] = cuda_swab32(h[6] ^ v[6] ^ v[14]);
		h[7] = cuda_swab32(h[7] ^ v[7] ^ v[15]);

		uint2 keccak_gpu_state[25] = {0};
		keccak_gpu_state[0].x = h[0];
		keccak_gpu_state[0].y = h[1];
		keccak_gpu_state[1].x = h[2];
		keccak_gpu_state[1].y = h[3];
		keccak_gpu_state[2].x = h[4];
		keccak_gpu_state[2].y = h[5];
		keccak_gpu_state[3].x = h[6];
		keccak_gpu_state[3].y = h[7];
		keccak_gpu_state[4] = UINT2(1, 0);

		keccak_gpu_state[16] = UINT2(0, 0x80000000);
		keccak_block(keccak_gpu_state);
		uint64_t *outputHash = (uint64_t *)Hash;
#pragma unroll 4
		for (int i = 0; i<4; i++)
			outputHash[i*threads + thread] = devectorize(keccak_gpu_state[i]);
	}



}


__global__ __launch_bounds__(256, 4)
void blake256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint32_t * Hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t h[8];
		//		uint32_t input[4];
#pragma unroll 8
		for (int i = 0; i<8; i++) { h[i] = cpu_h[i]; }

		uint32_t v[16];

		const uint32_t c_Padding[12] = {
			0x80000000, 0, 0, 0,
			0, 0, 0, 0,
			0, 1, 0, 640
		};
		const uint32_t  u256[16] =
		{
			0x243F6A88, 0x85A308D3,
			0x13198A2E, 0x03707344,
			0xA4093822, 0x299F31D0,
			0x082EFA98, 0xEC4E6C89,
			0x452821E6, 0x38D01377,
			0xBE5466CF, 0x34E90C6C,
			0xC0AC29B7, 0xC97C50DD,
			0x3F84D5B5, 0xB5470917
		};

		uint32_t m[16] =
		{
			c_data[0], c_data[1], c_data[2], nonce,
			c_Padding[0], c_Padding[1], c_Padding[2], c_Padding[3],
			c_Padding[4], c_Padding[5], c_Padding[6], c_Padding[7],
			c_Padding[8], c_Padding[9], c_Padding[10], c_Padding[11]
		};

#pragma unroll 8
		for (int i = 0; i < 8; i++)
			v[i] = h[i];

		v[8] = u256[0];
		v[9] = u256[1];
		v[10] = u256[2];
		v[11] = u256[3];
		v[12] = u256[4] ^ 640;
		v[13] = u256[5] ^ 640;
		v[14] = u256[6];
		v[15] = u256[7];

		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);

		//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		GSPREC(0, 4, 0x8, 0xC, 9, 0);
		GSPREC(1, 5, 0x9, 0xD, 5, 7);
		GSPREC(2, 6, 0xA, 0xE, 2, 4);
		GSPREC(3, 7, 0xB, 0xF, 10, 15);
		GSPREC(0, 5, 0xA, 0xF, 14, 1);
		GSPREC(1, 6, 0xB, 0xC, 11, 12);
		GSPREC(2, 7, 0x8, 0xD, 6, 8);
		GSPREC(3, 4, 0x9, 0xE, 3, 13);
		//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		GSPREC(0, 4, 0x8, 0xC, 2, 12);
		GSPREC(1, 5, 0x9, 0xD, 6, 10);
		GSPREC(2, 6, 0xA, 0xE, 0, 11);
		GSPREC(3, 7, 0xB, 0xF, 8, 3);
		GSPREC(0, 5, 0xA, 0xF, 4, 13);
		GSPREC(1, 6, 0xB, 0xC, 7, 5);
		GSPREC(2, 7, 0x8, 0xD, 15, 14);
		GSPREC(3, 4, 0x9, 0xE, 1, 9);

		//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		GSPREC(0, 4, 0x8, 0xC, 12, 5);
		GSPREC(1, 5, 0x9, 0xD, 1, 15);
		GSPREC(2, 6, 0xA, 0xE, 14, 13);
		GSPREC(3, 7, 0xB, 0xF, 4, 10);
		GSPREC(0, 5, 0xA, 0xF, 0, 7);
		GSPREC(1, 6, 0xB, 0xC, 6, 3);
		GSPREC(2, 7, 0x8, 0xD, 9, 2);
		GSPREC(3, 4, 0x9, 0xE, 8, 11);

		//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		GSPREC(0, 4, 0x8, 0xC, 13, 11);
		GSPREC(1, 5, 0x9, 0xD, 7, 14);
		GSPREC(2, 6, 0xA, 0xE, 12, 1);
		GSPREC(3, 7, 0xB, 0xF, 3, 9);
		GSPREC(0, 5, 0xA, 0xF, 5, 0);
		GSPREC(1, 6, 0xB, 0xC, 15, 4);
		GSPREC(2, 7, 0x8, 0xD, 8, 6);
		GSPREC(3, 4, 0x9, 0xE, 2, 10);
		//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
		GSPREC(0, 4, 0x8, 0xC, 6, 15);
		GSPREC(1, 5, 0x9, 0xD, 14, 9);
		GSPREC(2, 6, 0xA, 0xE, 11, 3);
		GSPREC(3, 7, 0xB, 0xF, 0, 8);
		GSPREC(0, 5, 0xA, 0xF, 12, 2);
		GSPREC(1, 6, 0xB, 0xC, 13, 7);
		GSPREC(2, 7, 0x8, 0xD, 1, 4);
		GSPREC(3, 4, 0x9, 0xE, 10, 5);
		//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
		GSPREC(0, 4, 0x8, 0xC, 10, 2);
		GSPREC(1, 5, 0x9, 0xD, 8, 4);
		GSPREC(2, 6, 0xA, 0xE, 7, 6);
		GSPREC(3, 7, 0xB, 0xF, 1, 5);
		GSPREC(0, 5, 0xA, 0xF, 15, 11);
		GSPREC(1, 6, 0xB, 0xC, 9, 14);
		GSPREC(2, 7, 0x8, 0xD, 3, 12);
		GSPREC(3, 4, 0x9, 0xE, 13, 0);
		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);

		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);

		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);

		h[0] = cuda_swab32(h[0] ^ v[0] ^ v[8]);
		h[1] = cuda_swab32(h[1] ^ v[1] ^ v[9]);
		h[2] = cuda_swab32(h[2] ^ v[2] ^ v[10]);
		h[3] = cuda_swab32(h[3] ^ v[3] ^ v[11]);
		h[4] = cuda_swab32(h[4] ^ v[4] ^ v[12]);
		h[5] = cuda_swab32(h[5] ^ v[5] ^ v[13]);
		h[6] = cuda_swab32(h[6] ^ v[6] ^ v[14]);
		h[7] = cuda_swab32(h[7] ^ v[7] ^ v[15]);

		Hash[((0 * threads) + thread)*2] = (h[0]);
		Hash[((0 * threads) + thread) * 2 + 1] = (h[1]);
		Hash[((1 * threads) + thread) * 2] = (h[2]);
		Hash[((1 * threads) + thread) * 2 + 1] = (h[3]);
		Hash[((2 * threads) + thread) * 2] = (h[4]);
		Hash[((2 * threads) + thread) * 2 + 1] = (h[5]);
		Hash[((3 * threads) + thread) * 2] = (h[6]);
		Hash[((3 * threads) + thread) * 2 + 1] = (h[7]);
	}
}

__host__
void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash)
{
	const uint32_t threadsperblock = 64;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	blake256_gpu_hash_80 <<<grid, block, 0, gpustream[thr_id]>>> (threads, startNonce, (uint32_t*)Hash);
	CUDA_SAFE_CALL(hipGetLastError());
}

__host__
void blake256_cpu_setBlock_80(int thr_id, uint32_t *pdata)
{
	uint32_t h[8] =
	{
		0x6A09E667, 0xBB67AE85,
		0x3C6EF372, 0xA54FF53A,
		0x510E527F, 0x9B05688C,
		0x1F83D9AB, 0x5BE0CD19
	};

	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_data), pdata + 16, 3 * sizeof(uint32_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]));

	blake256_compress1st(h, pdata);

	CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(cpu_h), h, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice, gpustream[thr_id]));
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
}

__host__
void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	blakeKeccak256_gpu_hash_80 << <grid, block, 0, gpustream[thr_id] >> > (threads, startNonce, (uint32_t *)Hash);
	CUDA_SAFE_CALL(hipGetLastError());
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
}