#include "hip/hip_runtime.h"
#include <memory.h>

#include "cuda_helper.h"

static __forceinline__ __device__
void Round512v35(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	const int ROT0, const int ROT1, const int ROT2, const int ROT3)
{
	p0 += p1; p1 = ROL2(p1, ROT0) ^ p0;
	p2 += p3; p3 = ROL2(p3, ROT1) ^ p2;
	p4 += p5; p5 = ROL2(p5, ROT2) ^ p4;
	p6 += p7; p7 = ROL2(p7, ROT3) ^ p6;
}

static __forceinline__ __device__
void Round_8_512v35(const uint2 *const __restrict__ ks, const uint2 *const __restrict__ ts,
	uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7, const int R)
{
	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 46, 36, 19, 37);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 33, 27, 14, 42);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 17, 49, 36, 39);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 44,  9, 54, 56);

	p0 += ks[(R+0) % 9];
	p1 += ks[(R+1) % 9];
	p2 += ks[(R+2) % 9];
	p3 += ks[(R+3) % 9];
	p4 += ks[(R+4) % 9];
	p5 += ks[(R+5) % 9] + ts[(R+0) % 3];
	p6 += ks[(R+6) % 9] + ts[(R+1) % 3];
	p7 += ks[(R+7) % 9] + make_uint2(R, 0);

	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 39, 30, 34, 24);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 13, 50, 10, 17);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 25, 29, 39, 43);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 8,  35, 56, 22);

	p0 += ks[(R+1) % 9];
	p1 += ks[(R+2) % 9];
	p2 += ks[(R+3) % 9];
	p3 += ks[(R+4) % 9];
	p4 += ks[(R+5) % 9];
	p5 += ks[(R+6) % 9] + ts[(R+1) % 3];
	p6 += ks[(R+7) % 9] + ts[(R+2) % 3];
	p7 += ks[(R+8) % 9] + make_uint2(R+1, 0);
}

static __forceinline__ __device__
void Round_8_512v35_final(const uint2 *const __restrict__ ks, const uint2 *const __restrict__ ts,
	uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7)
{
	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 46, 36, 19, 37);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 33, 27, 14, 42);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 17, 49, 36, 39);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 44, 9, 54, 56);

	p0 += ks[8];
	p1 += ks[0];
	p2 += ks[1];
	p3 += ks[2];
	p4 += ks[3];
	p5 += ks[4] + ts[2];
	p6 += ks[5] + ts[0];
	p7 += ks[6] + make_uint2(17, 0);

	Round512v35(p0, p1, p2, p3, p4, p5, p6, p7, 39, 30, 34, 24);
	Round512v35(p2, p1, p4, p7, p6, p5, p0, p3, 13, 50, 10, 17);
	Round512v35(p4, p1, p6, p3, p0, p5, p2, p7, 25, 29, 39, 43);
	Round512v35(p6, p1, p0, p7, p2, p5, p4, p3, 8,  35, 56, 22);

	p0 += ks[0];
	p1 += ks[1];
	p2 += ks[2];
	p3 += ks[3];
}



__global__ __launch_bounds__(256,4)
void skein256_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint64_t *outputHash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint2 skein_ks_parity = { 0xA9FC1A22, 0x1BD11BDA };

	const uint2 h2[9] = {
		{ 0x2FDB3E13, 0xCCD044A1 },
		{ 0x1A79A9EB, 0xE8359030 },
		{ 0x4F816E6F, 0x55AEA061 },
		{ 0xAE9B94DB, 0x2A2767A4 },
		{ 0x74DD7683, 0xEC06025E },
		{ 0xC4746251, 0xE7A436CD },
		{ 0x393AD185, 0xC36FBAF9 },
		{ 0x33EDFC13, 0x3EEDBA18 },
		{ 0xC73A4E2A, 0xB69D3CFC }
	};
	const uint2 t12[6] = {
		{ 0x20, 0 },
		{ 0, 0xf0000000 },
		{ 0x20, 0xf0000000 },
		{ 0x08, 0 },
		{ 0, 0xff000000 },
		{ 0x08, 0xff000000 }
	};

	if (thread < threads)
	{

		uint2 dt0,dt1,dt2,dt3;
		uint2 p0, p1, p2, p3, p4, p5, p6, p7;

		LOHI(dt0.x,dt0.y,outputHash[thread]);
		LOHI(dt1.x,dt1.y,outputHash[threads+thread]);
		LOHI(dt2.x,dt2.y,outputHash[2*threads+thread]);
		LOHI(dt3.x,dt3.y,outputHash[3*threads+thread]);

		p0 = h2[0] + dt0;
		p1 = h2[1] + dt1;
		p2 = h2[2] + dt2;
		p3 = h2[3] + dt3;
		p4 = h2[4];
		p5 = h2[5] + t12[0];
		p6 = h2[6] + t12[1];
		p7 = h2[7];

		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 1);
		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 3);
		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 5);
		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 7);
		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 9);
		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 11);
		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 13);
		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 15);
		Round_8_512v35(h2, t12, p0, p1, p2, p3, p4, p5, p6, p7, 17);

		p0 ^= dt0;
		p1 ^= dt1;
		p2 ^= dt2;
		p3 ^= dt3;

		uint2 h[9];
		h[0] = p0;
		h[1] = p1;
		h[2] = p2;
		h[3] = p3;
		h[4] = p4;
		h[5] = p5;
		h[6] = p6;
		h[7] = p7;
		h[8] = skein_ks_parity ^ h[0] ^ h[1] ^ h[2] ^ h[3] ^ h[4] ^ h[5] ^ h[6] ^ h[7];

		const uint2 *t = t12+3;
		p5 += t12[3];  //p5 already equal h[5]
		p6 += t12[4];

		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 1);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 3);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 5);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 7);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 9);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 11);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 13);
		Round_8_512v35(h, t, p0, p1, p2, p3, p4, p5, p6, p7, 15);
		Round_8_512v35_final(h, t, p0, p1, p2, p3, p4, p5, p6, p7);

		outputHash[thread]           = devectorize(p0);
		outputHash[threads+thread]   = devectorize(p1);
		outputHash[2*threads+thread] = devectorize(p2);
		outputHash[3*threads+thread] = devectorize(p3);
	}
}

__host__
void skein256_cpu_init(int thr_id, uint32_t threads)
{
}

__host__
void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash)
{
	const uint32_t threadsperblock = 32;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	skein256_gpu_hash_32<<<grid, block, 0, gpustream[thr_id]>>>(threads, startNounce, d_outputHash);
	CUDA_SAFE_CALL(hipGetLastError());
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
}

