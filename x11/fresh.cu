#include "hip/hip_runtime.h"
/**
 * Fresh algorithm
 */
extern "C" {
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}
#include "miner.h"
#include "cuda_helper.h"

// to test gpu hash on a null buffer
#define NULLTEST 0

extern void x11_shavite512_setBlock_80(int thr_id, void *pdata);
extern void x11_shavite512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
//extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void x11_echo512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *d_hash, uint32_t target, uint32_t *h_found);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *inpHashes,
											const uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse);

// CPU Hash
void fresh_hash(void *state, const void *input)
{
	// shavite-simd-shavite-simd-echo

	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashA hash
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, input, 80);
	sph_shavite512_close(&ctx_shavite, hashA);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hashA, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, hashB, 64);
	sph_shavite512_close(&ctx_shavite, hashA);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hashA, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hashB, 64);
	sph_echo512_close(&ctx_echo, hashA);

	memcpy(state, hash, 32);
}

extern int scanhash_fresh(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	static THREAD uint32_t *d_hash = nullptr;
	static THREAD uint32_t *h_found = nullptr;

	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];

	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, 1 << 19);
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 256 : 32;

	if (opt_benchmark)
		ptarget[7] = 0xf;

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == 1<<19)
			applog(LOG_INFO, "GPU #%d: using default intensity 19", device_map[thr_id]);
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (64 * sizeof(uint4)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		x11_simd512_cpu_init(thr_id, throughputmax);
		x11_echo512_cpu_init(thr_id, throughputmax);

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughputmax + 4));
		CUDA_SAFE_CALL(hipHostMalloc(&(h_found), 4 * sizeof(uint32_t)));

		cuda_check_cpu_init(thr_id, throughputmax);
		mining_has_stopped[thr_id] = false;

		init = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);
	
	x11_shavite512_setBlock_80(thr_id, (void*)endiandata);

	do {

		// GPU Hash

		x11_shavite512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash, simdthreads);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash, simdthreads);
		x11_echo512_cpu_hash_64_final(thr_id, throughput, pdata[19], d_hash, ptarget[7], h_found);
		hipStreamSynchronize(gpustream[thr_id]);

		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(h_found[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify){ be32enc(&endiandata[19], h_found[0]);
			fresh_hash(vhash64, endiandata);

			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_found[1] != 0xffffffff)
				{
					if(opt_verify){ be32enc(&endiandata[19], h_found[1]);
					fresh_hash(vhash64, endiandata);
					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						pdata[21] = h_found[1];
						res++;
						if (opt_benchmark)
							applog(LOG_INFO, "GPU #%d Found second nounce %08x", device_map[thr_id], h_found[1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[1]);
						}
					}

				}
				pdata[19] = h_found[0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", device_map[thr_id], h_found[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[0]);
				}
			}
		}
		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
