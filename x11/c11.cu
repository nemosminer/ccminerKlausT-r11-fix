#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
#include "cuda_helper.h"

#include <stdio.h>
#include <memory.h>

extern void quark_blake512_cpu_init(int thr_id);
extern void quark_blake512_cpu_setBlock_80(int thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_setBlock_80_multi(int thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_80_multi(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_bmw512_cpu_init(int thr_id, uint32_t threads);
extern void quark_bmw512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_init(int thr_id, uint32_t threads);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_keccak512_cpu_init(int thr_id, uint32_t threads);
extern void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void cuda_jh512Keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_luffaCubehash512_cpu_init(int thr_id, uint32_t threads);
extern void x11_luffaCubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_shavite512_cpu_init(int thr_id, uint32_t threads);
extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x11_echo512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *d_hash, uint32_t target, uint32_t *h_found);
extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *inpHashes,
										  const uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse);

extern "C" void c11hash(void *output, const void *input)
{
			// blake1-bmw2-grs3-skein4-jh5-keccak6-luffa7-cubehash8-shavite9-simd10-echo11
		sph_blake512_context ctx_blake;
		sph_bmw512_context ctx_bmw;
		sph_groestl512_context ctx_groestl;
		sph_jh512_context ctx_jh;
		sph_keccak512_context ctx_keccak;
		sph_skein512_context ctx_skein;
		sph_luffa512_context ctx_luffa;
		sph_cubehash512_context ctx_cubehash;
		sph_shavite512_context ctx_shavite;
		sph_simd512_context ctx_simd;
		sph_echo512_context ctx_echo;
		
		unsigned char hash[128];
		memset(hash, 0, sizeof hash);
		
		sph_blake512_init(&ctx_blake);
		sph_blake512(&ctx_blake, input, 80);
		sph_blake512_close(&ctx_blake, (void*)hash);
		
		sph_bmw512_init(&ctx_bmw);
		sph_bmw512(&ctx_bmw, (const void*)hash, 64);
		sph_bmw512_close(&ctx_bmw, (void*)hash);
		
		sph_groestl512_init(&ctx_groestl);
		sph_groestl512(&ctx_groestl, (const void*)hash, 64);
		sph_groestl512_close(&ctx_groestl, (void*)hash);
		
		sph_jh512_init(&ctx_jh);
		sph_jh512(&ctx_jh, (const void*)hash, 64);
		sph_jh512_close(&ctx_jh, (void*)hash);
		
		sph_keccak512_init(&ctx_keccak);
		sph_keccak512(&ctx_keccak, (const void*)hash, 64);
		sph_keccak512_close(&ctx_keccak, (void*)hash);
		
		sph_skein512_init(&ctx_skein);
		sph_skein512(&ctx_skein, (const void*)hash, 64);
		sph_skein512_close(&ctx_skein, (void*)hash);
		
		sph_luffa512_init(&ctx_luffa);
		sph_luffa512(&ctx_luffa, (const void*)hash, 64);
		sph_luffa512_close(&ctx_luffa, (void*)hash);
		
		sph_cubehash512_init(&ctx_cubehash);
		sph_cubehash512(&ctx_cubehash, (const void*)hash, 64);
		sph_cubehash512_close(&ctx_cubehash, (void*)hash);
		
		sph_shavite512_init(&ctx_shavite);
		sph_shavite512(&ctx_shavite, (const void*)hash, 64);
		sph_shavite512_close(&ctx_shavite, (void*)hash);
		
		sph_simd512_init(&ctx_simd);
		sph_simd512(&ctx_simd, (const void*)hash, 64);
		sph_simd512_close(&ctx_simd, (void*)hash);
		
		sph_echo512_init(&ctx_echo);
		sph_echo512(&ctx_echo, (const void*)hash, 64);
		sph_echo512_close(&ctx_echo, (void*)hash);
		
		memcpy(output, hash, 32);
}

static THREAD uint32_t *d_hash = nullptr;

int scanhash_c11(int thr_id, uint32_t *pdata,
				 uint32_t *ptarget, uint32_t max_nonce,
				 uint32_t *hashes_done)
{
	uint32_t foundnonces[2];
	const uint32_t first_nonce = pdata[19];

	hipDeviceProp_t props;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&props, device_map[thr_id]));
	static THREAD uint32_t throughputmax;

	if(opt_benchmark)
		ptarget[7] = 0x4f;

	static THREAD bool init = false;
	if(!init)
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));

		unsigned int intensity;
#if defined WIN32 && !defined _WIN64
		intensity = 256 * 256 * 16;
#else
		if(strstr(props.name, "Titan"))   intensity = (256 * 256 * 22);
		else if(strstr(props.name, "970"))		  intensity = (256 * 256 * 22);
		else if(strstr(props.name, "980"))    intensity = (256 * 256 * 22);
		else if(strstr(props.name, "1070"))   intensity = (256 * 256 * 22);
		else if(strstr(props.name, "1080"))   intensity = (256 * 256 * 22);
		else if(strstr(props.name, "750 Ti")) intensity = (256 * 256 * 20);
		else if(strstr(props.name, "750"))    intensity = (256 * 256 * 19);
		else if(strstr(props.name, "960"))    intensity = (256 * 256 * 19);
		else intensity = (256 * 256 * 19);
#endif
		throughputmax = device_intensity(device_map[thr_id], __func__, intensity);
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (64 * sizeof(uint4)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		quark_groestl512_cpu_init(thr_id, throughputmax);
		quark_bmw512_cpu_init(thr_id, throughputmax);
		x11_echo512_cpu_init(thr_id, throughputmax);
		x11_simd512_cpu_init(thr_id, throughputmax);

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * 4 * throughputmax));
		mining_has_stopped[thr_id] = false;
		init = true;
	}
	uint32_t throughput = min(throughputmax, max_nonce - first_nonce) & 0xfffffc00;
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 256 : 32;

	uint32_t endiandata[20];
	for(int k = 0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	quark_blake512_cpu_setBlock_80(thr_id, (uint64_t *)endiandata);

	do
	{

		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		cuda_jh512Keccak512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash, simdthreads);
		x11_echo512_cpu_hash_64_final(thr_id, throughput, pdata[19], d_hash, ptarget[7], foundnonces);
		hipStreamSynchronize(gpustream[thr_id]);
		if(stop_mining)
		{
			mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);
		}
		if (foundnonces[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify)
			{
				be32enc(&endiandata[19], foundnonces[0]);
				c11hash(vhash64, endiandata);
			}
			if(vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if(foundnonces[1] != 0xffffffff)
				{
					if(opt_verify)
					{
						be32enc(&endiandata[19], foundnonces[1]);
						c11hash(vhash64, endiandata);
					}
					if(vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundnonces[1];
						res++;
						if(opt_benchmark)
							applog(LOG_INFO, "GPU #%d: Found second nonce %08x", thr_id, foundnonces[1]);
					}
					else
					{
						if(vhash64[7] != Htarg)
						{
							applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundnonces[1]);
						}
					}
				}
				pdata[19] = foundnonces[0];
				if(opt_benchmark)
					applog(LOG_INFO, "GPU #%d: Found nonce %08x", thr_id, foundnonces[0]);
				return res;
			}
			else
			{
				if(vhash64[7] != Htarg)
				{
					applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundnonces[0]);
				}
			}
		}
		pdata[19] += throughput;
	} while(!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
