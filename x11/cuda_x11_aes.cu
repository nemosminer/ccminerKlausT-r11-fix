#include "hip/hip_runtime.h"
#include "cuda_helper.h"
__constant__ __align__(64) uint32_t d_AES0[256] = {
	0xA56363C6, 0x847C7CF8, 0x997777EE, 0x8D7B7BF6,
	0x0DF2F2FF, 0xBD6B6BD6, 0xB16F6FDE, 0x54C5C591,
	0x50303060, 0x03010102, 0xA96767CE, 0x7D2B2B56,
	0x19FEFEE7, 0x62D7D7B5, 0xE6ABAB4D, 0x9A7676EC,
	0x45CACA8F, 0x9D82821F, 0x40C9C989, 0x877D7DFA,
	0x15FAFAEF, 0xEB5959B2, 0xC947478E, 0x0BF0F0FB,
	0xECADAD41, 0x67D4D4B3, 0xFDA2A25F, 0xEAAFAF45,
	0xBF9C9C23, 0xF7A4A453, 0x967272E4, 0x5BC0C09B,
	0xC2B7B775, 0x1CFDFDE1, 0xAE93933D, 0x6A26264C,
	0x5A36366C, 0x413F3F7E, 0x02F7F7F5, 0x4FCCCC83,
	0x5C343468, 0xF4A5A551, 0x34E5E5D1, 0x08F1F1F9,
	0x937171E2, 0x73D8D8AB, 0x53313162, 0x3F15152A,
	0x0C040408, 0x52C7C795, 0x65232346, 0x5EC3C39D,
	0x28181830, 0xA1969637, 0x0F05050A, 0xB59A9A2F,
	0x0907070E, 0x36121224, 0x9B80801B, 0x3DE2E2DF,
	0x26EBEBCD, 0x6927274E, 0xCDB2B27F, 0x9F7575EA,
	0x1B090912, 0x9E83831D, 0x742C2C58, 0x2E1A1A34,
	0x2D1B1B36, 0xB26E6EDC, 0xEE5A5AB4, 0xFBA0A05B,
	0xF65252A4, 0x4D3B3B76, 0x61D6D6B7, 0xCEB3B37D,
	0x7B292952, 0x3EE3E3DD, 0x712F2F5E, 0x97848413,
	0xF55353A6, 0x68D1D1B9, 0x00000000, 0x2CEDEDC1,
	0x60202040, 0x1FFCFCE3, 0xC8B1B179, 0xED5B5BB6,
	0xBE6A6AD4, 0x46CBCB8D, 0xD9BEBE67, 0x4B393972,
	0xDE4A4A94, 0xD44C4C98, 0xE85858B0, 0x4ACFCF85,
	0x6BD0D0BB, 0x2AEFEFC5, 0xE5AAAA4F, 0x16FBFBED,
	0xC5434386, 0xD74D4D9A, 0x55333366, 0x94858511,
	0xCF45458A, 0x10F9F9E9, 0x06020204, 0x817F7FFE,
	0xF05050A0, 0x443C3C78, 0xBA9F9F25, 0xE3A8A84B,
	0xF35151A2, 0xFEA3A35D, 0xC0404080, 0x8A8F8F05,
	0xAD92923F, 0xBC9D9D21, 0x48383870, 0x04F5F5F1,
	0xDFBCBC63, 0xC1B6B677, 0x75DADAAF, 0x63212142,
	0x30101020, 0x1AFFFFE5, 0x0EF3F3FD, 0x6DD2D2BF,
	0x4CCDCD81, 0x140C0C18, 0x35131326, 0x2FECECC3,
	0xE15F5FBE, 0xA2979735, 0xCC444488, 0x3917172E,
	0x57C4C493, 0xF2A7A755, 0x827E7EFC, 0x473D3D7A,
	0xAC6464C8, 0xE75D5DBA, 0x2B191932, 0x957373E6,
	0xA06060C0, 0x98818119, 0xD14F4F9E, 0x7FDCDCA3,
	0x66222244, 0x7E2A2A54, 0xAB90903B, 0x8388880B,
	0xCA46468C, 0x29EEEEC7, 0xD3B8B86B, 0x3C141428,
	0x79DEDEA7, 0xE25E5EBC, 0x1D0B0B16, 0x76DBDBAD,
	0x3BE0E0DB, 0x56323264, 0x4E3A3A74, 0x1E0A0A14,
	0xDB494992, 0x0A06060C, 0x6C242448, 0xE45C5CB8,
	0x5DC2C29F, 0x6ED3D3BD, 0xEFACAC43, 0xA66262C4,
	0xA8919139, 0xA4959531, 0x37E4E4D3, 0x8B7979F2,
	0x32E7E7D5, 0x43C8C88B, 0x5937376E, 0xB76D6DDA,
	0x8C8D8D01, 0x64D5D5B1, 0xD24E4E9C, 0xE0A9A949,
	0xB46C6CD8, 0xFA5656AC, 0x07F4F4F3, 0x25EAEACF,
	0xAF6565CA, 0x8E7A7AF4, 0xE9AEAE47, 0x18080810,
	0xD5BABA6F, 0x887878F0, 0x6F25254A, 0x722E2E5C,
	0x241C1C38, 0xF1A6A657, 0xC7B4B473, 0x51C6C697,
	0x23E8E8CB, 0x7CDDDDA1, 0x9C7474E8, 0x211F1F3E,
	0xDD4B4B96, 0xDCBDBD61, 0x868B8B0D, 0x858A8A0F,
	0x907070E0, 0x423E3E7C, 0xC4B5B571, 0xAA6666CC,
	0xD8484890, 0x05030306, 0x01F6F6F7, 0x120E0E1C,
	0xA36161C2, 0x5F35356A, 0xF95757AE, 0xD0B9B969,
	0x91868617, 0x58C1C199, 0x271D1D3A, 0xB99E9E27,
	0x38E1E1D9, 0x13F8F8EB, 0xB398982B, 0x33111122,
	0xBB6969D2, 0x70D9D9A9, 0x898E8E07, 0xA7949433,
	0xB69B9B2D, 0x221E1E3C, 0x92878715, 0x20E9E9C9,
	0x49CECE87, 0xFF5555AA, 0x78282850, 0x7ADFDFA5,
	0x8F8C8C03, 0xF8A1A159, 0x80898909, 0x170D0D1A,
	0xDABFBF65, 0x31E6E6D7, 0xC6424284, 0xB86868D0,
	0xC3414182, 0xB0999929, 0x772D2D5A, 0x110F0F1E,
	0xCBB0B07B, 0xFC5454A8, 0xD6BBBB6D, 0x3A16162C
};

static __device__ __forceinline__
void aes_gpu_init(uint32_t *const sharedMemory)
{
	/* each thread startup will fill a uint32 */
	if (threadIdx.x < 256) {
		sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
		sharedMemory[threadIdx.x + 256] = ROL8(sharedMemory[threadIdx.x]);
		sharedMemory[threadIdx.x + 512] = ROL16(sharedMemory[threadIdx.x]);
		sharedMemory[threadIdx.x + 768] = ROL24(sharedMemory[threadIdx.x]);
	}
}

static __device__ __forceinline__
uint32_t bfe(uint32_t x, uint8_t bit, uint8_t numBits)
{
	uint32_t ret;
	asm("bfe.u32 %0, %1, %2, %3;" : "=r"(ret) : "r"(x), "r"((uint32_t)bit), "r"((uint32_t)numBits));
	return ret;
}

static __device__ __forceinline__
uint32_t bfi(uint32_t x, uint32_t a, uint32_t bit, uint32_t numBits)
{
	uint32_t ret;
	asm("bfi.b32 %0, %1, %2, %3, %4;" : "=r"(ret) : "r"(x), "r"(a), "r"(bit), "r"(numBits));
	return ret;
}

__device__ __forceinline__
static void aes_round(
const uint32_t *const __restrict__ sharedMemory,
const uint32_t x0, const uint32_t x1, const uint32_t x2, const uint32_t x3, const uint32_t k0,
	uint32_t &y0, uint32_t &y1, uint32_t &y2, uint32_t &y3 )
{
	const uint32_t a0 = (uint32_t) &sharedMemory[0];
	y0 = *(uint32_t *)(bfi(x0, a0, 2, 8))
		^ sharedMemory[bfe(x1, 8, 8) + 256]
		^ sharedMemory[bfe(x2, 16, 8) + 512]
		^ sharedMemory[bfe(x3, 24, 8) + 768] ^ k0;

	y1 = *(uint32_t *)(bfi(x1, a0, 2, 8))
		^sharedMemory[bfe(x2, 8, 8) + 256]
		^sharedMemory[bfe(x3, 16, 8) + 512]
		^ sharedMemory[bfe(x0, 24, 8) + 768];

	y2 = *(uint32_t *)(bfi(x2, a0, 2, 8))
	   ^sharedMemory[bfe(x3, 8, 8) + 256]
	   ^sharedMemory[bfe(x0, 16, 8) + 512]
	   ^ sharedMemory[bfe(x1, 24, 8) + 768];

	y3 = *(uint32_t *)(bfi(x3, a0, 2, 8))
	   ^ sharedMemory[bfe(x0, 8, 8) + 256]
	   ^ sharedMemory[bfe(x1, 16, 8) + 512]
	   ^ sharedMemory[bfe(x2, 24, 8) + 768];
}

__device__ __forceinline__
static void aes_round(
const uint32_t *const __restrict__ sharedMemory,
const uint32_t x0, const uint32_t x1, const uint32_t x2, const uint32_t x3,
	uint32_t &y0, uint32_t &y1, uint32_t &y2, uint32_t &y3)
{

	const uint32_t a0 = (uint32_t)&sharedMemory[0];
	y0 = *(uint32_t *)(bfi(x0, a0, 2, 8))
		^ sharedMemory[bfe(x1, 8, 8) + 256]
		^ sharedMemory[bfe(x2, 16, 8) + 512]
		^ sharedMemory[__byte_perm(x3, 0, 0x4443)+ 768];
	

	y1 = *(uint32_t *)(bfi(x1, a0, 2, 8))
		^ sharedMemory[bfe(x2, 8, 8) + 256]
		^ sharedMemory[bfe(x3, 16, 8) + 512]
		^ sharedMemory[bfe(x0, 24, 8) + 768];

	y2 = *(uint32_t *)(bfi(x2, a0, 2, 8))
		^ sharedMemory[bfe(x3, 8, 8) + 256]
		^ sharedMemory[bfe(x0, 16, 8) + 512]
		^ sharedMemory[bfe(x1, 24, 8) + 768];

	y3 = *(uint32_t *)(bfi(x3, a0, 2, 8))
		^ sharedMemory[bfe(x0, 8, 8) + 256]
		^ sharedMemory[bfe(x1, 16, 8) + 512]
		^ sharedMemory[bfe(x2, 24, 8) + 768];
}

