#include "hip/hip_runtime.h"
/**
 * S3 Hash (Also called 3S - Used by 1Coin)
 */

extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
}

#include "miner.h"
#include "cuda_helper.h"

#ifdef __cplusplus
#include <cstdint>
#else
#include <stdint.h>
#endif

extern void x11_shavite512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x11_shavite512_setBlock_80(int thr_id, void *pdata);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);
extern void quark_skein512_cpu_hash_64_final(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, uint32_t *h_found, uint32_t target);

/* CPU HASH */
void s3hash(void *output, const void *input)
{
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_skein512_context ctx_skein;

	unsigned char hash[64];

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, input, 80);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	memcpy(output, hash, 32);
}

/* Main S3 entry point */
extern int scanhash_s3(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	static THREAD uint32_t *d_hash = nullptr;
	static THREAD uint32_t *h_found = nullptr;

	const uint32_t first_nonce = pdata[19];
	unsigned int intensity = 20; // 256*256*8*2;
#ifdef WIN32
	// reduce by one the intensity on windows
	intensity--;
#endif
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, 1 << intensity);
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 256 : 32;
	if (opt_benchmark)
		ptarget[7] = 0x0000000fu;

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (64 * sizeof(uint4)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		x11_simd512_cpu_init(thr_id, throughputmax);
		quark_skein512_cpu_init(thr_id);

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughputmax));
		CUDA_SAFE_CALL(hipHostMalloc(&(h_found), 2 * sizeof(uint32_t)));

		cuda_check_cpu_init(thr_id, throughputmax);
		mining_has_stopped[thr_id] = false;

		init = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	x11_shavite512_setBlock_80(thr_id, (void*)endiandata);
	cuda_check_cpu_setTarget(ptarget, thr_id);

	do {
		x11_shavite512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash, simdthreads);
		quark_skein512_cpu_hash_64_final(thr_id, throughput, pdata[19], NULL, d_hash, h_found, ptarget[7]);

		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(h_found[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify){ be32enc(&endiandata[19], h_found[0]);
			s3hash(vhash64, endiandata);

			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (h_found[1] != 0xffffffff)
				{
					if(opt_verify){ be32enc(&endiandata[19], h_found[1]);
					s3hash(vhash64, endiandata);
					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						pdata[21] = h_found[1];
						res++;
						if (opt_benchmark)
							applog(LOG_INFO, "GPU #%d Found second nounce %08x", device_map[thr_id], h_found[1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[1]);
						}
					}

				}
				pdata[19] = h_found[0];
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nounce %08x", device_map[thr_id], h_found[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
				{
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[0]);
				}
			}
		}
		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
