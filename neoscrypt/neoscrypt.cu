#include "hip/hip_runtime.h"
#include <string.h>
#include "cuda_helper.h"
#include "miner.h"
#include "sph/neoscrypt.h"

extern void neoscrypt_setBlockTarget(int thr_id, uint32_t* pdata, const void *target);
extern void neoscrypt_cpu_init_2stream(int thr_id, uint32_t threads);
extern void neoscrypt_cpu_hash_k4_2stream(bool stratum, int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *result);
//extern void neoscrypt_cpu_hash_k4_52(int stratum, int thr_id, int threads, uint32_t startNounce, int order, uint32_t* foundnonce);
extern void get_cuda_arch_neo_tpruvot(int *version);
extern void get_cuda_arch_neo(int *version); 
extern int cuda_arch[MAX_GPUS];
void neoscrypt_init(int thr_id, uint32_t threads);
void neoscrypt_setBlockTarget_tpruvot(int thr_id, uint32_t* const pdata, uint32_t* const target);
void neoscrypt_hash_tpruvot(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resNonces, bool stratum);

int scanhash_neoscrypt(bool stratum, int thr_id, uint32_t *pdata,
					   uint32_t *ptarget, uint32_t max_nonce,
					   uint32_t *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput;
	static THREAD uint32_t throughputmax;

	static THREAD volatile bool init = false;
	static THREAD uint32_t hw_errors = 0;
	static THREAD uint32_t *foundNonce = nullptr;
	static THREAD bool use_tpruvot = false;

	if(opt_benchmark)
	{
		ptarget[7] = 0x01ff;
		stratum = 0;
	}

	if(!init)
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, device_map[thr_id]);
		unsigned int cc = props.major * 10 + props.minor;
		if(cc <= 30)
		{
			applog(LOG_ERR, "GPU #%d: this gpu is not supported", device_map[thr_id]);
			mining_has_stopped[thr_id] = true;
			proper_exit(2);
		}

		unsigned int intensity = (256 * 64 * 1); // -i 14
		if(strstr(props.name, " Xp"))
		{
			intensity = 256 * 64 * 5;
			use_tpruvot = true;
		}
		else if(strstr(props.name, "1080 Ti"))
		{
			intensity = 256 * 64 * 5;
			use_tpruvot = true;
		}
		else if(strstr(props.name, "1080"))
		{
			intensity = 256 * 64 * 5;
			use_tpruvot = true;
		}
		else if(strstr(props.name, "P104"))
		{
			intensity = 256 * 64 * 5;
			use_tpruvot = true;
		}
		else if(strstr(props.name, "P106"))
		{
			intensity = 256 * 64 * 5;
		}
		else if(strstr(props.name, "1070"))
		{
			intensity = 256 * 64 * 5;
		}
		else if(strstr(props.name, "970"))
		{
			intensity = (256 * 64 * 5);
		}
		else if(strstr(props.name, "980"))
		{
			intensity = (256 * 64 * 5);
		}
		else if(strstr(props.name, "980 Ti"))
		{
			intensity = (256 * 64 * 5);
		}
		else if(strstr(props.name, "750 Ti"))
		{
			intensity = (256 * 64 * 3);
		}
		else if(strstr(props.name, "750"))
		{
			intensity = (256 * 64 * 1);
		}
		else if(strstr(props.name, "960"))
		{
			intensity = (256 * 64 * 2);
		}
		else if(strstr(props.name, "950"))
		{
			intensity = (256 * 64 * 2);
		}
		if(cc == 70 || cc == 60) // Tesla P100/V100 or Titan V
		{
			intensity = 256 * 64 * 5;
			use_tpruvot = true;
		}

		throughputmax = device_intensity(device_map[thr_id], __func__, intensity) / 2;
		//		hipDeviceSetCacheConfig(hipFuncCachePreferL1);	
		if(throughputmax == intensity/2)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax*2));
		CUDA_SAFE_CALL(hipHostMalloc(&foundNonce, 2 * 4));

#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (32 * 128 * sizeof(uint64_t)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			proper_exit(2);
		}
#endif
		if(use_tpruvot)
		{
			get_cuda_arch_neo_tpruvot(&cuda_arch[thr_id]);
			neoscrypt_init(thr_id, throughputmax);
		}
		else
		{
			get_cuda_arch_neo(&cuda_arch[thr_id]);
			neoscrypt_cpu_init_2stream(thr_id, throughputmax);
		}
		mining_has_stopped[thr_id] = false;
		init = true;
	}
	throughput = min(throughputmax, (max_nonce - first_nonce) / 2) & 0xffffff00;

	uint32_t endiandata[20];
	for(int k = 0; k < 20; k++)
	{
		if(stratum)
			be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
		else endiandata[k] = pdata[k];
	}
	if(use_tpruvot)
		neoscrypt_setBlockTarget_tpruvot(thr_id, endiandata, ptarget);
	else
		neoscrypt_setBlockTarget(thr_id, endiandata, ptarget);
	

	do
	{
		if(use_tpruvot)
			neoscrypt_hash_tpruvot(thr_id, throughput, pdata[19], foundNonce, stratum);
		else
			neoscrypt_cpu_hash_k4_2stream(stratum, thr_id, throughput, pdata[19], foundNonce);
		if(stop_mining)
		{
			mining_has_stopped[thr_id] = true; pthread_exit(nullptr);
		}
		if(foundNonce[0] != 0xffffffff)
		{
			uint32_t vhash64[8]={0};
			if(opt_verify)
			{
				if(stratum)
					be32enc(&endiandata[19], foundNonce[0]);
				else
					endiandata[19] = foundNonce[0];
				neoscrypt((unsigned char*)endiandata, (unsigned char*)vhash64, 0x80000620);
			}
			if(vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget))
			{
				*hashes_done = pdata[19] - first_nonce + throughput;
				int res = 1;
				if(opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nonce %08x", device_map[thr_id], foundNonce[0]);
				pdata[19] = foundNonce[0];
				if(foundNonce[1] != 0xffffffff)
				{
					if(opt_verify)
					{
						if(stratum)
						{
							be32enc(&endiandata[19], foundNonce[1]);
						}
						else
						{
							endiandata[19] = foundNonce[1];
						}
						neoscrypt((unsigned char*)endiandata, (unsigned char*)vhash64, 0x80000620);
					}
					if(vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundNonce[1];
						res++;
						if(opt_benchmark)
							applog(LOG_INFO, "GPU #%d: Found second nonce %08x", device_map[thr_id], foundNonce[1]);
					}
					else
					{
						if(vhash64[7] != ptarget[7])
						{
							applog(LOG_WARNING, "GPU #%d: Second nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce[1]);
							hw_errors++;
						}
					}

				}
				return res;
			}
			else
			{
				if(vhash64[7] != ptarget[7])
				{
					applog(LOG_WARNING, "GPU #%d: Nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce[0]);
					hw_errors++;
				}
			}
//						if(hw_errors > 0) applog(LOG_WARNING, "Hardware errors: %u", hw_errors);
		}
		pdata[19] += throughput;
	} while(!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}

