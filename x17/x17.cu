#include "hip/hip_runtime.h"
/*
 * X17 algorithm built on cbuchner1's original X11
 *
 */

extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"

#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"

#include "sph/sph_sha2.h"
#include "sph/sph_haval.h"
}

#include "miner.h"
#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];
static THREAD uint32_t *h_found = nullptr;

extern void quark_blake512_cpu_init(int thr_id);
extern void quark_blake512_cpu_setBlock_80(int thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_setBlock_80_multi(int thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_80_multi(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_bmw512_cpu_init(int thr_id, uint32_t threads);
extern void quark_bmw512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void cuda_jh512Keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_luffaCubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x13_hamsi512_cpu_init(int thr_id, uint32_t threads);
extern void x13_hamsi512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int flag);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x17_sha512_cpu_init(int thr_id, uint32_t threads);
extern void x17_sha512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce,uint64_t *d_hash);

extern void x17_haval256_cpu_init(int thr_id, uint32_t threads);
extern void x17_haval256_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, uint32_t target, uint32_t *result);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *inpHashes,
											const uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse);

// X17 Hashfunktion
void x17hash(void *output, const void *input)
{
	// blake1-bmw2-grs3-skein4-jh5-keccak6-luffa7-cubehash8-shavite9-simd10-echo11-hamsi12-fugue13-shabal14-whirlpool15-sha512-haval17

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_fugue512_context ctx_fugue;
	sph_shabal512_context ctx_shabal;
	sph_whirlpool_context ctx_whirlpool;
	sph_sha512_context ctx_sha512;
	sph_haval256_5_context ctx_haval;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, hash);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, (const void*) hash, 64);
	sph_luffa512_close (&ctx_luffa, hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, hash);

	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, (const void*) hash, 64);
	sph_hamsi512_close(&ctx_hamsi, hash);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, (const void*) hash, 64);
	sph_fugue512_close(&ctx_fugue, hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, (const void*) hash, 64);
	sph_shabal512_close(&ctx_shabal, hash);

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool (&ctx_whirlpool, (const void*) hash, 64);
	sph_whirlpool_close(&ctx_whirlpool, hash);

	sph_sha512_init(&ctx_sha512);
	sph_sha512(&ctx_sha512,(const void*) hash, 64);
	sph_sha512_close(&ctx_sha512,(void*) hash);

	sph_haval256_5_init(&ctx_haval);
	sph_haval256_5(&ctx_haval,(const void*) hash, 64);
	sph_haval256_5_close(&ctx_haval,hash);

	memcpy(output, hash, 32);
}

static volatile bool init[MAX_GPUS] = { false };

extern int scanhash_x17(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	int intensity = 256 * 256 * 9;
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 256 : 32;
	if (device_sm[device_map[thr_id]] == 520)  intensity = 256 * 256 * 15;
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, intensity); // 19=256*256*8;
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;

	if (opt_benchmark)
		ptarget[7] = 0x03f;

	if (!init[thr_id])
	{
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (64 * sizeof(uint4)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		quark_groestl512_cpu_init(thr_id, throughputmax);
		quark_skein512_cpu_init(thr_id);
		quark_bmw512_cpu_init(thr_id, throughputmax);
		x11_simd512_cpu_init(thr_id, throughputmax);
		x11_echo512_cpu_init(thr_id, throughputmax);
		x13_hamsi512_cpu_init(thr_id, throughputmax);
		x13_fugue512_cpu_init(thr_id, throughputmax);
		x15_whirlpool_cpu_init(thr_id, throughputmax, 0);
		x17_sha512_cpu_init(thr_id, throughputmax);
		x17_haval256_cpu_init(thr_id, throughputmax);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16ULL * sizeof(uint32_t) * throughputmax));
		CUDA_SAFE_CALL(hipHostMalloc(&(h_found), 2 * sizeof(uint32_t)));
 
		mining_has_stopped[thr_id] = false;
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, (uint64_t *)endiandata);
	cuda_check_cpu_setTarget(ptarget, thr_id);

	do {
		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id]);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id]);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id]);
		cuda_jh512Keccak512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, pdata[19],d_hash[thr_id]);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], simdthreads);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id]);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19],d_hash[thr_id]);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19],d_hash[thr_id]);
		x17_sha512_cpu_hash_64(thr_id, throughput, pdata[19], (uint64_t*)d_hash[thr_id]);
		x17_haval256_cpu_hash_64(thr_id, throughput, pdata[19], d_hash[thr_id], ptarget[7], h_found);

		if(stop_mining)	{	mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);	}
		if(h_found[0] != 0xffffffff)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8] = {0};
			if(opt_verify)
			{
				be32enc(&endiandata[19], h_found[0]);
				x17hash(vhash64, endiandata);

			} if(vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if(h_found[1] != 0xffffffff)
				{
					if(opt_verify)
					{
						be32enc(&endiandata[19], h_found[1]);
						x13hash(vhash64, endiandata);
					} if(vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						pdata[21] = h_found[1];
						res++;
						if(opt_benchmark)
							applog(LOG_INFO, "GPU #%d Found second nonce %08x", device_map[thr_id], h_found[1]);
					}
					else
					{
						if(vhash64[7] != Htarg)
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[1]);
						}
					}

				}
				pdata[19] = h_found[0];
				if(opt_benchmark)
					applog(LOG_INFO, "GPU #%d Found nonce %08x", device_map[thr_id], h_found[0]);
				return res;
			}
			else
			{
				if(vhash64[7] != Htarg)
				{
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], h_found[0]);
				}
			}
		}
		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
