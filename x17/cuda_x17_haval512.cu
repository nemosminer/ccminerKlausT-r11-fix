#include "hip/hip_runtime.h"
/*
 * Haval-512 for X17
 *
 * Built on cbuchner1's implementation, actual hashing code
 * heavily based on phm's sgminer
 *
 */

/*
 * Haval-512 kernel implementation.
 *
 * ==========================(LICENSE BEGIN)============================
 *
 * Copyright (c) 2014  djm34
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
 * CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
 * TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
 * SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * ===========================(LICENSE END)=============================
 *
 * @author   phm <phm@inbox.com>
 */
#include <stdio.h>
#include <memory.h>

#define USE_SHARED 1

#include "cuda_helper.h"

static uint32_t *d_nonce[MAX_GPUS];

#define SPH_ROTL32(x, n)   ROTL32(x, n)
#define SPH_ROTR32(x, n)   ROTR32(x, n)

#define F1(x6, x5, x4, x3, x2, x1, x0) \
	(((x1) & ((x0) ^ (x4))) ^ ((x2) & (x5)) ^ ((x3) & (x6)) ^ (x0))

#define F2(x6, x5, x4, x3, x2, x1, x0) \
	(((x2) & (((x1) & ~(x3)) ^ ((x4) & (x5)) ^ (x6) ^ (x0))) \
	^ ((x4) & ((x1) ^ (x5))) ^ ((x3 & (x5)) ^ (x0)))

#define F3(x6, x5, x4, x3, x2, x1, x0) \
	(((x3) & (((x1) & (x2)) ^ (x6) ^ (x0))) \
	^ ((x1) & (x4)) ^ ((x2) & (x5)) ^ (x0))

#define F4(x6, x5, x4, x3, x2, x1, x0) \
	(((x3) & (((x1) & (x2)) ^ ((x4) | (x6)) ^ (x5))) \
	^ ((x4) & ((~(x2) & (x5)) ^ (x1) ^ (x6) ^ (x0))) \
	^ ((x2) & (x6)) ^ (x0))

#define F5(x6, x5, x4, x3, x2, x1, x0) \
	(((x0) & ~(((x1) & (x2) & (x3)) ^ (x5))) \
	^ ((x1) & (x4)) ^ ((x2) & (x5)) ^ ((x3) & (x6)))

#define STEP1(x7, x6, x5, x4, x3, x2, x1, x0, w) { \
		uint32_t t = F1(x3, x4, x1, x0, x5, x2, x6); \
		(x7) =(SPH_ROTR32(t, 7) + SPH_ROTR32((x7), 11) \
			+ (w)); \
	}

#define STEP2(x7, x6, x5, x4, x3, x2, x1, x0, w, c) { \
		uint32_t t = F2(x6, x2, x1, x0, x3, x4, x5); \
		(x7) =(SPH_ROTR32(t, 7) + SPH_ROTR32((x7), 11) \
			+ (w) + (c)); \
	}

#define STEP3(x7, x6, x5, x4, x3, x2, x1, x0, w, c) { \
		uint32_t t = F3(x2, x6, x0, x4, x3, x1, x5); \
		(x7) =(SPH_ROTR32(t, 7) + SPH_ROTR32((x7), 11) \
			+ (w) + (c)); \
	}

#define STEP4(x7, x6, x5, x4, x3, x2, x1, x0, w, c) { \
		uint32_t t = F4(x1, x5, x3, x2, x0, x4, x6); \
		(x7) =(SPH_ROTR32(t, 7) + SPH_ROTR32((x7), 11) \
			+ (w) + (c)); \
	}

#define STEP5(x7, x6, x5, x4, x3, x2, x1, x0, w, c) { \
		uint32_t t = F5(x2, x5, x0, x6, x4, x3, x1); \
		(x7) =(SPH_ROTR32(t, 7) + SPH_ROTR32((x7), 11) \
			+ (w) + (c)); \
	}

__global__
void x17_haval256_gpu_hash_64(uint32_t threads, uint32_t startNounce, const uint64_t *const __restrict__ g_hash, uint32_t target, uint32_t *const __restrict__ ret)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		uint32_t *inpHash = (uint32_t*)&g_hash[8 * thread];
		uint32_t hash[16];

		uint32_t buf[32] = {0};

		uint32_t s0 = 0x243F6A88;
		uint32_t s1 = 0x85A308D3;
		uint32_t s2 = 0x13198A2E;
		uint32_t s3 = 0x03707344;
		uint32_t s4 = 0xA4093822;
		uint32_t s5 = 0x299F31D0;
		uint32_t s6 = 0x082EFA98;
		uint32_t s7 = 0xEC4E6C89;

#pragma unroll
		for(int i = 0; i<16; i++)
		{
			hash[i] = inpHash[i];
		}

		///////// input big /////////////////////

#pragma unroll
		for(int i = 0; i<16; i++)
		{
				buf[i] = hash[i];
		}

		buf[16] = 0x00000001;
		buf[29] = 0x40290000;
		buf[30] = 0x00000200;

		STEP1(s7, s6, s5, s4, s3, s2, s1, s0, buf[ 0]); 
		STEP1(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 1]);
		STEP1(s5, s4, s3, s2, s1, s0, s7, s6, buf[ 2]);
		STEP1(s4, s3, s2, s1, s0, s7, s6, s5, buf[ 3]);
		STEP1(s3, s2, s1, s0, s7, s6, s5, s4, buf[ 4]);
		STEP1(s2, s1, s0, s7, s6, s5, s4, s3, buf[ 5]);
		STEP1(s1, s0, s7, s6, s5, s4, s3, s2, buf[ 6]);
		STEP1(s0, s7, s6, s5, s4, s3, s2, s1, buf[ 7]);
		STEP1(s7, s6, s5, s4, s3, s2, s1, s0, buf[ 8]);
		STEP1(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 9]);
		STEP1(s5, s4, s3, s2, s1, s0, s7, s6, buf[10]);
		STEP1(s4, s3, s2, s1, s0, s7, s6, s5, buf[11]);
		STEP1(s3, s2, s1, s0, s7, s6, s5, s4, buf[12]);
		STEP1(s2, s1, s0, s7, s6, s5, s4, s3, buf[13]);
		STEP1(s1, s0, s7, s6, s5, s4, s3, s2, buf[14]);
		STEP1(s0, s7, s6, s5, s4, s3, s2, s1, buf[15]);
		STEP1(s7, s6, s5, s4, s3, s2, s1, s0, buf[16]);
		STEP1(s6, s5, s4, s3, s2, s1, s0, s7, buf[17]);
		STEP1(s5, s4, s3, s2, s1, s0, s7, s6, buf[18]);
		STEP1(s4, s3, s2, s1, s0, s7, s6, s5, buf[19]);
		STEP1(s3, s2, s1, s0, s7, s6, s5, s4, buf[20]);
		STEP1(s2, s1, s0, s7, s6, s5, s4, s3, buf[21]);
		STEP1(s1, s0, s7, s6, s5, s4, s3, s2, buf[22]);
		STEP1(s0, s7, s6, s5, s4, s3, s2, s1, buf[23]);
		STEP1(s7, s6, s5, s4, s3, s2, s1, s0, buf[24]);
		STEP1(s6, s5, s4, s3, s2, s1, s0, s7, buf[25]);
		STEP1(s5, s4, s3, s2, s1, s0, s7, s6, buf[26]);
		STEP1(s4, s3, s2, s1, s0, s7, s6, s5, buf[27]);
		STEP1(s3, s2, s1, s0, s7, s6, s5, s4, buf[28]);
		STEP1(s2, s1, s0, s7, s6, s5, s4, s3, buf[29]);
		STEP1(s1, s0, s7, s6, s5, s4, s3, s2, buf[30]);
		STEP1(s0, s7, s6, s5, s4, s3, s2, s1, buf[31]);

		STEP2(s7, s6, s5, s4, s3, s2, s1, s0, buf[ 5], SPH_C32(0x452821E6));
		STEP2(s6, s5, s4, s3, s2, s1, s0, s7, buf[14], SPH_C32(0x38D01377));
		STEP2(s5, s4, s3, s2, s1, s0, s7, s6, buf[26], SPH_C32(0xBE5466CF));
		STEP2(s4, s3, s2, s1, s0, s7, s6, s5, buf[18], SPH_C32(0x34E90C6C));
		STEP2(s3, s2, s1, s0, s7, s6, s5, s4, buf[11], SPH_C32(0xC0AC29B7));
		STEP2(s2, s1, s0, s7, s6, s5, s4, s3, buf[28], SPH_C32(0xC97C50DD));
		STEP2(s1, s0, s7, s6, s5, s4, s3, s2, buf[ 7], SPH_C32(0x3F84D5B5));
		STEP2(s0, s7, s6, s5, s4, s3, s2, s1, buf[16], SPH_C32(0xB5470917));
		STEP2(s7, s6, s5, s4, s3, s2, s1, s0, buf[ 0], SPH_C32(0x9216D5D9));
		STEP2(s6, s5, s4, s3, s2, s1, s0, s7, buf[23], SPH_C32(0x8979FB1B));
		STEP2(s5, s4, s3, s2, s1, s0, s7, s6, buf[20], SPH_C32(0xD1310BA6));
		STEP2(s4, s3, s2, s1, s0, s7, s6, s5, buf[22], SPH_C32(0x98DFB5AC));
		STEP2(s3, s2, s1, s0, s7, s6, s5, s4, buf[ 1], SPH_C32(0x2FFD72DB));
		STEP2(s2, s1, s0, s7, s6, s5, s4, s3, buf[10], SPH_C32(0xD01ADFB7));
		STEP2(s1, s0, s7, s6, s5, s4, s3, s2, buf[ 4], SPH_C32(0xB8E1AFED));
		STEP2(s0, s7, s6, s5, s4, s3, s2, s1, buf[ 8], SPH_C32(0x6A267E96));
		STEP2(s7, s6, s5, s4, s3, s2, s1, s0, buf[30], SPH_C32(0xBA7C9045));
		STEP2(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 3], SPH_C32(0xF12C7F99));
		STEP2(s5, s4, s3, s2, s1, s0, s7, s6, buf[21], SPH_C32(0x24A19947));
		STEP2(s4, s3, s2, s1, s0, s7, s6, s5, buf[ 9], SPH_C32(0xB3916CF7));
		STEP2(s3, s2, s1, s0, s7, s6, s5, s4, buf[17], SPH_C32(0x0801F2E2));
		STEP2(s2, s1, s0, s7, s6, s5, s4, s3, buf[24], SPH_C32(0x858EFC16));
		STEP2(s1, s0, s7, s6, s5, s4, s3, s2, buf[29], SPH_C32(0x636920D8));
		STEP2(s0, s7, s6, s5, s4, s3, s2, s1, buf[ 6], SPH_C32(0x71574E69));
		STEP2(s7, s6, s5, s4, s3, s2, s1, s0, buf[19], SPH_C32(0xA458FEA3));
		STEP2(s6, s5, s4, s3, s2, s1, s0, s7, buf[12], SPH_C32(0xF4933D7E));
		STEP2(s5, s4, s3, s2, s1, s0, s7, s6, buf[15], SPH_C32(0x0D95748F));
		STEP2(s4, s3, s2, s1, s0, s7, s6, s5, buf[13], SPH_C32(0x728EB658));
		STEP2(s3, s2, s1, s0, s7, s6, s5, s4, buf[ 2], SPH_C32(0x718BCD58));
		STEP2(s2, s1, s0, s7, s6, s5, s4, s3, buf[25], SPH_C32(0x82154AEE));
		STEP2(s1, s0, s7, s6, s5, s4, s3, s2, buf[31], SPH_C32(0x7B54A41D));
		STEP2(s0, s7, s6, s5, s4, s3, s2, s1, buf[27], SPH_C32(0xC25A59B5));
		STEP3(s7, s6, s5, s4, s3, s2, s1, s0, buf[19], SPH_C32(0x9C30D539));
		STEP3(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 9], SPH_C32(0x2AF26013));
		STEP3(s5, s4, s3, s2, s1, s0, s7, s6, buf[ 4], SPH_C32(0xC5D1B023));
		STEP3(s4, s3, s2, s1, s0, s7, s6, s5, buf[20], SPH_C32(0x286085F0));
		STEP3(s3, s2, s1, s0, s7, s6, s5, s4, buf[28], SPH_C32(0xCA417918));
		STEP3(s2, s1, s0, s7, s6, s5, s4, s3, buf[17], SPH_C32(0xB8DB38EF));
		STEP3(s1, s0, s7, s6, s5, s4, s3, s2, buf[ 8], SPH_C32(0x8E79DCB0));
		STEP3(s0, s7, s6, s5, s4, s3, s2, s1, buf[22], SPH_C32(0x603A180E));
		STEP3(s7, s6, s5, s4, s3, s2, s1, s0, buf[29], SPH_C32(0x6C9E0E8B));
		STEP3(s6, s5, s4, s3, s2, s1, s0, s7, buf[14], SPH_C32(0xB01E8A3E));
		STEP3(s5, s4, s3, s2, s1, s0, s7, s6, buf[25], SPH_C32(0xD71577C1));
		STEP3(s4, s3, s2, s1, s0, s7, s6, s5, buf[12], SPH_C32(0xBD314B27));
		STEP3(s3, s2, s1, s0, s7, s6, s5, s4, buf[24], SPH_C32(0x78AF2FDA));
		STEP3(s2, s1, s0, s7, s6, s5, s4, s3, buf[30], SPH_C32(0x55605C60));
		STEP3(s1, s0, s7, s6, s5, s4, s3, s2, buf[16], SPH_C32(0xE65525F3));
		STEP3(s0, s7, s6, s5, s4, s3, s2, s1, buf[26], SPH_C32(0xAA55AB94));
		STEP3(s7, s6, s5, s4, s3, s2, s1, s0, buf[31], SPH_C32(0x57489862));
		STEP3(s6, s5, s4, s3, s2, s1, s0, s7, buf[15], SPH_C32(0x63E81440));
		STEP3(s5, s4, s3, s2, s1, s0, s7, s6, buf[ 7], SPH_C32(0x55CA396A));
		STEP3(s4, s3, s2, s1, s0, s7, s6, s5, buf[ 3], SPH_C32(0x2AAB10B6));
		STEP3(s3, s2, s1, s0, s7, s6, s5, s4, buf[ 1], SPH_C32(0xB4CC5C34));
		STEP3(s2, s1, s0, s7, s6, s5, s4, s3, buf[ 0], SPH_C32(0x1141E8CE));
		STEP3(s1, s0, s7, s6, s5, s4, s3, s2, buf[18], SPH_C32(0xA15486AF));
		STEP3(s0, s7, s6, s5, s4, s3, s2, s1, buf[27], SPH_C32(0x7C72E993));
		STEP3(s7, s6, s5, s4, s3, s2, s1, s0, buf[13], SPH_C32(0xB3EE1411));
		STEP3(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 6], SPH_C32(0x636FBC2A));
		STEP3(s5, s4, s3, s2, s1, s0, s7, s6, buf[21], SPH_C32(0x2BA9C55D));
		STEP3(s4, s3, s2, s1, s0, s7, s6, s5, buf[10], SPH_C32(0x741831F6));
		STEP3(s3, s2, s1, s0, s7, s6, s5, s4, buf[23], SPH_C32(0xCE5C3E16));
		STEP3(s2, s1, s0, s7, s6, s5, s4, s3, buf[11], SPH_C32(0x9B87931E));
		STEP3(s1, s0, s7, s6, s5, s4, s3, s2, buf[ 5], SPH_C32(0xAFD6BA33));
		STEP3(s0, s7, s6, s5, s4, s3, s2, s1, buf[ 2], SPH_C32(0x6C24CF5C));

		STEP4(s7, s6, s5, s4, s3, s2, s1, s0, buf[24], SPH_C32(0x7A325381));
		STEP4(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 4], SPH_C32(0x28958677));
		STEP4(s5, s4, s3, s2, s1, s0, s7, s6, buf[ 0], SPH_C32(0x3B8F4898));
		STEP4(s4, s3, s2, s1, s0, s7, s6, s5, buf[14], SPH_C32(0x6B4BB9AF));
		STEP4(s3, s2, s1, s0, s7, s6, s5, s4, buf[ 2], SPH_C32(0xC4BFE81B));
		STEP4(s2, s1, s0, s7, s6, s5, s4, s3, buf[ 7], SPH_C32(0x66282193));
		STEP4(s1, s0, s7, s6, s5, s4, s3, s2, buf[28], SPH_C32(0x61D809CC));
		STEP4(s0, s7, s6, s5, s4, s3, s2, s1, buf[23], SPH_C32(0xFB21A991));
		STEP4(s7, s6, s5, s4, s3, s2, s1, s0, buf[26], SPH_C32(0x487CAC60));
		STEP4(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 6], SPH_C32(0x5DEC8032));
		STEP4(s5, s4, s3, s2, s1, s0, s7, s6, buf[30], SPH_C32(0xEF845D5D));
		STEP4(s4, s3, s2, s1, s0, s7, s6, s5, buf[20], SPH_C32(0xE98575B1));
		STEP4(s3, s2, s1, s0, s7, s6, s5, s4, buf[18], SPH_C32(0xDC262302));
		STEP4(s2, s1, s0, s7, s6, s5, s4, s3, buf[25], SPH_C32(0xEB651B88));
		STEP4(s1, s0, s7, s6, s5, s4, s3, s2, buf[19], SPH_C32(0x23893E81));
		STEP4(s0, s7, s6, s5, s4, s3, s2, s1, buf[ 3], SPH_C32(0xD396ACC5));
		STEP4(s7, s6, s5, s4, s3, s2, s1, s0, buf[22], SPH_C32(0x0F6D6FF3));
		STEP4(s6, s5, s4, s3, s2, s1, s0, s7, buf[11], SPH_C32(0x83F44239));
		STEP4(s5, s4, s3, s2, s1, s0, s7, s6, buf[31], SPH_C32(0x2E0B4482));
		STEP4(s4, s3, s2, s1, s0, s7, s6, s5, buf[21], SPH_C32(0xA4842004));
		STEP4(s3, s2, s1, s0, s7, s6, s5, s4, buf[ 8], SPH_C32(0x69C8F04A));
		STEP4(s2, s1, s0, s7, s6, s5, s4, s3, buf[27], SPH_C32(0x9E1F9B5E));
		STEP4(s1, s0, s7, s6, s5, s4, s3, s2, buf[12], SPH_C32(0x21C66842));
		STEP4(s0, s7, s6, s5, s4, s3, s2, s1, buf[ 9], SPH_C32(0xF6E96C9A));
		STEP4(s7, s6, s5, s4, s3, s2, s1, s0, buf[ 1], SPH_C32(0x670C9C61));
		STEP4(s6, s5, s4, s3, s2, s1, s0, s7, buf[29], SPH_C32(0xABD388F0));
		STEP4(s5, s4, s3, s2, s1, s0, s7, s6, buf[ 5], SPH_C32(0x6A51A0D2));
		STEP4(s4, s3, s2, s1, s0, s7, s6, s5, buf[15], SPH_C32(0xD8542F68));
		STEP4(s3, s2, s1, s0, s7, s6, s5, s4, buf[17], SPH_C32(0x960FA728));
		STEP4(s2, s1, s0, s7, s6, s5, s4, s3, buf[10], SPH_C32(0xAB5133A3));
		STEP4(s1, s0, s7, s6, s5, s4, s3, s2, buf[16], SPH_C32(0x6EEF0B6C));
		STEP4(s0, s7, s6, s5, s4, s3, s2, s1, buf[13], SPH_C32(0x137A3BE4));

		STEP5(s7, s6, s5, s4, s3, s2, s1, s0, buf[27], SPH_C32(0xBA3BF050));
		STEP5(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 3], SPH_C32(0x7EFB2A98));
		STEP5(s5, s4, s3, s2, s1, s0, s7, s6, buf[21], SPH_C32(0xA1F1651D));
		STEP5(s4, s3, s2, s1, s0, s7, s6, s5, buf[26], SPH_C32(0x39AF0176));
		STEP5(s3, s2, s1, s0, s7, s6, s5, s4, buf[17], SPH_C32(0x66CA593E));
		STEP5(s2, s1, s0, s7, s6, s5, s4, s3, buf[11], SPH_C32(0x82430E88));
		STEP5(s1, s0, s7, s6, s5, s4, s3, s2, buf[20], SPH_C32(0x8CEE8619));
		STEP5(s0, s7, s6, s5, s4, s3, s2, s1, buf[29], SPH_C32(0x456F9FB4));
			
		STEP5(s7, s6, s5, s4, s3, s2, s1, s0, buf[19], SPH_C32(0x7D84A5C3));
		STEP5(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 0], SPH_C32(0x3B8B5EBE));
		STEP5(s5, s4, s3, s2, s1, s0, s7, s6, buf[12], SPH_C32(0xE06F75D8));
		STEP5(s4, s3, s2, s1, s0, s7, s6, s5, buf[ 7], SPH_C32(0x85C12073));
		STEP5(s3, s2, s1, s0, s7, s6, s5, s4, buf[13], SPH_C32(0x401A449F));
		STEP5(s2, s1, s0, s7, s6, s5, s4, s3, buf[ 8], SPH_C32(0x56C16AA6));
		STEP5(s1, s0, s7, s6, s5, s4, s3, s2, buf[31], SPH_C32(0x4ED3AA62));
		STEP5(s0, s7, s6, s5, s4, s3, s2, s1, buf[10], SPH_C32(0x363F7706));
			
		STEP5(s7, s6, s5, s4, s3, s2, s1, s0, buf[ 5], SPH_C32(0x1BFEDF72));
		STEP5(s6, s5, s4, s3, s2, s1, s0, s7, buf[ 9], SPH_C32(0x429B023D));
		STEP5(s5, s4, s3, s2, s1, s0, s7, s6, buf[14], SPH_C32(0x37D0D724));
		STEP5(s4, s3, s2, s1, s0, s7, s6, s5, buf[30], SPH_C32(0xD00A1248));
		STEP5(s3, s2, s1, s0, s7, s6, s5, s4, buf[18], SPH_C32(0xDB0FEAD3));
		STEP5(s2, s1, s0, s7, s6, s5, s4, s3, buf[ 6], SPH_C32(0x49F1C09B));
		STEP5(s1, s0, s7, s6, s5, s4, s3, s2, buf[28], SPH_C32(0x075372C9));
		STEP5(s0, s7, s6, s5, s4, s3, s2, s1, buf[24], SPH_C32(0x80991B7B));
			
		STEP5(s7, s6, s5, s4, s3, s2, s1, s0, buf[ 2], SPH_C32(0x25D479D8));
		/*
		STEP5(s6, s5, s4, s3, s2, s1, s0, s7, buf[23], SPH_C32(0xF6E8DEF7));
		STEP5(s5, s4, s3, s2, s1, s0, s7, s6, buf[16], SPH_C32(0xE3FE501A));
		STEP5(s4, s3, s2, s1, s0, s7, s6, s5, buf[22], SPH_C32(0xB6794C3B));
		STEP5(s3, s2, s1, s0, s7, s6, s5, s4, buf[ 4], SPH_C32(0x976CE0BD));
		STEP5(s2, s1, s0, s7, s6, s5, s4, s3, buf[ 1], SPH_C32(0x04C006BA));
		STEP5(s1, s0, s7, s6, s5, s4, s3, s2, buf[25], SPH_C32(0xC1A94FB6));
		STEP5(s0, s7, s6, s5, s4, s3, s2, s1, buf[15], SPH_C32(0x409F60C4));

		inpHash[0] = s0 + 0x243F6A88;
		inpHash[1] = s1 + 0x85A308D3;
		inpHash[2] = s2 + 0x13198A2E;
		inpHash[3] = s3 + 0x03707344;
		inpHash[4] = s4 + 0xA4093822;
		inpHash[5] = s5 + 0x299F31D0;
		inpHash[6] = s6 + 0x082EFA98;
		inpHash[7] = s7 + 0xEC4E6C89;
		*/
		if(s7 + 0xEC4E6C89 <= target)
		{
			uint32_t tmp = atomicExch(ret, startNounce + thread);
			if(tmp != 0xffffffff)
				ret[1] = tmp;
		}

	} // threads
}

__host__
void x17_haval256_cpu_init(int thr_id, uint32_t threads)
{
	hipMalloc(&d_nonce[thr_id], 2 * sizeof(uint32_t));
}

__host__
void x17_haval256_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce,  uint32_t *d_hash, uint32_t target, uint32_t *result)
{
	const uint32_t threadsperblock = 512;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	CUDA_SAFE_CALL(hipMemsetAsync(d_nonce[thr_id], 0xff, 2 * sizeof(uint32_t), gpustream[thr_id]));

	x17_haval256_gpu_hash_64 <<<grid, block, 0, gpustream[thr_id] >>>(threads, startNounce, (uint64_t*)d_hash, target, d_nonce[thr_id]);
	CUDA_SAFE_CALL(hipMemcpyAsync(result, d_nonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost, gpustream[thr_id]));
	CUDA_SAFE_CALL(hipStreamSynchronize(gpustream[thr_id]));

}
