#include "hip/hip_runtime.h"
/*
 * X15 algorithm (CHC, BBC, X15C)
 * Added in ccminer by Tanguy Pruvot - 2014
 */

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"

#include "sph/sph_hamsi.h"
#include "sph/sph_fugue.h"
#include "sph/sph_shabal.h"
#include "sph/sph_whirlpool.h"
}

#include "miner.h"

#include "cuda_helper.h"

extern void quark_blake512_cpu_init(int thr_id);
extern void quark_blake512_cpu_setBlock_80(int thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_setBlock_80_multi(int thr_id, uint64_t *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void quark_blake512_cpu_hash_80_multi(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void quark_bmw512_cpu_init(int thr_id, uint32_t threads);
extern void quark_bmw512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_groestl512_cpu_init(int thr_id, uint32_t threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_keccak512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void quark_jh512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash);

extern void x11_luffaCubehash512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce,  uint32_t *d_hash);

extern void x11_shavite512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern int  x11_simd512_cpu_init(int thr_id, uint32_t threads);
extern void x11_simd512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, const uint32_t simdthreads);

extern void x11_echo512_cpu_init(int thr_id, uint32_t threads);
extern void x11_echo512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x13_hamsi512_cpu_init(int thr_id, uint32_t threads);
extern void x13_hamsi512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x13_fugue512_cpu_init(int thr_id, uint32_t threads);
extern void x13_fugue512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x14_shabal512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void x15_whirlpool_cpu_free(int thr_id);

extern void quark_compactTest_cpu_init(int thr_id, uint32_t threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, const uint32_t *inpHashes,
											const uint32_t *d_noncesTrue, uint32_t *nrmTrue, uint32_t *d_noncesFalse, uint32_t *nrmFalse);

// X15 CPU Hash function
void x15hash(void *output, const void *input)
{
	sph_blake512_context     ctx_blake;
	sph_bmw512_context       ctx_bmw;
	sph_groestl512_context   ctx_groestl;
	sph_jh512_context        ctx_jh;
	sph_keccak512_context    ctx_keccak;
	sph_skein512_context     ctx_skein;
	sph_luffa512_context     ctx_luffa;
	sph_cubehash512_context  ctx_cubehash;
	sph_shavite512_context   ctx_shavite;
	sph_simd512_context      ctx_simd;
	sph_echo512_context      ctx_echo;
	sph_hamsi512_context     ctx_hamsi;
	sph_fugue512_context     ctx_fugue;
	sph_shabal512_context    ctx_shabal;
	sph_whirlpool_context    ctx_whirlpool;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, hash, 64);
	sph_bmw512_close(&ctx_bmw, hashB);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, hashB, 64);
	sph_groestl512_close(&ctx_groestl, hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, hash, 64);
	sph_skein512_close(&ctx_skein, hashB);

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, hashB, 64);
	sph_jh512_close(&ctx_jh, hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, hash, 64);
	sph_keccak512_close(&ctx_keccak, hashB);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512(&ctx_luffa, hashB, 64);
	sph_luffa512_close(&ctx_luffa, hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, hash, 64);
	sph_cubehash512_close(&ctx_cubehash, hashB);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, hashB, 64);
	sph_shavite512_close(&ctx_shavite, hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hash, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hashB, 64);
	sph_echo512_close(&ctx_echo, hash);

	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, hash, 64);
	sph_hamsi512_close(&ctx_hamsi, hashB);

	sph_fugue512_init(&ctx_fugue);
	sph_fugue512(&ctx_fugue, hashB, 64);
	sph_fugue512_close(&ctx_fugue, hash);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, hash, 64);
	sph_shabal512_close(&ctx_shabal, hashB);

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, hashB, 64);
	sph_whirlpool_close(&ctx_whirlpool, hash);

	memcpy(output, hash, 32);
}

extern int scanhash_x15(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	static THREAD uint32_t *d_hash = nullptr;

	const uint32_t first_nonce = pdata[19];

	int intensity = 256 * 256 * 13;
	if (device_sm[device_map[thr_id]] == 520)  intensity = 256 * 256 * 22;
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, intensity); // 19=256*256*8;
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;
	uint32_t simdthreads = (device_sm[device_map[thr_id]] > 500) ? 256 : 32;

	if (opt_benchmark)
		ptarget[7] = 0x0fF;

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == intensity)
			applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (64 * sizeof(uint4)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		quark_groestl512_cpu_init(thr_id, throughputmax);
		quark_skein512_cpu_init(thr_id);
		quark_bmw512_cpu_init(thr_id, throughputmax);
		x11_simd512_cpu_init(thr_id, throughputmax);
		x11_echo512_cpu_init(thr_id, throughputmax);
		x13_hamsi512_cpu_init(thr_id, throughputmax);
		x13_fugue512_cpu_init(thr_id, throughputmax);
		x15_whirlpool_cpu_init(thr_id, throughputmax, 0);

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughputmax));

		cuda_check_cpu_init(thr_id, throughputmax);
		mining_has_stopped[thr_id] = false;
		init = true;
	}
	
	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, (uint64_t *)endiandata);
	cuda_check_cpu_setTarget(ptarget, thr_id);

	do {
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash);
		x11_luffaCubehash512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash,simdthreads);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19],  d_hash);
		x13_hamsi512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x13_fugue512_cpu_hash_64(thr_id, throughput, pdata[19],  d_hash);
		x14_shabal512_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);

		uint32_t foundNonce = cuda_check_hash(thr_id, throughput, pdata[19], d_hash);
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			/* check now with the CPU to confirm */
			if(opt_verify){ be32enc(&endiandata[19], foundNonce);
			x15hash(vhash64, endiandata);

			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				int res = 1;
				uint32_t secNonce = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash, foundNonce);
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (secNonce != 0)
				{
					if(opt_verify){ be32enc(&endiandata[19], secNonce);
					x15hash(vhash64, endiandata);
					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = secNonce;
						res++;
						if (opt_benchmark)
							applog(LOG_INFO, "GPU #%d: found nounce %08x", device_map[thr_id], secNonce);
					}
					else
					{
						applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], secNonce);
					}
				}
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d: found nounce %08x", device_map[thr_id], foundNonce);
				pdata[19] = foundNonce;
				return res;
			}
			else
			{
				applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;

	return 0;
}
