#include "hip/hip_runtime.h"
/*
 * whirlpool routine (djm)
 * whirlpoolx routine (provos alexis)
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
}
#include "miner.h"


#include "cuda_helper.h"

extern void whirlpoolx_cpu_init(int thr_id, uint32_t threads);
extern void whirlpoolx_setBlock_80(int thr_id, void *pdata, const void *ptarget);
extern void cpu_whirlpoolx(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *foundNonce);
extern void whirlpoolx_precompute(int thr_id);

// CPU Hash function
extern "C" void whirlxHash(void *state, const void *input)
{

	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[64];
	unsigned char hash_xored[32];

	memset(hash, 0, sizeof(hash));

	sph_whirlpool_init(&ctx_whirlpool);
	sph_whirlpool(&ctx_whirlpool, input, 80);
	sph_whirlpool_close(&ctx_whirlpool, hash);

    
	for (uint32_t i = 0; i < 32; i++){
	        hash_xored[i] = hash[i] ^ hash[i + 16];
	}
	memcpy(state, hash_xored, 32);
}

int scanhash_whirlpoolx(int thr_id, uint32_t *pdata, uint32_t *ptarget, uint32_t max_nonce, uint32_t *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, (1 << 27));
	uint32_t throughput = min(throughputmax, max_nonce - first_nonce) & 0xfffffc00;

	if (opt_benchmark)
		ptarget[7] = 0x5;

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == 1<<27)
			applog(LOG_INFO, "GPU #%d: using default intensity 27", device_map[thr_id]);
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		whirlpoolx_cpu_init(thr_id, throughputmax);
		mining_has_stopped[thr_id] = false;
		init = true;
	}

	for (int k=0; k < 20; k++)
	{
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpoolx_setBlock_80(thr_id, (void*)endiandata, &ptarget[6]);
	whirlpoolx_precompute(thr_id);
	do {
		uint32_t foundNonce[2];
		cpu_whirlpoolx(thr_id, throughput, pdata[19], foundNonce);
		CUDA_SAFE_CALL(hipGetLastError());
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundNonce[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			/* check now with the CPU to confirm */
			if(opt_verify){ be32enc(&endiandata[19], foundNonce[0]);
			whirlxHash(vhash64, endiandata);
			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != UINT32_MAX)
				{
					if(opt_verify){ be32enc(&endiandata[19], foundNonce[1]);
					whirlxHash(vhash64, endiandata);
					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundNonce[1];
						res++;
						if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found nonce %08x", device_map[thr_id], foundNonce[1]);
					}
					else
					{
						if (vhash64[7] != Htarg)
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce[1]);
					}
				}
				if (opt_benchmark)
					applog(LOG_INFO, "GPU #%d: found nonce %08x", device_map[thr_id], foundNonce[0], vhash64[7]);
				pdata[19] = foundNonce[0];
				return res;
			}
			else
			{
				if(vhash64[7] != Htarg)
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce[0]);
			}
		}
		pdata[19] += throughput;
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));
	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
