#include "hip/hip_runtime.h"
/*
 * whirlpool routine djm&SP
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
}
#include "miner.h"
#include "cuda_helper.h"

extern void x15_whirlpool_cpu_init(int thr_id, uint32_t threads, int mode);
extern void x15_whirlpool_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce,  uint32_t *d_hash);

extern void whirlpool512_setBlock_80(int thr_id, void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern uint32_t* whirlpool512_cpu_finalhash_64(int thr_id, uint32_t threads, uint32_t startNounce,  uint32_t *d_hash);


// CPU Hash function
void wcoinhash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	// shavite 1
	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, input, 80);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hashB);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hashB, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	memcpy(state, hash, 32);
}

extern int scanhash_whc(int thr_id, uint32_t *pdata,
    uint32_t *ptarget, uint32_t max_nonce,
    uint32_t *hashes_done)
{
	static THREAD uint32_t *d_hash = nullptr;

	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, 1U << 20); // 19=256*256*8;
	uint32_t throughput = min(throughputmax, (max_nonce - first_nonce)) & 0xfffffc00;

	if (opt_benchmark)
		ptarget[7] = 0x0000ff;

	static THREAD volatile bool init = false;
	if(!init)
	{
		if(throughputmax == 1<<20)
			applog(LOG_INFO, "GPU #%d: using default intensity 20", device_map[thr_id]);
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		CUDA_SAFE_CALL(hipDeviceReset());
		CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
#if defined WIN32 && !defined _WIN64
		// 2GB limit for hipMalloc
		if(throughputmax > 0x7fffffffULL / (16 * sizeof(uint32_t)))
		{
			applog(LOG_ERR, "intensity too high");
			mining_has_stopped[thr_id] = true;
			hipStreamDestroy(gpustream[thr_id]);
			proper_exit(2);
		}
#endif

		CUDA_SAFE_CALL(hipMalloc(&d_hash, 16ULL * sizeof(uint32_t) * throughputmax));
		x15_whirlpool_cpu_init(thr_id, throughputmax, 1 /* old whirlpool */);
		mining_has_stopped[thr_id] = false;

		init = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], pdata[k]);
	}

	whirlpool512_setBlock_80(thr_id, (void*)endiandata, ptarget);

	do {
		uint32_t* foundNonce;

		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], d_hash);

		foundNonce = whirlpool512_cpu_finalhash_64(thr_id, throughput, pdata[19],  d_hash);
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundNonce[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8]={0};
			if(opt_verify){ be32enc(&endiandata[19], foundNonce[0]);
			wcoinhash(vhash64, endiandata);
			} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != UINT32_MAX)
				{
					if(opt_verify){ be32enc(&endiandata[19], foundNonce[1]);
					wcoinhash(vhash64, endiandata);
					} if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{

						if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found second nounce %08x", device_map[thr_id], foundNonce[1]);
						pdata[21] = foundNonce[1];
						res++;
					}
					else
					{
						if (vhash64[7] != Htarg)
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce[1]);
					}
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: found nounce %08x", device_map[thr_id], foundNonce[0]);

				return res;
			}
			else
			{
				if (vhash64[7] != Htarg)
					applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce[0]);
			}
		}
		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce ;
	return 0;
}
